#include "hip/hip_runtime.h"
/*******************************************************************************
 * Copyright (c) 2015-2018 Skymind, Inc.
 *
 * This program and the accompanying materials are made available under the
 * terms of the Apache License, Version 2.0 which is available at
 * https://www.apache.org/licenses/LICENSE-2.0.
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS, WITHOUT
 * WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied. See the
 * License for the specific language governing permissions and limitations
 * under the License.
 *
 * SPDX-License-Identifier: Apache-2.0
 ******************************************************************************/

#ifndef NDARRAY_CPP
#define NDARRAY_CPP

#include "../NDArray.h"
#include "../NDArrayFactory.h"
#include "NativeOpExecutioner.h"
#include <memory/Workspace.h>
#include <memory/MemoryRegistrator.h>
#include <ops.h>
#include <ops/gemm.h>
#include <pointercast.h>
#include <stdexcept>
#include <memory>
#include <helpers/logger.h>
#include <loops/pairwise_transform.h>
#include <loops/transform_same.h>
#include <loops/random.h>
#include <loops/broadcasting.h>
#include <indexing/NDIndex.h>
#include <indexing/IndicesList.h>
#include <helpers/ShapeUtils.h>
#include <sstream>
#include <helpers/ArrayUtils.h>
#include <MmulHelper.h>
#include <helpers/threshold.h>
#include <exceptions/datatype_exception.h>
#include <exceptions/cuda_exception.h>
#include <specials_cuda.h>
#include <loops/special_kernels.h>
#include "../NDArray.hpp"

namespace nd4j {

////////////////////////////////////////////////////////////////////////
void* NDArray::operator new(size_t i) {
    if (nd4j::memory::MemoryRegistrator::getInstance()->hasWorkspaceAttached()) {
        nd4j::memory::Workspace* ws = nd4j::memory::MemoryRegistrator::getInstance()->getWorkspace();
        return ws->allocateBytes((Nd4jLong) i);
    } else {
        auto p = malloc(i);
        
        CHECK_ALLOC(p, "Failed to allocate new NDArray", i);
        return p;
    }
}

////////////////////////////////////////////////////////////////////////
void NDArray::operator delete(void* p) {
    
    if (!nd4j::memory::MemoryRegistrator::getInstance()->hasWorkspaceAttached())
        free(p);
}


////////////////////////////////////////////////////////////////////////
// copy constructor
NDArray::NDArray(const NDArray& other) {
    
    _context = other._context;    

    setShapeInfo(ShapeBuilders::copyShapeInfo(other._shapeInfo, false, _context->getWorkspace()));
    _isShapeAlloc = true;

    ALLOCATE_SPECIAL(_bufferD, _context->getWorkspace(), _length * sizeOfT(), int8_t);
    _isBuffDAlloc = true;

    if(other.isActualOnHostSide()) {
        auto res = hipMemcpy(_bufferD, other._buffer, _length * sizeOfT(), hipMemcpyHostToDevice);
        if (res != 0)
            throw cuda_exception::build("hipMemcpy failed", res);
    } else {
        auto res = hipMemcpy(_bufferD, other._bufferD, _length * sizeOfT(), hipMemcpyDeviceToDevice);
        if (res != 0)
            throw cuda_exception::build("hipMemcpy failed", res);
    }        

    tickWriteDevice();
}

////////////////////////////////////////////////////////////////////////
void NDArray::lazyAllocateBuffer() const {
    
    if (lengthOf() > 0) {
        NDArray* constThis = const_cast<NDArray*>(this);
        if (_buffer == nullptr && !this->isEmpty()) {
            //int8_t* pB = const_cast<int8_t*>(_buffer);
            ALLOCATE(constThis->_buffer, _context->getWorkspace(), this->lengthOf() * this->sizeOfT(), int8_t);
            //const_cast<NDArray*>(this)->_buffer = pB;
            constThis->_isBuffAlloc = true;
            syncToHost();
        }
    }
}   

////////////////////////////////////////////////////////////////////////
// scalar constructor
NDArray::NDArray(nd4j::DataType dtype, nd4j::graph::LaunchContext* context) {

    setShapeInfo(ShapeBuilders::createScalarShapeInfo(dtype, context->getWorkspace()));

    ALLOCATE_SPECIAL(_bufferD, context->getWorkspace(), sizeOfT(), int8_t);
    _isBuffDAlloc = true;
    hipMemset(_bufferD, 0, sizeOfT());    
    
    tickWriteDevice();
}

////////////////////////////////////////////////////////////////////////
// creates new NDArray using shape information from "shapeInfo" array, set all elements in new array to be zeros
NDArray::NDArray(Nd4jLong* shapeInfo, const nd4j::DataType dtype, const bool copyStrides, nd4j::graph::LaunchContext* context, const bool isShapeAlloc) {
    
    if (shapeInfo == nullptr)
        throw std::runtime_error("NDArray constructor: can't be initalized without shapeinfo");

    if ((int) shapeInfo[0] > MAX_RANK)
        throw std::invalid_argument("Rank of NDArray can't exceed 32");

    _context = context;        

    if(!isShapeAlloc) {
        setShapeInfo(ShapeBuilders::copyShapeInfo(shapeInfo, copyStrides, _context->getWorkspace()), dtype);
    }
    else {
        setShapeInfo(shapeInfo, dtype);        
        if(!copyStrides)
            shape::updateStrides(_shapeInfo, shape::order(shapeInfo));         
    }

    _isShapeAlloc = true;
   
    ALLOCATE_SPECIAL(_bufferD, _context->getWorkspace(), _length * sizeOfT(), int8_t);
    hipMemset(_bufferD, 0, _length * sizeOfT());
    _isBuffDAlloc = true;        
   
    tickWriteDevice();
}

////////////////////////////////////////////////////////////////////////
NDArray::NDArray(const char order, const std::vector<Nd4jLong> &shape, const std::vector<double>& data, nd4j::DataType dtype, nd4j::graph::LaunchContext* context) {

    if (shape.empty())
        throw std::runtime_error("NDArray constructor: input shape is empty !");

    if ((int) shape.size() > MAX_RANK)
        throw std::invalid_argument("Rank of NDArray can't exceed 32");

    _context = context;

    setShapeInfo(ShapeBuilders::createShapeInfo(dtype, order, shape, _context->getWorkspace()));
    _isShapeAlloc = true;

    if (_length != data.size()) {
        nd4j_printf("NDArray constructor: data size [%i] doesn't match shape length [%i]\n", data.size(), _length);
        throw std::runtime_error("Data size doesn't match shape");
    }

    ALLOCATE(_buffer, _context->getWorkspace(), _length * DataTypeUtils::sizeOf(dtype), int8_t);
    _isBuffAlloc = true;
    
    ALLOCATE_SPECIAL(_bufferD, _context->getWorkspace(), _length * DataTypeUtils::sizeOf(dtype), int8_t);
    _isBuffDAlloc = true;    

    for(Nd4jLong i=0; i < _length; ++i) {
        BUILD_SINGLE_PARTIAL_SELECTOR(dtype, templatedDoubleAssign<, double>(_buffer, i, reinterpret_cast<const void *>(data.data()), i), LIBND4J_TYPES);
    }
        
    syncToDevice();
    tickReadHost();
}

////////////////////////////////////////////////////////////////////////
NDArray::NDArray(const char order, const std::vector<Nd4jLong> &shape, nd4j::DataType dtype, nd4j::graph::LaunchContext* context) {

    if (shape.empty())
        throw std::runtime_error("NDArray constructor: input shape is empty !");

    if ((int) shape.size() > MAX_RANK)
        throw std::invalid_argument("Rank of NDArray can't exceed 32");

    _context = context;

    setShapeInfo(ShapeBuilders::createShapeInfo(dtype, order, shape, _context->getWorkspace()));
    _isShapeAlloc = true;

    ALLOCATE_SPECIAL(_bufferD, _context->getWorkspace(), _length * sizeOfT(), int8_t);
    hipMemset(_bufferD, '\0', _length * sizeOfT()); // zero all memory
    _isBuffDAlloc = true;    

    tickWriteDevice();
}

////////////////////////////////////////////////////////////////////////
NDArray::NDArray(const NDArray *other, const bool copyStrides, nd4j::graph::LaunchContext* context) {

    _context = context;
    
    setShapeInfo(ShapeBuilders::copyShapeInfo(other->_shapeInfo, copyStrides, _context->getWorkspace()));
    _isShapeAlloc = true;
    
    ALLOCATE_SPECIAL(_bufferD, _context->getWorkspace(), _length * sizeOfT(), int8_t);
    _isBuffDAlloc = true;    

    tickWriteDevice();
}

////////////////////////////////////////////////////////////////////////
NDArray::NDArray(void* buffer, const char order, const std::vector<Nd4jLong> &shape,  nd4j::DataType dtype, nd4j::graph::LaunchContext* context) {
    
    if (shape.empty())
        throw std::runtime_error("NDArray constructor: input shape is empty !");
        
    if ((int) shape.size() > MAX_RANK)
        throw std::invalid_argument("Rank of NDArray can't exceed 32");
    
    _context = context;

    setShapeInfo(ShapeBuilders::createShapeInfo(dtype, order, shape, _context->getWorkspace()));
    _isShapeAlloc = true;

    _buffer = reinterpret_cast<int8_t *>(buffer);
    
    ALLOCATE_SPECIAL(_bufferD, _context->getWorkspace(), _length * sizeOfT(), int8_t);
    _isBuffDAlloc = true;
    
    if(_buffer != nullptr)
        hipMemcpy(_bufferD, _buffer, _length * sizeOfT(), hipMemcpyHostToDevice);
        
    tickWriteDevice();
    tickReadHost(); 
}

////////////////////////////////////////////////////////////////////////
// assignment operator
    NDArray& NDArray::operator=(const NDArray& other) {

    if (this == &other)
        return *this;    

    if (shape::equalsSoft(_shapeInfo, other._shapeInfo) && _dataType == other._dataType) {
        if(!isEmpty())
            this->assign(&other);
    }
    else {
        
        if(_context->getWorkspace() == nullptr) {
            
            if(_isBuffAlloc) delete []_buffer;
            if(_isShapeAlloc) delete []_shapeInfo;

            if(_isBuffDAlloc)  RELEASE_SPECIAL(_bufferD, nullptr);
            if(_isShapeDAlloc) RELEASE_SPECIAL(_shapeInfoD, nullptr);
        }
               
        _context= other._context;
        _buffer = nullptr;
              
        setShapeInfo(ShapeBuilders::copyShapeInfo(other._shapeInfo, false, _context->getWorkspace()));    
        _isShapeAlloc = true;

        ALLOCATE_SPECIAL(_bufferD, _context->getWorkspace(), _length * sizeOfT(), int8_t);
        _isBuffDAlloc = true;        
                
        this->assign(&other);
    }

    return *this;
}

//////////////////////////////////////////////////////////////////////////
// perform array transformation
    // void NDArray::applyTransform(nd4j::transform::FloatOps op, void *extraParams) {
    //     applyTransform(op, this, extraParams);
    // }

    // void NDArray::applyTransform(nd4j::transform::AnyOps op, void *extraParams) {
    //     applyTransform(op, this, extraParams);
    // }

    // void NDArray::applyTransform(nd4j::transform::SameOps op, void *extraParams) {
    //     applyTransform(op, this, extraParams);
    // }

    // void NDArray::applyTransform(nd4j::transform::BoolOps op, void *extraParams) {
    //     applyTransform(op, this, extraParams);
    // }

    // void NDArray::applyTransform(nd4j::transform::StrictOps op, void *extraParams) {
    //     applyTransform(op, this, extraParams);
    // }

    // perform array transformation

/*
    template<typename T>
    template<typename OpName>
    void NDArray<T>::applyRandom(nd4j::random::RandomBuffer *buffer, NDArray<T>* y, NDArray<T>* z, T* extraArgs) {
        Nd4jPointer state = (Nd4jPointer) buffer;
        if (y == nullptr && z == nullptr) {
            // we're executing indexed z here
            functions::random::RandomFunction<T>::template execTransform<OpName>(state, this->buffer(), this->shapeInfo(), extraArgs);
        } else if (y == nullptr && z != nullptr) {
            // XZ case
            functions::random::RandomFunction<T>::template execTransform<OpName>(state, this->buffer(), this->shapeInfo(), z->buffer(), z->shapeInfo(), extraArgs);
        } else if (y != nullptr && z != nullptr) {
            // XYZ case
            functions::random::RandomFunction<T>::template execTransform<OpName>(state, this->buffer(), this->shapeInfo(), y->buffer(), y->shapeInfo(), z->buffer(), z->shapeInfo(), extraArgs);
        }
    }
    */

    //////////////////////////////////////////////////////////////////////////
    void NDArray::applyTrueBroadcast(nd4j::BroadcastBoolOpsTuple op, const NDArray* other, NDArray* target, const bool checkTargetShape, ExtraArguments *extraArgs) const {
        if (isS())
            throw std::runtime_error("NDArray::applyTrueBroadcast bool: you can't use this method on String array!");
        if(target == nullptr || other == nullptr)
            throw std::runtime_error("NDArray::applyTrueBroadcast bool method: target or other = nullptr !");

        NDArray::prepareSpecialUse({target}, {this, other});

        if (isScalar()) {
            NDArray temp(target->_shapeInfo, _dataType, false, _context);
            temp.assign(this);
            temp.applyPairwiseTransform(op.p, other, target,  extraArgs);
            return;
        }
        if (other->isScalar()) {
            this->applyScalarArr(op.s, other, target, extraArgs);
            return;
        }

        const NDArray* min(nullptr), *max(nullptr);
        if(this->rankOf() >= other->rankOf()) {
            max = this;
            min = other;
        }
        else {
            max = other;
            min = this;
        }

        if(checkTargetShape) {
            Nd4jLong* newShapeInfo = nullptr;
            if(!ShapeUtils::evalBroadcastShapeInfo(*max, *min, false, newShapeInfo, _context->getWorkspace()))          // the rank of target array must be equal to max->rankOf)()
                throw std::runtime_error("NDArray::applyTrueBroadcast method: the shapes of this and other arrays are not suitable for broadcast operation !");
            if(!shape::equalsSoft(target->_shapeInfo, newShapeInfo) || target->_dataType != DataType::BOOL)
                throw std::runtime_error("NDArray::applyTrueBroadcast bool method: the shape or type of target array is wrong !");
            if(_dataType != other->_dataType)
                throw std::invalid_argument("NDArray::applyTrueBroadcast bool method: this and other arrays must have the same type !");

            // if workspace is not null - do not call delete.
            if (_context->getWorkspace() == nullptr)
                delete[] newShapeInfo;
        }

        NDArray* pTarget = (max->_dataType == target->_dataType) ? target : new NDArray(target->ordering(), target->getShapeAsVector(), max->_dataType, target->_context);
        // check whether max array has to be tiled
        if(!max->isSameShape(target)) {
            // evaluate repeating dimensions for tile operation
            std::vector<Nd4jLong> repeatMax(max->rankOf());
            for(int i = 1; i <= max->rankOf(); ++i)
                repeatMax[i-1] = (target->_shapeInfo[i] / max->_shapeInfo[i]);
            max->tile(repeatMax, *pTarget);
        }
        else
            pTarget->assign(max);

        // check whether min array has to be tiled
        std::vector<Nd4jLong> repeatMin(min->rankOf());
        int product = 1;
        for(int i = min->rankOf(); i >=1 ; --i) {
            repeatMin[i-1] = (target->_shapeInfo[target->rankOf() - min->rankOf() + i] / min->_shapeInfo[i]);
            product *= repeatMin[i-1];
        }

        auto pMin = const_cast<NDArray *>(min);
        if(product != 1 )
            pMin = new NDArray(min->tile(repeatMin));


        std::vector<int> sameDims = ShapeUtils::getDimsWithSameShape(*target, *pMin);

        if(max == this) {
            pTarget->applyBroadcast(op.b, sameDims, pMin, target, extraArgs);
        }
        else {
            auto dimsToExclude = ShapeUtils::evalDimsToExclude(target->rankOf(), sameDims);
            const auto numOfSubArrs = ShapeUtils::getNumOfSubArrs(target->_shapeInfo, dimsToExclude);

            for(Nd4jLong i = 0; i < numOfSubArrs; ++i) {
                NDArray targetSubArr = (*target)(i, dimsToExclude);
                if (pTarget == target)
                    pMin->applyPairwiseTransform(op.p, &targetSubArr, &targetSubArr, extraArgs);
                else {
                    NDArray pTargetSubArr = (*pTarget)(i, dimsToExclude);
                    pMin->applyPairwiseTransform(op.p, &pTargetSubArr, &targetSubArr, extraArgs);
                }
            }
        }

        if(pMin != min)
            delete pMin;
        if(pTarget != target)
            delete pTarget;
    }

    //////////////////////////////////////////////////////////////////////////
    void NDArray::applyTrueBroadcast(nd4j::BroadcastOpsTuple op, const NDArray* other, NDArray* target, const bool checkTargetShape, ExtraArguments *extraArgs) const {
        if (isS())
            throw std::runtime_error("NDArray::applyTrueBroadcast: you can't use this method on String array!");
        if(target == nullptr || other == nullptr)
            throw std::runtime_error("NDArray::applyTrueBroadcast method: target or other = nullptr !");
        if(((op.s == scalar::Divide || op.s == scalar::FloorDiv || op.s == scalar::FloorMod) && other->isB()) || (op.s == scalar::ReverseDivide && this->isB()))
            throw std::runtime_error("NDArray::applyTrueBroadcast method: you can't divide by bool array !");


        NDArray::prepareSpecialUse({target}, {this, other});

        if (isScalar()) {
            target->assign(this);
            target->applyPairwiseTransform(op.p, *other, extraArgs);
            return;
        }
        if (other->isScalar()) {
            const_cast<NDArray*>(this)->applyScalarArr(op.s, other, target, extraArgs);
            return;
        }

        const NDArray* min(nullptr), *max(nullptr);
        if(this->rankOf() >= other->rankOf()) {
            max = this;
            min = other;
        }
        else {
            max = other;
            min = this;
        }

        if(checkTargetShape) {
            Nd4jLong* newShapeInfo = nullptr;
            if(!ShapeUtils::evalBroadcastShapeInfo(*max, *min, false, newShapeInfo, _context->getWorkspace()))          // the rank of target array must be equal to max->rankOf)()
                throw std::runtime_error("NDArray::applyTrueBroadcast method: the shapes of this and other arrays are not suitable for broadcast operation !");
            if(!shape::equalsTypesAndShapesSoft(target->getShapeInfo(), newShapeInfo))
                throw std::runtime_error("NDArray::applyTrueBroadcast method: the shape or type of target array is wrong !");
            shape::printShapeInfo(newShapeInfo);
            // if workspace is not null - do not call delete.
            if (_context->getWorkspace() == nullptr)
                delete[] newShapeInfo;
        }

        NDArray* pTarget = (max->_dataType == target->_dataType) ? target : new NDArray(target->ordering(), target->getShapeAsVector(), max->_dataType, target->_context);
        // check whether max array has to be tiled
        if(!max->isSameShape(target)) {
            // evaluate repeating dimensions for tile operation
            std::vector<Nd4jLong> repeatMax(max->rankOf());
            for(int i = 1; i <= max->rankOf(); ++i) {
                repeatMax[i - 1] = (target->_shapeInfo[i] / max->_shapeInfo[i]);
                //nd4j_printf("repeatMax[%i] = %i\n", i - 1, repeatMax[i - 1]);
            }
            max->tile(repeatMax, *pTarget);
        }
        else
            pTarget->assign(max);

        // check whether min array has to be tiled
        std::vector<Nd4jLong> repeatMin(min->rankOf());
        int product = 1;
        for(int i = min->rankOf(); i >=1 ; --i) {
            repeatMin[i-1] = (target->_shapeInfo[target->rankOf() - min->rankOf() + i] / min->_shapeInfo[i]);
            product *= repeatMin[i-1];
        }
        auto pMin = const_cast<NDArray *>(min);
        if(product != 1 ) {
            auto localMin = min->tile(repeatMin);
            pMin = new NDArray(localMin);
        }

        std::vector<int> sameDims = ShapeUtils::getDimsWithSameShape(*target, *pMin);
        //max->syncToDevice();
        //pMin->syncToDevice(); // tile has a problem with syncing data to device
        //pMin->printBuffer("MIN BUFFER");
        //min->printBuffer("Min buffer");
        //max->printBuffer("MAX BUFFER");
//        if (sameDims.size() == max->rankOf()) {
//            target->syncToDevice();
//            max->applyPairwiseTransform(op.p, pMin, target, extraArgs);
//            target->printBuffer("TARGET");
//        }
        if(max == this) {
            pTarget->applyBroadcast(op.b, sameDims, pMin, target, extraArgs);
        }
        else {
            auto dimsToExclude = ShapeUtils::evalDimsToExclude(target->rankOf(), sameDims);
            const auto numOfSubArrs = ShapeUtils::getNumOfSubArrs(target->_shapeInfo, dimsToExclude);

            for(Nd4jLong i = 0; i < numOfSubArrs; ++i) {
                auto targetSubArr = (*target)(i, dimsToExclude);
                if(pTarget == target)
                    pMin->applyPairwiseTransform(op.p, &targetSubArr, &targetSubArr, extraArgs);
                else {
                    auto pTargetSubArr = (*pTarget)(i, dimsToExclude);
                    pMin->applyPairwiseTransform(op.p, &pTargetSubArr, &targetSubArr, extraArgs);
                }
            }
        }

        if(pMin != min)
            delete pMin;
         if(pTarget != target)
            delete pTarget;
    }

    //////////////////////////////////////////////////////////////////////////
    // return array which is broadcasted from this and argument array
    NDArray* NDArray::broadcast(const NDArray& other) {
	    // the orders must be the same
	    char order = ordering();
	    if(order != other.ordering())
		    throw std::runtime_error("Broadcast method: arrays have different orders!");

	    // recognize shapes with smaller and bigger rank
	    Nd4jLong* biggerShapeInfo = nullptr;
	    Nd4jLong* smallerShapeInfo = nullptr;
	    int smallerRank, biggerRank;
	    if (rankOf() > other.rankOf()) {
		    biggerShapeInfo = _shapeInfo;
		    biggerRank = shape::rank(_shapeInfo);
		    smallerShapeInfo = other._shapeInfo;
		    smallerRank = shape::rank(other._shapeInfo);
	    }
	    else {
		    biggerShapeInfo = other._shapeInfo;
		    biggerRank = shape::rank(other._shapeInfo);
		    smallerShapeInfo = _shapeInfo;
		    smallerRank = shape::rank(_shapeInfo);
	    }

	    // check shapes on consistency
	    int diff = biggerRank - smallerRank;
	    for (int i = smallerRank; i<=1; --i)
		    if(biggerShapeInfo[diff+i] != smallerShapeInfo[i] && biggerShapeInfo[i] != 1 && smallerShapeInfo[i] != 1)
			    throw std::runtime_error("Broadcast method: arrays have incompatible shapes !");

		// create and fill ret shapeInfo
	    auto shapeInfoNew = new Nd4jLong[shape::shapeInfoLength(biggerRank)];
	    memcpy(shapeInfoNew, biggerShapeInfo, shape::shapeInfoByteLength(biggerRank));
	    for (int i = smallerRank; i>=1; --i)
		    if(shapeInfoNew[diff+i] == 1 || smallerShapeInfo[i] == 1)
			    shapeInfoNew[diff+i] *= smallerShapeInfo[i];

	    auto ret = new NDArray(shapeInfoNew, true, _context);
        ShapeUtils::updateStridesAndType(ret->getShapeInfo(), DataTypeUtils::pickPairwiseResultType(_dataType, other._dataType), order);
	    delete []shapeInfoNew;

    	return ret;
    }


    //////////////////////////////////////////////////////////////////////////
    // check whether array's rows (arg=0) or columns (arg=1) create orthogonal basis
    bool NDArray::hasOrthonormalBasis(const int arg) {
        if (isS())
            throw std::runtime_error("NDArray::hasOrthonormalBasis: you can't use this method on String array!");
	    if(rankOf() !=2 )
		    throw std::runtime_error("NDArray::hasOrthBasis method: rank of ndarray is not equal 2 !");

	    if(arg!=0  && arg!=1)
		    throw std::runtime_error("NDArray::hasOrthBasis method: input argument is not equal to 0 or 1 !");

	    const double eps = 1e-5;
        double dot = 0.f;

        if(arg) {					// check whether columns create orthogonal basis
		    for(int j=0; j<columns()-1; ++j)
			    for(int k=j+1; k<columns(); ++k) {
				    for(int i=0; i<rows(); ++i)
					    dot += e<double>(i,j)*e<double>(i,k);

				    if(nd4j::math::nd4j_abs(dot) > eps )
					    return false;

				    dot = 0.f;
			    }

			    for(int j=0; j<columns(); ++j)	{	// check whether norm of column vector = 1
			        for(int i=0; i<rows(); ++i)
				        dot += e<double>(i,j)*e<double>(i,j);
			    if(dot != 0.f && nd4j::math::nd4j_abs(nd4j::math::nd4j_sqrt<double, double>(dot) - 1.f) > eps)
				    return false;

			    dot = 0.f;
		    }
	    }
	    else {						// check whether rows create orthogonal basis
		    for(int i=0; i<rows()-1; ++i)
			    for(int k=i+1; k<rows(); ++k) {
				    for(int j=0; j<columns(); ++j)
					    dot += e<double>(i,j)*e<double>(k,j);

				    if(nd4j::math::nd4j_abs(dot) > eps )
					    return false;

				    dot = 0.;
			    }

		        for(int i=0; i<rows(); ++i) {		// check whether norm of row vector = 1
			        for(int j=0; j<columns(); ++j)
					    dot += e<double>(i,j)*e<double>(i,j);

			        if(dot!= 0. && nd4j::math::nd4j_abs(nd4j::math::nd4j_sqrt<double, double>(dot) - 1.) > eps)
				        return false;
			        dot = 0.;
		        }
	        }
	    return true;
    }

    template <typename T>
    std::vector<T> NDArray::asVectorT() {
        std::vector<T> result(this->lengthOf());

#pragma omp parallel for simd
        for (int e = 0; e < this->lengthOf(); e++)
            result[e] = this->e<T>(e);

        return result;
    }
    BUILD_SINGLE_TEMPLATE(template std::vector, NDArray::asVectorT(), LIBND4J_TYPES);

    ////////////////////////////////////////////////////////////////////////
    template<typename T>
    void NDArray::setValueInDiagMatrix(const T& value, const int diag, const char direction) {
        if (isS())
            throw std::runtime_error("NDArray::setValueInDiagMatrix: you can't use this method on String array!");
        if(rankOf() != 2)
           throw std::runtime_error("NDArray::setValueInDiagMatrix method: array must have rank = 2, but got " + toStringValue(rankOf()) + " instead !");
        hipStream_t* stream = _context->getCudaStream();
        const auto rows = sizeAt(0);
        const auto cols = sizeAt(1);
        if (!isActualOnDeviceSide())
            syncToDevice();
        NDArray val = NDArrayFactory::create(value, _context);
        switch(direction) {
            case 'u':                           // fill upper triangular block
                BUILD_SINGLE_SELECTOR(_dataType, setDiagonalValueUpper, ((void*)_bufferD, _shapeInfoD, val, diag, rows, cols,  *stream), LIBND4J_TYPES);
                break;

            case 'l':                           // fill lower triangular block
                BUILD_SINGLE_SELECTOR(_dataType, setDiagonalValueLower, ((void*)_bufferD, _shapeInfoD, val, diag, rows, cols, *stream), LIBND4J_TYPES);
                break;
            default:
                throw std::string("NDArray::setValueInDiagMatrix method: wrong value of direction argument, expected is 'u' or 'l', but got " + std::string(1,direction) + " instead !");
        }
        tickWriteDevice();
    }
    template void NDArray::setValueInDiagMatrix(const double& value, const int diag, const char direction);
    template void NDArray::setValueInDiagMatrix(const float& value, const int diag, const char direction);
    template void NDArray::setValueInDiagMatrix(const float16& value, const int diag, const char direction);
    template void NDArray::setValueInDiagMatrix(const bfloat16& value, const int diag, const char direction);
    template void NDArray::setValueInDiagMatrix(const Nd4jLong& value, const int diag, const char direction);
    template void NDArray::setValueInDiagMatrix(const int& value, const int diag, const char direction);
    template void NDArray::setValueInDiagMatrix(const int16_t& value, const int diag, const char direction);
    template void NDArray::setValueInDiagMatrix(const uint8_t& value, const int diag, const char direction);
    template void NDArray::setValueInDiagMatrix(const int8_t& value, const int diag, const char direction);
    template void NDArray::setValueInDiagMatrix(const bool& value, const int diag, const char direction);

    


    //////////////////////////////////////////////////////////////////////////
// set new order and shape in case of suitable array length
    bool NDArray::reshapei(const char order, const std::vector<Nd4jLong>& cshape) {

        // check firstly whether cshape is identical to shape of array, if yes then reshape is unnecessary
        if(order == ordering() && rankOf() == cshape.size()) {
            bool areShapesSame = true;
            for(int i = 0; i < cshape.size(); ++i)
                if(cshape[i] != sizeAt(i)) {
                    areShapesSame = false;
                    break;
                }
            if(areShapesSame)
                return areShapesSame;
        }

        std::vector<Nd4jLong> shape(cshape);
        int rank = shape.size();

        // looking for negative in shape

        int numberNegativesOnes = 0;

        Nd4jLong* shape_ = shape.data();
        for (int i = 0; i < (int) shape.size(); i++) {
            if (shape[i] < 0) {
                if (numberNegativesOnes >= 1)
                    throw std::runtime_error("Only one dimension can be negative at once");

                numberNegativesOnes++;

                int shapeLength = 1;
                for (int j = 0; j < (int) shape.size(); j++)
                    if (i != j)
                        shapeLength *= shape_[j];

                Nd4jLong realShape = nd4j::math::nd4j_abs<int>(lengthOf() / shapeLength);
                auto thisNewShape = new Nd4jLong[shape.size()];

                for (int j = 0; j < (int) shape.size(); j++)
                    if (i != j)
                        thisNewShape[j] = shape_[j];
                    else
                        thisNewShape[j] = realShape;

                shape_ = thisNewShape;
            }
        }

        for (int e = 0; e < (int) shape.size(); e++)
            shape[e] = shape_[e];

        if (numberNegativesOnes > 0)
            delete[] shape_;

        int arrLength = 1;
        for(const auto& item : shape)
            arrLength *= item;

        if(_bufferD==nullptr || arrLength != this->lengthOf()) {
            this->printShapeInfo("Mismatched shape");
            nd4j::Logger::printv("Shape requested: ", shape);
            nd4j_debug("Requested length in reshape: %i; Existing length: %i;\n", arrLength, this->lengthOf());
            throw std::runtime_error("Bad shape!");
        }

        int shapeLength = shape::shapeInfoLength(rank);
        // remember old values

        // we can do this only if there was no permute applied, or there are no weird strides
        if (shape::canReshape(this->rankOf(), this->_shapeInfo, shape.size(), shape.data(), order == 'f')) {                        
            Nd4jLong *shapeInfoNew;            
            ALLOCATE(shapeInfoNew, _context->getWorkspace(), shape::shapeInfoLength(rank), Nd4jLong);            
            shape::reshapeCF(this->rankOf(), this->_shapeInfo, shape.size(), shape.data(), order == 'f', shapeInfoNew);            
            
            setShapeInfo(shapeInfoNew, dataType());      
            _isShapeAlloc = true;
        } 
        else {
            Nd4jLong *shapeInfoNew = ShapeBuilders::createShapeInfo(dataType(), order, shape, _context->getWorkspace());
            NDArray temp(shapeInfoNew, true, _context, true);                    
            this->applyTransform(transform::Copy, &temp, nullptr);            
            temp.tickWriteDevice();
            *this = std::move(temp);
        }
        return true;
    }

    ////////////////////////////////////////////////////////////////////////
    void NDArray::setIdentity() {
        if (isS())
            throw std::runtime_error("NDArray::setIdentity: you can't use this method on String array!");

        if (rankOf() != 2)
            throw std::runtime_error("NDArray::setIdentity: method should work only for 2D tensors. But " + toStringValue(rankOf()) + " was given.");

        this->assign(1.);

        setValueInDiagMatrix(0.f, 1, 'u');
        setValueInDiagMatrix(0.f, -1, 'l');

        //setValueInDiagMatrix(0.f, 1, 'l');
//#pragma omp parallel for if(minDim > Environment::getInstance()->elementwiseThreshold()) schedule(guided)
//        for(int i = 0; i < minDim; ++i)
//            templatedSet<float>(_buffer, i*offset, this->dataType(), &v);
    }

    template <typename T>
    void NDArray::templatedSet(void *buffer, const Nd4jLong xOfsset, nd4j::DataType dtype, const void *value) {
        BUILD_SINGLE_PARTIAL_SELECTOR(dtype, templatedSet< , T>(buffer, xOfsset, value), LIBND4J_TYPES);
    }
    BUILD_SINGLE_TEMPLATE(template void NDArray::templatedSet, (void *buffer, const Nd4jLong xOfsset, nd4j::DataType dtype, const void *value), LIBND4J_TYPES);



    template <typename T>
    void NDArray::templatedSwap(void *xBuffer, void *yBuffer, Nd4jLong length) {
        auto x = reinterpret_cast<T *>(xBuffer);
        auto y = reinterpret_cast<T *>(yBuffer);

#pragma omp parallel for simd schedule(static)
        for (int i = 0; i < length; ++i) {
            auto temp = x[i];
            x[i] = y[i];
            y[i] = temp;
        }
    }
    BUILD_SINGLE_TEMPLATE(template void NDArray::templatedSwap, (void *xBuffer, void *yBuffer, Nd4jLong length), LIBND4J_TYPES);

    ////////////////////////////////////////////////////////////////////////////////////////////////////////////////////
    void NDArray::swapUnsafe(NDArray& other) {
        auto xType = this->dataType();

        if (xType != other.dataType())
            throw std::runtime_error("NDArray::swapUnsage method: both arrays must have the same data type");

        if(_buffer == nullptr || other._buffer == nullptr)
            throw std::runtime_error("NDArray::swapUnsafe method: input array should not be empty!");

        // if(_buffer == other._buffer)
        //     throw std::runtime_error("NDArray::swapUnsafe method: the buffers of input arrays should not point on the same address!");

        if(lengthOf() != other.lengthOf())
            throw std::runtime_error("NDArray::swapUnsafe method: input arrays should have the same length!");

        BUILD_SINGLE_SELECTOR(xType, templatedSwapUnsafe, (this->_bufferD, this->_shapeInfoD, other.specialBuffer(), other.specialShapeInfo(), _context->getCudaStream()), LIBND4J_TYPES);
    }

    void NDArray::streamline(char o) {
        char order = o == 'a' ? this->ordering() : o;
        if (!isActualOnDeviceSide())
            syncToDevice();

        Nd4jLong *newShape;
        Nd4jLong* newShapeD;
        Nd4jLong rank = this->rankOf();
        ALLOCATE(newShape, this->_context->getWorkspace(), shape::shapeInfoLength(rank), Nd4jLong);
        ALLOCATE_SPECIAL(newShapeD, this->_context->getWorkspace(), shape::shapeInfoLength(rank), Nd4jLong);

        int8_t *newBuffer = nullptr;
        int8_t* newBufferD;
        ///ALLOCATE(newBuffer, this->_context->getWorkspace(), this->lengthOf() * sizeOfT(), int8_t);
        ALLOCATE_SPECIAL(newBufferD, this->_context->getWorkspace(), this->lengthOf() * sizeOfT(), int8_t);

        std::vector<Nd4jLong> shape(this->rankOf());
        for (int e = 0; e < this->rankOf(); e++)
            shape[e] = this->sizeAt(e);

        if (order == 'c')
            shape::shapeBuffer(this->rankOf(), dataType(), shape.data(), newShape);
        else
            shape::shapeBufferFortran(this->rankOf(), dataType(), shape.data(), newShape);

        hipMemcpy(newShapeD, newShape, shape::shapeInfoByteLength(rank), hipMemcpyHostToDevice); // move data from host shape to device
        if (!isView()) {
            NativeOpExecutioner::execTransformSame(_context, transform::Copy, _buffer, _shapeInfo, _bufferD, _shapeInfoD, newBuffer, newShape, newBufferD, newShapeD, nullptr, nullptr, nullptr);
            //memcpy(_buffer, newBuffer, this->lengthOf() * sizeOfT());

            if (_isBuffAlloc) {
                RELEASE(this->_buffer, this->_context->getWorkspace());
            }
            if (_isBuffDAlloc) {
                RELEASE_SPECIAL(_bufferD, this->_context->getWorkspace());
            }
            if (_isShapeAlloc)
                RELEASE(this->_shapeInfo, this->_context->getWorkspace());

            if (_isShapeDAlloc)
                RELEASE_SPECIAL(_shapeInfoD, this->_context->getWorkspace());

            _buffer == nullptr;
            _shapeInfo = newShape;
            setSpecialBuffers(newBufferD, newShapeD);
            //this->_buffer = newBuffer;
            this->_isBuffAlloc = false;
            this->_isBuffDAlloc = true;
            this->_isShapeDAlloc = true;
            this->_isShapeAlloc = true;
        } else {
            NativeOpExecutioner::execTransformSame(_context, transform::Copy, _buffer, _shapeInfo, _bufferD, _shapeInfoD, newBuffer, newShape, newBufferD, newShapeD, nullptr, nullptr, nullptr);

            if (_isBuffAlloc)
                RELEASE(this->_buffer, this->_context->getWorkspace());
            if (_isShapeAlloc)
                RELEASE(this->_shapeInfo, this->_context->getWorkspace());

            if (_isBuffDAlloc)
                RELEASE_SPECIAL(_bufferD, this->_context->getWorkspace());
            if (_isShapeAlloc)
                RELEASE_SPECIAL(_shapeInfoD, this->_context->getWorkspace());
            _buffer = nullptr;
            //setBuffer(newBuffer);
            _shapeInfo = newShape;
            setSpecialBuffers(newBufferD, newShapeD);
            this->_isBuffAlloc = false;
            this->_isShapeAlloc = true;
            this->_isBuffDAlloc = true;
            this->_isShapeDAlloc = true;
        }

        //tickReadHost();
        tickWriteDevice();
    }

    void NDArray::applyPairwiseTransform(nd4j::pairwise::Ops op, const NDArray* other, NDArray *target, ExtraArguments *extraParams) const{
        if (isS())
            throw std::runtime_error("NDArray::applyPairwiseTransform: you can't use this method on String array!");
        if (other->lengthOf() != target->lengthOf())
            throw std::invalid_argument("NDArray::applyPairwiseTransform method - lengths of arrays are mismatched");
        if (target->_dataType != this->_dataType && target->_dataType != other->_dataType)
            throw std::invalid_argument("NDArray::applyPairwiseTransform method - type of target array must be the same as type of this or other array !");
        if (_context == nullptr)
            throw std::runtime_error("Launch context cannot be NULL!!!");
        if (_context->getCudaStream() == nullptr)
            throw std::runtime_error("CUDA stream cannot be NULL!!!");


        if (!this->isActualOnDeviceSide())
            this->syncToDevice();

        if (!other->isActualOnDeviceSide())
            other->syncToDevice();

        NativeOpExecutioner::execPairwiseTransform(_context, op, this->_buffer, this->_shapeInfo, this->_bufferD, this->_shapeInfoD, other->_buffer, other->_shapeInfo, other->_bufferD, other->_shapeInfoD, target->_buffer, target->_shapeInfo, target->_bufferD, target->_shapeInfoD, extraParams != nullptr ? extraParams->argumentsAsT(target->dataType()) : nullptr);

        if(target)
            target->tickWriteDevice();
        else
            this->tickWriteDevice();

        if (extraParams != nullptr)
            this->synchronize();
    }

////////////////////////////////////////////////////////////////////////
    void NDArray::syncToHost() const {
        
        if(isEmpty()) return;        
        
        if (_buffer == nullptr && !this->isEmpty()) {
//            const_cast<NDArray*>(this)->lazyAllocateBuffer();
            //NDArray* constThis =  const_cast<NDArray*>(this); // not recommended solution
            //ALLOCATE(constThis->_buffer, _context->getWorkspace(), (getOffset(_length - 1) + 1) * sizeOfT(), int8_t);
            //constThis->_isBuffAlloc = true;
            throw std::runtime_error("Cannot sync to host due host buffer is not allocated yet.");
        }
        else if (lengthOf() == 0) {
            printf("sync with zero lenght is not needed.");
            return;
        }

        auto res = hipStreamSynchronize(*_context->getCudaStream());
        if (res != 0)
            throw cuda_exception::build("syncToHost failed to to some previous kernel failre", res);

        if (ews() != 1) {
            // FIXME: ^$%@#$%@#$!@#!!!!!!!!!!!
            for (Nd4jLong i = 0; i < _length; i++) {
                auto offset = getOffset(i) * sizeOfT();
                hipMemcpy(_buffer + offset, _bufferD + offset, sizeOfT(), hipMemcpyDeviceToHost);
            }
        }
        else
            hipMemcpy(_buffer, _bufferD, _length * sizeOfT(), hipMemcpyDeviceToHost);
        
        tickReadHost();
    }

////////////////////////////////////////////////////////////////////////
    void NDArray::syncToDevice() const {
        
        if(isEmpty()) return;

        if (_bufferD == nullptr) {
            NDArray* constThis =  const_cast<NDArray*>(this); // not recommended solution
            void* p = constThis->_bufferD;
            ALLOCATE_SPECIAL(p, _context->getWorkspace(), (getOffset(_length - 1) + 1) * sizeOfT(), int8_t);
            constThis->_isBuffDAlloc = true;
        }

         if (ews() != 1) {
            for (Nd4jLong i = 0; i < _length; i++) {
                auto offset = getOffset(i) * sizeOfT();
                hipMemcpy(_bufferD + offset, _buffer + offset, sizeOfT(), hipMemcpyHostToDevice);
            }
        }
        else
            hipMemcpy(_bufferD, _buffer, _length * sizeOfT(), hipMemcpyHostToDevice);
                
        tickReadDevice();        
    }

    void NDArray::syncShape() const {
        hipMemcpy(_shapeInfoD, _shapeInfo, shape::shapeInfoByteLength(_shapeInfo), hipMemcpyHostToDevice);
    }

    template <typename X, typename Y>
    void NDArray::templatedDoubleAssign(void *xBuffer, const Nd4jLong xOffset, const void *yBuffer, const Nd4jLong yOffset) const {
        auto x = reinterpret_cast<X *>(xBuffer);
        const auto y = reinterpret_cast<const Y *>(yBuffer);
        if (x && y)
        *(reinterpret_cast<X*>(xBuffer) + xOffset) = static_cast<X>(*(reinterpret_cast<Y const*>(yBuffer) + yOffset));
    }
    BUILD_DOUBLE_TEMPLATE(template void NDArray::templatedDoubleAssign, (void *xBuffer, const Nd4jLong xOffset, const void *yBuffer, const Nd4jLong yOffset) const, LIBND4J_TYPES, LIBND4J_TYPES);

    // This method assigns values of given NDArray to this one
    void NDArray::assign(const NDArray& other) {

        if (this == &other)
            return;

        // if (!Environment::getInstance()->isExperimentalBuild() && (this->dataType() != other.dataType() && other.dataType() != DataType::BOOL)) {
        //     throw datatype_exception::build("NDArray::assign: cannot assign array of different types", this->dataType(), other.dataType());
        // }

        if (other.isScalar()) {
            if(this->isScalar()) {
                if (!this->isEmpty() && !other.isEmpty()) {
                    BUILD_DOUBLE_SELECTOR(_dataType, other._dataType, templatedDoubleAssign,
                                          (_buffer, 0, other._buffer, 0), LIBND4J_TYPES, LIBND4J_TYPES);
                    if (!isActualOnDeviceSide())
                    syncToDevice();
                    tickWriteHost();
                }
                else if (this->isEmpty() != other.isEmpty()) { // need assign non-empty scalar to empty
                    if (other.isEmpty()) {
                        ArrayOptions::setPropertyBit(this->_shapeInfo, ARRAY_EMPTY);
                        syncShape();
                    }
                    else
                        *this = other;
                }
            }
            else {
                if (!isActualOnDeviceSide())
                syncToDevice();
                NativeOpExecutioner::execScalar(_context, scalar::CopyPws, _buffer, _shapeInfo, _bufferD, _shapeInfoD, _buffer, _shapeInfo, _bufferD, _shapeInfoD, other._buffer, other._shapeInfo, other._bufferD, other._shapeInfoD, nullptr);
            }
            tickWriteDevice();
            return;
        }

        if (other._length != _length) {
            auto shapeThis = ShapeUtils::shapeAsString(this);
            auto shapeThat = ShapeUtils::shapeAsString(&other);
            nd4j_printf("Can't assign new value to the array: this shape %s; other shape: %s\n", shapeThis.c_str(), shapeThat.c_str());
            throw std::runtime_error("Lengths of arrays are mismatched");
        }

        // memcpy is allowed only for same order && same ews (being equal to 1)
        if (ordering() == other.ordering() && _dataType == other._dataType && ews() == 1 && other.ews() == 1)
            hipMemcpy(_bufferD, other._bufferD, _length * sizeOfT(), hipMemcpyDeviceToDevice);
        else 
            NativeOpExecutioner::execTransformAny(_context, transform::Assign, nullptr, other._shapeInfo, other._bufferD, other._shapeInfoD, nullptr, _shapeInfo, _bufferD, _shapeInfoD, nullptr, nullptr, nullptr);

        //syncToHost();

        tickWriteDevice();
    }

    ////////////////////////////////////////////////////////////////////////
// This method returns new copy of this NDArray, optionally in different order
    NDArray* NDArray::dup(const char newOrder) {

        char order = newOrder == 'a' ? ordering() : newOrder;

        auto outShapeInfo = ShapeBuilders::createShapeInfo(_dataType, order, getShapeAsVector(), _context->getWorkspace());        
        auto result = new NDArray(outShapeInfo, true, _context, true);
        result->assign(*this);

        return result;
    }

    void NDArray::synchronize() const {
        auto res = hipStreamSynchronize(*(_context->getCudaStream()));
        if (res != 0)
            throw std::runtime_error("Synchronization failed");
    }

//////////////////////////////////////////////////////////////////////////
    template <>
    utf8string NDArray::e(const Nd4jLong i) const {
        if (i >= _length)
            throw std::invalid_argument("NDArray::e(i): input index is out of array length !");

        if (!isS())
            throw std::runtime_error("This method is available for String arrays only");

        lazyAllocateBuffer();
        if(!isActualOnHostSide()) 
            syncToHost();

        tickReadHost();
        auto rp = getOffset(i);
        
        return *(reinterpret_cast<utf8string**>(_buffer)[rp]);
    }

//////////////////////////////////////////////////////////////////////////
    template <>
    std::string NDArray::e(const Nd4jLong i) const {

        const_cast<NDArray*>(this)->lazyAllocateBuffer();
        if(!isActualOnHostSide())
            syncToHost();

        auto u = e<utf8string>(i);
        std::string r(u._buffer);
        tickReadHost();
        
        return r;
    }

//////////////////////////////////////////////////////////////////////////
    template <typename T>
    T NDArray::e(const Nd4jLong i) const {

        if (i >= _length)
            throw std::invalid_argument("NDArray::e(i): input index is out of array length !");

        const_cast<NDArray*>(this)->lazyAllocateBuffer();
        if(!isActualOnHostSide())
            syncToHost();

        auto rp = getOffset(i);
        tickReadHost();

        BUILD_SINGLE_PARTIAL_SELECTOR(this->dataType(), return templatedGet<, T>(this->_buffer, rp), LIBND4J_TYPES);
        
//        return static_cast<T>(119);
    }
    BUILD_SINGLE_UNCHAINED_TEMPLATE(template , NDArray::e(const Nd4jLong) const, LIBND4J_TYPES);
    //BUILD_DOUBLE_TEMPLATE(template void NDArray::templatedSet, (void *buffer, const Nd4jLong *indices, Y value), LIBND4J_TYPES, LIBND4J_TYPES);


//////////////////////////////////////////////////////////////////////////
// Returns value from 2D matrix by coordinates/indexes
    template <typename T>
    T NDArray::e(const Nd4jLong i, const Nd4jLong j) const {
        if (rankOf() != 2 || i >= shapeOf()[0] || j >= shapeOf()[1])
            throw std::invalid_argument("NDArray::e(i,j): one of input indexes is out of array length or rank!=2 !");

        const_cast<NDArray*>(this)->lazyAllocateBuffer();
        if(!isActualOnHostSide()) 
            syncToHost();

        auto xType = this->dataType();
        Nd4jLong coords[2] = {i, j};
        auto xOffset = shape::getOffset(0, shapeOf(), stridesOf(), coords, rankOf());
        tickReadHost();
        //return (*this)(i, j);
        BUILD_SINGLE_PARTIAL_SELECTOR(xType, return templatedGet<, T>(this->_buffer, xOffset), LIBND4J_TYPES);
                
        return static_cast<T>(119);
    }
    BUILD_SINGLE_UNCHAINED_TEMPLATE(template , NDArray::e(const Nd4jLong, const Nd4jLong) const, LIBND4J_TYPES);

//////////////////////////////////////////////////////////////////////////
// returns value from 3D tensor by coordinates
    template <typename T>
    T NDArray::e(const Nd4jLong i, const Nd4jLong j, const Nd4jLong k) const {
        //return (*this)(i, j, k);
        if (rankOf() != 3 || i >= shapeOf()[0] || j >= shapeOf()[1] || k >= shapeOf()[2])
            throw std::invalid_argument("NDArray::e(i,j,k): one of input indexes is out of array length or rank!=3 !");

        const_cast<NDArray*>(this)->lazyAllocateBuffer();
        if(!isActualOnHostSide()) 
            syncToHost();

        auto xType = this->dataType();
        Nd4jLong coords[3] = {i, j, k};
        auto xOffset = shape::getOffset(0, shapeOf(), stridesOf(), coords, rankOf());
        tickReadHost();
        BUILD_SINGLE_PARTIAL_SELECTOR(xType, return templatedGet<, T>(this->_buffer, xOffset), LIBND4J_TYPES);
                
        return static_cast<T>(119);
    }
    BUILD_SINGLE_UNCHAINED_TEMPLATE(template , NDArray::e(const Nd4jLong, const Nd4jLong, const Nd4jLong) const, LIBND4J_TYPES);

//////////////////////////////////////////////////////////////////////////
    // returns value from 3D tensor by coordinates
    template <typename T>
    T NDArray::e(const Nd4jLong i, const Nd4jLong j, const Nd4jLong k, const Nd4jLong l) const {
        //return (*this)(i, j, k);
        if (rankOf() != 4 || i >= shapeOf()[0] || j >= shapeOf()[1] || k >= shapeOf()[2] || l >= shapeOf()[3])
            throw std::invalid_argument("NDArray::e(i,j,k,l): one of input indexes is out of array length or rank!=4 !");

        const_cast<NDArray*>(this)->lazyAllocateBuffer();
        if(!isActualOnHostSide()) 
            syncToHost();

        auto xType = this->dataType();
        Nd4jLong coords[4] = {i, j, k, l};
        auto xOffset = shape::getOffset(0, shapeOf(), stridesOf(), coords, rankOf());
        tickReadHost();
        BUILD_SINGLE_PARTIAL_SELECTOR(xType, return templatedGet<, T>(this->_buffer, xOffset), LIBND4J_TYPES);
        
        return static_cast<T>(119);
    }
    BUILD_SINGLE_UNCHAINED_TEMPLATE(template , NDArray::e(const Nd4jLong, const Nd4jLong, const Nd4jLong, const Nd4jLong) const, LIBND4J_TYPES);

//////////////////////////////////////////////////////////////////////////
NDArray NDArray::e(const Nd4jLong i) const {
    if (i >= _length)
        throw std::invalid_argument("scalar NDArray::e(i): input index is out of array length !");

    NDArray scalar(_dataType, _context);
    
    if(isActualOnHostSide()) {
        hipMemcpy(scalar._bufferD, bufferWithOffset(getOffset(i)), sizeOfT(), hipMemcpyHostToDevice);
        tickReadHost();
    }
    else {
        hipMemcpy(scalar._bufferD, specialBufferWithOffset(getOffset(i)), sizeOfT(), hipMemcpyDeviceToDevice);
        tickReadDevice();
    }

    scalar.tickWriteDevice();
    return scalar;
}    

////////////////////////////////////////////////////////////////////////
#ifndef __JAVACPP_HACK__

    template<typename T>
    void NDArray::applyTriplewiseLambda(NDArray* second, NDArray *third, const std::function<T(T, T, T)>& func, NDArray* target) {
        if (target == nullptr)
            target = this;

        if (second == nullptr) {
            nd4j_printf("applyTriplewiseLambda requires three operands to be valid NDArrays, but Second is NULL\n","");
            throw std::runtime_error("second is null");
        }

        if (third == nullptr) {
            nd4j_printf("applyTriplewiseLambda requires three operands to be valid NDArrays, but Third is NULL\n","");
            throw std::runtime_error("third is null");
        }
        if(_dataType != DataTypeUtils::fromT<T>())
            throw std::runtime_error("NDArray::applyTriplewiseLambda<T> method: wrong template parameter T, its type should be the same as type of this array!");
        if(_dataType != second->_dataType || _dataType != third->_dataType || _dataType != target->_dataType)
            throw std::runtime_error("NDArray::applyTriplewiseLambda<T> method: bother four arrays (this, second, third, target) should have the same type !");

        if (this->lengthOf() != second->lengthOf() || this->lengthOf() != third->lengthOf() || !this->isSameShape(second) || !this->isSameShape(third)) {
            nd4j_printf("applyPairwiseLambda requires both operands to have the same shape\n","");
            throw std::runtime_error("Shapes mismach");
        }

        auto f = this->bufferAsT<T>();
        auto s = second->bufferAsT<T>();
        auto t = third->bufferAsT<T>();
        auto z = target->bufferAsT<T>();

        if (this->ordering() == second->ordering() && this->ordering() == third->ordering()  && this->ordering() == target->ordering() && (this->ews() == 1 && target->ews() == 1) && this->ews() == second->ews() && this->ews() == third->ews()) {
#pragma omp parallel for simd schedule(static)
            for (Nd4jLong e = 0; e < this->lengthOf(); e++)
                z[e] = func(f[e], s[e], t[e]);
        } else {
            if (f == z) {

#pragma omp parallel for schedule(guided)
                for (int e = 0; e < this->lengthOf(); e++) {

                    auto tOffset = this->getOffset(e);
                    auto uOffset = second->getOffset(e);
                    auto vOffset = third->getOffset(e);

                    f[tOffset] = func(f[tOffset], s[uOffset], t[vOffset]);
                }
            } else {

#pragma omp parallel for schedule(guided)
                for (int e = 0; e < this->lengthOf(); e++) {

                    auto tOffset = this->getOffset(e);
                    auto uOffset = second->getOffset(e);
                    auto vOffset = third->getOffset(e);
                    auto zOffset = target->getOffset(e);

                    z[zOffset] = func(f[tOffset], s[uOffset], t[vOffset]);
                }
            }
        }
        target->tickWriteDevice();
    }
    template void NDArray::applyTriplewiseLambda(NDArray* second, NDArray *third, const std::function<double (double, double, double)>& func, NDArray* target);
    template void NDArray::applyTriplewiseLambda(NDArray* second, NDArray *third, const std::function<float (float, float, float)>& func, NDArray* target);
    template void NDArray::applyTriplewiseLambda(NDArray* second, NDArray *third, const std::function<float16 (float16, float16, float16)>& func, NDArray* target);
    template void NDArray::applyTriplewiseLambda(NDArray* second, NDArray *third, const std::function<bfloat16 (bfloat16, bfloat16, bfloat16)>& func, NDArray* target);
    template void NDArray::applyTriplewiseLambda(NDArray* second, NDArray *third, const std::function<Nd4jLong (Nd4jLong, Nd4jLong, Nd4jLong)>& func, NDArray* target);
    template void NDArray::applyTriplewiseLambda(NDArray* second, NDArray *third, const std::function<int (int, int, int)>& func, NDArray* target);
    template void NDArray::applyTriplewiseLambda(NDArray* second, NDArray *third, const std::function<int16_t (int16_t, int16_t, int16_t)>& func, NDArray* target);
    template void NDArray::applyTriplewiseLambda(NDArray* second, NDArray *third, const std::function<uint8_t (uint8_t, uint8_t, uint8_t)>& func, NDArray* target);
    template void NDArray::applyTriplewiseLambda(NDArray* second, NDArray *third, const std::function<int8_t (int8_t, int8_t, int8_t)>& func, NDArray* target);
    template void NDArray::applyTriplewiseLambda(NDArray* second, NDArray *third, const std::function<bool (bool, bool, bool)>& func, NDArray* target);


    template<typename T>
    void NDArray::applyPairwiseLambda(NDArray* other, const std::function<T(T, T)>& func, NDArray* target) {
        if (target == nullptr)
            target = this;

        if (other == nullptr) {
            nd4j_printf("applyPairwiseLambda requires both operands to be valid NDArrays, but Y is NULL\n","");
            throw std::runtime_error("Other is null");
        }

        if(_dataType != DataTypeUtils::fromT<T>())
            throw std::runtime_error("NDArray::applyPairwiseLambda<T> method: wrong template parameter T, its type should be the same as type of this array!");
        if(_dataType != other->_dataType || _dataType != target->_dataType)
            throw std::runtime_error("NDArray::applyPairwiseLambda<T> method: all three arrays (this, other, target) must have the same type !");

        if (this->lengthOf() != other->lengthOf()) {
            nd4j_printf("applyPairwiseLambda requires both operands to have the same shape\n","");
            throw std::runtime_error("Shapes mismach");
        }

        auto f = this->bufferAsT<T>();
        auto s = other->bufferAsT<T>();
        auto z = target->bufferAsT<T>();

        if (this->ordering() == other->ordering() && this->ordering() == target->ordering() && (this->ews() == 1 && target->ews() == 1) && this->ews() == other->ews()) {
#pragma omp parallel for simd schedule(guided)
            for (int e = 0; e < this->lengthOf(); e++)
                z[e] = func(f[e], s[e]);
        } else {
            if (f == z) {

#pragma omp parallel for schedule(guided)
                for (int e = 0; e < this->lengthOf(); e++) {

                    auto xOffset = this->getOffset(e);
                    auto yOffset = other->getOffset(e);

                    f[xOffset] = func(f[xOffset], s[yOffset]);
                }
            } else {

#pragma omp parallel for schedule(guided)
                for (int e = 0; e < this->lengthOf(); e++) {

                    auto xOffset = this->getOffset(e);
                    auto yOffset = other->getOffset(e);
                    auto zOffset = target->getOffset(e);

                    z[zOffset] = func(f[xOffset], s[yOffset]);
                }
            }
        }
        target->tickWriteDevice();
    }
    template void NDArray::applyPairwiseLambda(NDArray* other, const std::function<double (double, double)>& func, NDArray* target);
    template void NDArray::applyPairwiseLambda(NDArray* other, const std::function<float (float, float)>& func, NDArray* target);
    template void NDArray::applyPairwiseLambda(NDArray* other, const std::function<float16 (float16, float16)>& func, NDArray* target);
    template void NDArray::applyPairwiseLambda(NDArray* other, const std::function<bfloat16 (bfloat16, bfloat16)>& func, NDArray* target);
    template void NDArray::applyPairwiseLambda(NDArray* other, const std::function<Nd4jLong (Nd4jLong, Nd4jLong)>& func, NDArray* target);
    template void NDArray::applyPairwiseLambda(NDArray* other, const std::function<int (int, int)>& func, NDArray* target);
    template void NDArray::applyPairwiseLambda(NDArray* other, const std::function<int16_t (int16_t, int16_t)>& func, NDArray* target);
    template void NDArray::applyPairwiseLambda(NDArray* other, const std::function<uint8_t (uint8_t, uint8_t)>& func, NDArray* target);
    template void NDArray::applyPairwiseLambda(NDArray* other, const std::function<int8_t (int8_t, int8_t)>& func, NDArray* target);
    template void NDArray::applyPairwiseLambda(NDArray* other, const std::function<bool (bool, bool)>& func, NDArray* target);


////////////////////////////////////////////////////////////////////////
    template<typename T>
    void NDArray::applyLambda(const std::function<T(T)>& func, NDArray* target) {
        if (target == nullptr)
            target = this;

        if(_dataType != DataTypeUtils::fromT<T>())
            throw std::runtime_error("NDArray::applyLambda<T> method: wrong template parameter T, its type should be the same as type of this array!");
        if(_dataType != target->_dataType)
            throw std::runtime_error("NDArray::applyLambda<T> method: types of this and target array should match !");

        auto f = this->bufferAsT<T>();
        auto z = target->bufferAsT<T>();

        if (this->ordering() == target->ordering() && (this->ews() == 1 && target->ews() == 1)) {
#pragma omp parallel for simd schedule(guided)
            for (int e = 0; e < this->lengthOf(); e++)
                z[e] = func(f[e]);
        } else {
            if (f == z) {

#pragma omp parallel for schedule(guided)
                for (int e = 0; e < this->lengthOf(); e++) {

                    auto xOffset = this->getOffset(e);

                    f[xOffset] = func(f[xOffset]);
                }
            } else {

#pragma omp parallel for schedule(guided)
                for (int e = 0; e < this->lengthOf(); e++) {

                    auto xOffset = this->getOffset(e);
                    auto zOffset = target->getOffset(e);

                    z[zOffset] = func(f[xOffset]);
                }
            }
        }
        target->tickWriteDevice();
    }
    template void NDArray::applyLambda(const std::function<double(double)>& func, NDArray* target);
    template void NDArray::applyLambda(const std::function<float(float)>& func, NDArray* target);
    template void NDArray::applyLambda(const std::function<float16(float16)>& func, NDArray* target);
    template void NDArray::applyLambda(const std::function<bfloat16(bfloat16)>& func, NDArray* target);
    template void NDArray::applyLambda(const std::function<Nd4jLong(Nd4jLong)>& func, NDArray* target);
    template void NDArray::applyLambda(const std::function<int16_t(int16_t)>& func, NDArray* target);
    template void NDArray::applyLambda(const std::function<int32_t(int32_t)>& func, NDArray* target);
    template void NDArray::applyLambda(const std::function<uint8_t(uint8_t)>& func, NDArray* target);
    template void NDArray::applyLambda(const std::function<int8_t(int8_t)>& func, NDArray* target);
    template void NDArray::applyLambda(const std::function<bool(bool)>& func, NDArray* target);

    template<typename T>
    void NDArray::applyIndexedLambda(const std::function<T(Nd4jLong, T)>& func, NDArray* target) {
        if (target == nullptr)
            target = this;

        if(_dataType != DataTypeUtils::fromT<T>())
            throw std::runtime_error("NDArray::applyIndexedLambda<T> method: wrong template parameter T, its type should be the same as type of this array!");
        if(_dataType != target->_dataType)
            throw std::runtime_error("NDArray::applyIndexedLambda<T> method: types of this and target array should match !");

        auto f = this->bufferAsT<T>();
        auto z = target->bufferAsT<T>();

        if (this->ordering() == target->ordering() && (this->ews() == 1 && target->ews() == 1)) {
#pragma omp parallel for simd schedule(guided)
            for (Nd4jLong e = 0; e < this->lengthOf(); e++)
                z[e] = func(e, f[e]);
        } else {
            if (f == z) {

#pragma omp parallel for schedule(guided)
                for (Nd4jLong e = 0; e < this->lengthOf(); e++) {

                    auto xOffset = this->getOffset(e);

                    f[xOffset] = func(e, f[xOffset]);
                }
            } else {

#pragma omp parallel for schedule(guided)
                for (Nd4jLong e = 0; e < this->lengthOf(); e++) {

                    auto xOffset = this->getOffset(e);
                    auto zOffset = target->getOffset(e);

                    z[zOffset] = func(e, f[xOffset]);
                }
            }
        }
        target->tickWriteDevice();
    }
    template void NDArray::applyIndexedLambda(const std::function<double(Nd4jLong, double)>& func, NDArray* target);
    template void NDArray::applyIndexedLambda(const std::function<float(Nd4jLong, float)>& func, NDArray* target);
    template void NDArray::applyIndexedLambda(const std::function<float16(Nd4jLong, float16)>& func, NDArray* target);
    template void NDArray::applyIndexedLambda(const std::function<bfloat16(Nd4jLong, bfloat16)>& func, NDArray* target);
    template void NDArray::applyIndexedLambda(const std::function<Nd4jLong(Nd4jLong, Nd4jLong)>& func, NDArray* target);
    template void NDArray::applyIndexedLambda(const std::function<int(Nd4jLong, int)>& func, NDArray* target);
    template void NDArray::applyIndexedLambda(const std::function<int16_t(Nd4jLong, int16_t)>& func, NDArray* target);
    template void NDArray::applyIndexedLambda(const std::function<uint8_t (Nd4jLong, uint8_t)>& func, NDArray* target);
    template void NDArray::applyIndexedLambda(const std::function<int8_t(Nd4jLong, int8_t)>& func, NDArray* target);
    template void NDArray::applyIndexedLambda(const std::function<bool(Nd4jLong, bool)>& func, NDArray* target);


    template<typename T>
    void NDArray::applyIndexedPairwiseLambda(NDArray* other, const std::function<T(Nd4jLong, T, T)>& func, NDArray* target) {
        if (target == nullptr)
            target = this;

        if (other == nullptr) {
            nd4j_printf("applyIndexedPairwiseLambda requires both operands to be valid NDArrays, but Y is NULL\n","");
            throw std::runtime_error("Other is null");
        }
        if(_dataType != DataTypeUtils::fromT<T>())
            throw std::runtime_error("NDArray::applyIndexedPairwiseLambda<T> method: wrong template parameter T, its type should be the same as type of this array!");
        if(_dataType != target->_dataType)
            throw std::runtime_error("NDArray::applyIndexedPairwiseLambda<T> method: types of this and target array should match !");
        if (this->lengthOf() != other->lengthOf()) {
            nd4j_printf("applyIndexedPairwiseLambda requires both operands to have the same shape\n","");
            throw std::runtime_error("Shapes mismach");
        }

        auto f = this->bufferAsT<T>();
        auto s = other->bufferAsT<T>();
        auto z = target->bufferAsT<T>();

        if (this->ordering() == other->ordering() && this->ordering() == target->ordering() && (this->ews() == 1 && target->ews() == 1) && this->ews() == other->ews()) {
#pragma omp parallel for simd schedule(guided)
            for (Nd4jLong e = 0; e < this->lengthOf(); e++)
                z[e] = func((Nd4jLong) e, f[e], s[e]);
        } else {
            if (f == z) {

#pragma omp parallel for schedule(guided)
                for (int e = 0; e < this->lengthOf(); e++) {

                    auto xOffset = this->getOffset(e);
                    auto yOffset = other->getOffset(e);

                    f[xOffset] = func((Nd4jLong) e, f[xOffset], s[yOffset]);
                }
            } else {

#pragma omp parallel for schedule(guided)
                for (int e = 0; e < this->lengthOf(); e++) {

                    auto xOffset = this->getOffset(e);
                    auto yOffset = other->getOffset(e);
                    auto zOffset = target->getOffset(e);

                    z[zOffset] = func((Nd4jLong) e, f[xOffset], s[yOffset]);
                }
            }
        }
        target->tickWriteDevice();
    }
    template void NDArray::applyIndexedPairwiseLambda(NDArray* other, const std::function<double (Nd4jLong, double, double)>& func, NDArray* target);
    template void NDArray::applyIndexedPairwiseLambda(NDArray* other, const std::function<float (Nd4jLong, float, float)>& func, NDArray* target);
    template void NDArray::applyIndexedPairwiseLambda(NDArray* other, const std::function<float16 (Nd4jLong, float16, float16)>& func, NDArray* target);
    template void NDArray::applyIndexedPairwiseLambda(NDArray* other, const std::function<bfloat16 (Nd4jLong, bfloat16, bfloat16)>& func, NDArray* target);
    template void NDArray::applyIndexedPairwiseLambda(NDArray* other, const std::function<Nd4jLong (Nd4jLong, Nd4jLong, Nd4jLong)>& func, NDArray* target);
    template void NDArray::applyIndexedPairwiseLambda(NDArray* other, const std::function<int (Nd4jLong, int, int)>& func, NDArray* target);
    template void NDArray::applyIndexedPairwiseLambda(NDArray* other, const std::function<int16_t (Nd4jLong, int16_t, int16_t)>& func, NDArray* target);
    template void NDArray::applyIndexedPairwiseLambda(NDArray* other, const std::function<uint8_t (Nd4jLong, uint8_t, uint8_t)>& func, NDArray* target);
    template void NDArray::applyIndexedPairwiseLambda(NDArray* other, const std::function<int8_t (Nd4jLong, int8_t, int8_t)>& func, NDArray* target);
    template void NDArray::applyIndexedPairwiseLambda(NDArray* other, const std::function<bool (Nd4jLong, bool, bool)>& func, NDArray* target);
#endif

//////////////////////////////////////////////////////////////////////////
// perform array transformation
    void NDArray::applyTransform(nd4j::transform::FloatOps op, NDArray *target, ExtraArguments *extraParams) {

        if (isS())
            throw std::runtime_error("NDArray::applyTransform FloatOps: you can't use this method on String array!");

        if (target == nullptr)
            target = this;

        if (!target->isR())
            throw std::runtime_error("NDArray::applyTransform FloatOps: target array must have one of FLOAT types");

        NativeOpExecutioner::execTransformFloat(_context, op, _buffer, _shapeInfo, _bufferD, _shapeInfoD, target->_buffer, target->_shapeInfo, target->_bufferD, target->_shapeInfoD, extraParams != nullptr ? extraParams->argumentsAsT(target->dataType()) : nullptr, nullptr, nullptr);
        target->tickWriteDevice();
    }

    void NDArray::applyTransform(nd4j::transform::AnyOps op, NDArray *target, ExtraArguments *extraParams) {

        if (isS())
            throw std::runtime_error("NDArray::applyTransform AnyOps: you can't use this method on String array!");

        if (target == nullptr)
            target = this;

//        NDArray::registerSpecialUse({target}, {this});
        NativeOpExecutioner::execTransformAny(_context, op, _buffer, _shapeInfo, _bufferD, _shapeInfoD, target->_buffer, target->_shapeInfo, target->_bufferD, target->_shapeInfoD, extraParams != nullptr ? extraParams->argumentsAsT(target->dataType()) : nullptr, nullptr, nullptr);
        target->tickWriteDevice();
    }

    void NDArray::applyTransform(nd4j::transform::SameOps op, NDArray *target, ExtraArguments *extraParams) {
        nd4j_printf("Same op %i transform:\n", (int)op);
        if (isS())
            throw std::runtime_error("NDArray::applyTransform SameOps: you can't use this method on String array!");

        if (target == nullptr)
            target = this;

        if (target->dataType() != this->dataType())
            throw std::runtime_error("NDArray::applyTransform SameOps: target array must have the same data type as original array");
//        NDArray::registerSpecialUse({target}, {this});
        NativeOpExecutioner::execTransformSame(_context, op, this->_buffer, this->_shapeInfo, this->_bufferD, this->_shapeInfoD, target->_buffer, target->_shapeInfo, target->_bufferD, target->_shapeInfoD, extraParams != nullptr ? extraParams->argumentsAsT(target->dataType()) : nullptr, nullptr, nullptr);
        target->tickWriteDevice();
    }

    void NDArray::applyTransform(nd4j::transform::BoolOps op, NDArray *target, ExtraArguments *extraParams) {
        if (isS())
            throw std::runtime_error("NDArray::applyTransform BoolOps: you can't use this method on String array!");

        if (target == nullptr)
            target = this;

        if (!target->isB())
            throw std::runtime_error("NDArray::applyTransform BoolOps: target array must have one of BOOL types");

        NDArray::registerSpecialUse({target}, {this});
        NativeOpExecutioner::execTransformBool(_context, op, this->_buffer, this->_shapeInfo, _bufferD, _shapeInfoD, target->_buffer, target->_shapeInfo, target->_bufferD, target->_shapeInfoD, extraParams != nullptr ? extraParams->argumentsAsT(this->dataType()) : nullptr, nullptr, nullptr);
    }

    void NDArray::applyTransform(nd4j::transform::StrictOps op, NDArray *target, ExtraArguments *extraParams) {
        if (isS())
            throw std::runtime_error("NDArray::applyTransform StrictOps: you can't use this method on String array!");

        if (target == nullptr)
            target = this;

        if (!this->isR() || !target->isR() || (this->dataType() != target->dataType()))
            throw std::runtime_error("NDArray::applyTransform StrictOps: both Source and Target array must have same FLOAT type !");

        NDArray::registerSpecialUse({target}, {this});
        NativeOpExecutioner::execTransformStrict(_context, op, this->_buffer, this->_shapeInfo, _bufferD, _shapeInfoD, target->_buffer, target->_shapeInfo, target->_bufferD, target->_shapeInfoD, extraParams != nullptr ? extraParams->argumentsAsT(target->dataType()) : nullptr, nullptr, nullptr);
    }

//////////////////////////////////////////////////////////////////////////
// perform array transformation
    // void NDArray::applyTransform(nd4j::transform::FloatOps op, void *extraParams) {
    //     applyTransform(op, this, extraParams);
    // }

    // void NDArray::applyTransform(nd4j::transform::AnyOps op, void *extraParams) {
    //     applyTransform(op, this, extraParams);
    // }

    // void NDArray::applyTransform(nd4j::transform::SameOps op, void *extraParams) {
    //     applyTransform(op, this, extraParams);
    // }

    // void NDArray::applyTransform(nd4j::transform::BoolOps op, void *extraParams) {
    //     applyTransform(op, this, extraParams);
    // }

    // void NDArray::applyTransform(nd4j::transform::StrictOps op, void *extraParams) {
    //     applyTransform(op, this, extraParams);
    // }

    // perform array transformation
    NDArray NDArray::transform(nd4j::transform::FloatOps op, void *extraParams) const {
        if (isS())
            throw std::runtime_error("NDArray::transform FloatOps: you can't use this method on String array!");

        NDArray result(this->ordering(), getShapeAsVector(), DataTypeUtils::pickFloatingType(dataType()), this->_context);
        NativeOpExecutioner::execTransformFloat(_context, op, this->_buffer, this->_shapeInfo, this->_bufferD, this->_shapeInfoD, result._buffer, result._shapeInfo, result._bufferD, result._shapeInfoD, extraParams, nullptr, nullptr);
        return result;
    }

    NDArray NDArray::transform(nd4j::transform::SameOps op, void *extraParams) const {
        if (isS())
            throw std::runtime_error("NDArray::transform SameOps: you can't use this method on String array!");

        NDArray result(this->_shapeInfo, false, this->_context);
        NativeOpExecutioner::execTransformSame(_context, op, this->_buffer, this->_shapeInfo, this->_bufferD, this->_shapeInfoD, result._buffer, result._shapeInfo, result._bufferD, result._shapeInfoD, extraParams, nullptr, nullptr);
        return result;
    }

    NDArray NDArray::transform(nd4j::transform::StrictOps op, void *extraParams) const {
        if (!this->isR())
            throw std::runtime_error("Source array must have one of FLOAT types");

        NDArray result(this->_shapeInfo, false, this->_context);
        NativeOpExecutioner::execTransformStrict(_context, op, this->_buffer, this->_shapeInfo, this->_bufferD, this->_shapeInfoD, result._buffer, result._shapeInfo, result._bufferD, result._shapeInfoD, extraParams, nullptr, nullptr);
        return result;
    }

    NDArray NDArray::transform(nd4j::transform::BoolOps op, void *extraParams) const {
        if (isS())
            throw std::runtime_error("NDArray::transform BoolOps: you can't use this method on String array!");

        NDArray result(this->ordering(), getShapeAsVector(), nd4j::DataType::BOOL, this->_context);
        NativeOpExecutioner::execTransformBool(_context, op, this->_buffer, this->_shapeInfo, this->_bufferD, this->_shapeInfoD, result._buffer, result._shapeInfo, result._bufferD, result._shapeInfoD, extraParams, nullptr, nullptr);
        return result;
    }

//////////////////////////////////////////////////////////////////////////
// perform pairwise transformation
    void NDArray::applyPairwiseTransform(nd4j::pairwise::Ops op, const NDArray& other, ExtraArguments *extraParams) {
        applyPairwiseTransform(op, &other, this, extraParams);
    }

    void NDArray::applyPairwiseTransform(nd4j::pairwise::BoolOps op, const NDArray *other, NDArray *target, ExtraArguments *extraParams) const{
        if (isS())
            throw std::runtime_error("NDArray::applyPairwiseTransform BoolOps: you can't use this method on String array!");
        if (other->lengthOf() != target->lengthOf())
            throw std::invalid_argument("NDArray::applyPairwiseTransform BoolOps method - lengths of arrays are mismatched");
        if (!target->isB())
            throw std::invalid_argument("NDArray::applyPairwiseTransform BoolOps method - result must have bool type");
        if (_dataType != other->_dataType)
            throw std::invalid_argument("NDArray::applyPairwiseTransform BoolOps method - this and other arrays must have the same type !");

        NDArray::registerSpecialUse({target}, {this,other});
        NativeOpExecutioner::execPairwiseBoolTransform(_context, op, this->_buffer, this->_shapeInfo, this->_bufferD, this->_shapeInfoD, other->_buffer, other->_shapeInfo, other->_bufferD, other->_shapeInfoD, target->_buffer, target->_shapeInfo, target->_bufferD, target->_shapeInfoD, extraParams != nullptr ? extraParams->argumentsAsT(target->dataType()) : nullptr);
    }

//////////////////////////////////////////////////////////////////////////
    void NDArray::applyScalarArr(nd4j::scalar::BoolOps op, const NDArray* scalar, NDArray *target, ExtraArguments *extraParams) const {
        if (isS())
            throw std::runtime_error("NDArray::applyScalarArr BoolOps: you can't use this method on String array!");
        if (target == nullptr || !target->isB())
            throw std::invalid_argument("NDArray::applyScalarArr bool method: target is nullptr or has not bool type!");
        if (_dataType != scalar->_dataType) {
            nd4j_printf("This dtype: [%i]; scalar dtype: [%i]\n", this->_dataType, scalar->_dataType);
            throw std::invalid_argument("NDArray::applyScalarArr bool method: this and scalar arrays must have the same type!");
        }
        if (!this->isActualOnDeviceSide())
            this->syncToDevice();

        if (!scalar->isActualOnDeviceSide())
            scalar->syncToDevice();
        NDArray::registerSpecialUse({target}, {this, scalar});
        NativeOpExecutioner::execScalarBool(_context, op, _buffer, _shapeInfo, _bufferD, _shapeInfoD, target->_buffer, target->_shapeInfo, target->_bufferD, target->_shapeInfoD, scalar->_buffer, scalar->_shapeInfo, scalar->_bufferD, scalar->_shapeInfoD, extraParams != nullptr ? extraParams->argumentsAsT(target->dataType()): nullptr);
    }

    template <typename T>
    void NDArray::applyScalar(nd4j::scalar::BoolOps op, const T scalar, NDArray *target, ExtraArguments *extraParams) const {

        auto scalarArr = NDArrayFactory::create<T>(scalar, _context);
        applyScalarArr(op, &scalarArr, target, extraParams);
    }

    template <> void NDArray::applyScalar(nd4j::scalar::BoolOps op, const NDArray* scalar, NDArray *target, ExtraArguments *extraParams) const { throw std::runtime_error("NDArray::applyScalar<NDArray*> method: do not use me!");}
    template void NDArray::applyScalar<double>(nd4j::scalar::BoolOps op, const double scalar, NDArray *target, ExtraArguments *extraParams) const;
    template void NDArray::applyScalar<float>(nd4j::scalar::BoolOps op, const float scalar, NDArray *target, ExtraArguments *extraParams) const;
    template void NDArray::applyScalar<float16>(nd4j::scalar::BoolOps op, const float16 scalar, NDArray *target, ExtraArguments *extraParams) const;
    template void NDArray::applyScalar<bfloat16>(nd4j::scalar::BoolOps op, const bfloat16 scalar, NDArray *target, ExtraArguments *extraParams) const;
    template void NDArray::applyScalar<Nd4jLong>(nd4j::scalar::BoolOps op, const Nd4jLong scalar, NDArray *target, ExtraArguments *extraParams) const;
    template void NDArray::applyScalar<int>(nd4j::scalar::BoolOps op, const int scalar, NDArray *target, ExtraArguments *extraParams) const;
    template void NDArray::applyScalar<int16_t>(nd4j::scalar::BoolOps op, const int16_t scalar, NDArray *target, ExtraArguments *extraParams) const;
    template void NDArray::applyScalar<int8_t>(nd4j::scalar::BoolOps op, const int8_t scalar, NDArray *target, ExtraArguments *extraParams) const;
    template void NDArray::applyScalar<uint8_t>(nd4j::scalar::BoolOps op, const uint8_t scalar, NDArray *target, ExtraArguments *extraParams) const;
    template void NDArray::applyScalar<bool>(nd4j::scalar::BoolOps op, const bool scalar, NDArray *target, ExtraArguments *extraParams) const;

//////////////////////////////////////////////////////////////////////////
    void NDArray::applyScalarArr(nd4j::scalar::Ops op, const NDArray* scalar, NDArray* target, ExtraArguments *extraParams) {
        if (isS())
            throw std::runtime_error("NDArray::applyScalarArr: you can't use this method on String array!");
        if (!scalar->isScalar())
            throw std::invalid_argument("NDArray::applyScalarArr method: operand is not a scalar!");
        if(target == nullptr)
            target = this;
        if(target->_dataType != DataTypeUtils::pickPairwiseResultType(_shapeInfo, scalar->_shapeInfo) && !(target->_dataType == this->_dataType || target->_dataType == scalar->_dataType))
            throw std::invalid_argument("NDArray::applyScalarArr method: wrong type of target array!");

        if (!this->isActualOnDeviceSide())
            this->syncToDevice();

        if (!scalar->isActualOnDeviceSide())
            scalar->syncToDevice();
        NDArray::registerSpecialUse({target}, {this,scalar});
        NativeOpExecutioner::execScalar(_context, op, _buffer, _shapeInfo, _bufferD, _shapeInfoD, target->_buffer, target->_shapeInfo, target->_bufferD, target->_shapeInfoD, scalar->getBuffer(), scalar->getShapeInfo(), scalar->_bufferD, scalar->_shapeInfoD, extraParams != nullptr ? extraParams->argumentsAsT(target->dataType()) : nullptr);
    }

    template <typename T>
    void NDArray::applyScalar(nd4j::scalar::Ops op, const T scalar, NDArray *target, ExtraArguments *extraParams) {

        auto scalarArr = NDArrayFactory::create<T>(this->dataType(), scalar, this->_context);
        applyScalarArr(op, &scalarArr, target, extraParams);
    }

    template <> void NDArray::applyScalar(nd4j::scalar::Ops op, const NDArray* scalar, NDArray *target, ExtraArguments *extraParams) { throw std::runtime_error("NDArray::applyScalar<NDArray*> method: do not use me!");}
    template void NDArray::applyScalar(nd4j::scalar::Ops op, const double scalar, NDArray *target, ExtraArguments *extraParams);
    template void NDArray::applyScalar(nd4j::scalar::Ops op, const float scalar, NDArray *target, ExtraArguments *extraParams);
    template void NDArray::applyScalar(nd4j::scalar::Ops op, const float16 scalar, NDArray *target, ExtraArguments *extraParams);
    template void NDArray::applyScalar(nd4j::scalar::Ops op, const bfloat16 scalar, NDArray *target, ExtraArguments *extraParams);
    template void NDArray::applyScalar(nd4j::scalar::Ops op, const Nd4jLong scalar, NDArray *target, ExtraArguments *extraParams);
    template void NDArray::applyScalar(nd4j::scalar::Ops op, const int scalar, NDArray *target, ExtraArguments *extraParams);
    template void NDArray::applyScalar(nd4j::scalar::Ops op, const int16_t scalar, NDArray *target, ExtraArguments *extraParams);
    template void NDArray::applyScalar(nd4j::scalar::Ops op, const int8_t scalar, NDArray *target, ExtraArguments *extraParams);
    template void NDArray::applyScalar(nd4j::scalar::Ops op, const uint8_t scalar, NDArray *target, ExtraArguments *extraParams);
    template void NDArray::applyScalar(nd4j::scalar::Ops op, const bool scalar, NDArray *target, ExtraArguments *extraParams);

    //////////////////////////////////////////////////////////////////////////
    void NDArray::applyBroadcast(nd4j::broadcast::Ops op, const std::vector<int>& dimensions, const NDArray* tadArray, NDArray* target, ExtraArguments* extraArgs) {
        if (isS())
            throw std::runtime_error("NDArray::applyBroadcast: you can't use this method on String array!");
        if(((op == broadcast::Divide || op == broadcast::FloorDiv || op == broadcast::FloorMod) && tadArray->isB()) || (op == broadcast::ReverseDivide && this->isB()))
            throw std::runtime_error("NDArray::applyBroadcast: you can't divide by array!");

        if (dimensions.size() == 0)
            return;
        auto result = (NDArray*)this;// == nullptr ? this : target;
        if (target != nullptr)
            result = target;

        if(result->_dataType != DataTypeUtils::pickPairwiseResultType(_shapeInfo, tadArray->_shapeInfo))
            throw std::invalid_argument("NDArray::applyBroadcast method: wrong type of target array !");
        if(!result->isSameShape(this))
            throw std::invalid_argument("NDArray::applyBroadcast method: this and target arrays must have the same shape !");

        std::vector<int> copy(dimensions);

        //if (dimensions.size() > 1)
        //    std::sort(copy.begin(), copy.end());

        Nd4jLong tadLength = shape::tadLength(this->_shapeInfo, copy.data(), (int) copy.size());
        if (tadLength != tadArray->lengthOf())
            throw std::runtime_error("NDArray::applyBroadcast method: tad length mismatch !");

        shape::TAD tad(this->_shapeInfo, copy.data(), copy.size());
        tad.createTadOnlyShapeInfo();
        tad.createOffsets();
        if (!this->isActualOnDeviceSide())
            this->syncToDevice();

        if (!tadArray->isActualOnDeviceSide())
            tadArray->syncToDevice();

        // prepare input arrays for prepareDataForCuda function
        std::vector<std::pair<void*,size_t>> hostData;
        hostData.emplace_back(copy.data(), copy.size() * sizeof(int));							// 0 -- dimensions
        hostData.emplace_back(tad.tadOnlyShapeInfo, shape::shapeInfoByteLength(tad.tadOnlyShapeInfo));	// 1 -- xTadShapeInfo
        hostData.emplace_back(tad.tadOffsets, tad.numTads * sizeof(Nd4jLong));							// 2 -- xTadOffsets
        std::vector<void*> devicePtrs(hostData.size(), nullptr);

        // create cuda stream and LaunchContext
        hipError_t cudaResult;
        //hipStream_t stream;
        //cudaResult = hipStreamCreate(&stream);	ASSERT_EQ(0, cudaResult);
        //hipStream_t* stream = this->getContext()->getCudaStream();
        // allocate required amount of global device memory and copy host data to it
//    cudaResult = allocateDeviceMem(*pLc, devicePtrs, hostData);	ASSERT_EQ(0, cudaResult);
        for(int i = 0; i < devicePtrs.size(); ++i) {

            cudaResult = hipMalloc(reinterpret_cast<void **>(&devicePtrs[i]), hostData[i].second);
            if(cudaResult != 0) throw cuda_exception::build("Cannot allocate memory for tads on device", cudaResult);
            cudaResult = hipMemcpy(devicePtrs[i], hostData[i].first, hostData[i].second, hipMemcpyHostToDevice);
            if(cudaResult != 0) throw cuda_exception::build("Cannot copy memory block for tads on device", cudaResult);
        }

        //NDArray::registerSpecialUse({result}, {this, tadArray});

        // call cuda kernel which calculates result
        // TODO: eventually we want separate tads here
        NativeOpExecutioner::execBroadcast(_context, op, this->_buffer, this->_shapeInfo, this->_bufferD, this->_shapeInfoD, tadArray->_buffer, tadArray->_shapeInfo, tadArray->_bufferD, tadArray->_shapeInfoD, result->_buffer, result->_shapeInfo, result->_bufferD, result->_shapeInfoD, (int*)devicePtrs[0], (int)copy.size(), (Nd4jLong*)devicePtrs[1], (Nd4jLong*)devicePtrs[2], nullptr, nullptr);
        result->tickWriteDevice();
    }

    //////////////////////////////////////////////////////////////////////////
    void NDArray::applyBroadcast(nd4j::broadcast::BoolOps op, const std::vector<int>& dimensions, const NDArray* tadArray, NDArray* target, ExtraArguments* extraArgs) {
        if (isS())
            throw std::runtime_error("NDArray::applyBroadcast BoolOps: you can't use this method on String array!");

        if (dimensions.size() == 0)
            return;

        auto result = target == nullptr ? this : target;

        if(result->_dataType != DataType::BOOL)
            throw std::invalid_argument("NDArray::applyBroadcast bool method: type of target array must be BOOL!");
        if(!result->isSameShape(this))
            throw std::invalid_argument("NDArray::applyBroadcast bool method: this and other arrays must have the same shape !");
        if(_dataType != tadArray->_dataType)
            throw std::invalid_argument("NDArray::applyBroadcast bool method: this and tad arrays must have the same type !");

        std::vector<int> copy(dimensions);

        if (dimensions.size() > 1)
            std::sort(copy.begin(), copy.end());

        Nd4jLong tadLength = shape::tadLength(this->_shapeInfo, copy.data(), (int) copy.size());
        if (tadLength != tadArray->lengthOf())
            throw std::runtime_error("Tad length mismatch");

        shape::TAD tad(this->_shapeInfo, copy.data(), copy.size());
        tad.createTadOnlyShapeInfo();
        tad.createOffsets();
        if (!this->isActualOnDeviceSide())
            this->syncToDevice();

        if (!tadArray->isActualOnDeviceSide())
            tadArray->syncToDevice();
        // prepare input arrays for prepareDataForCuda function
        std::vector<std::pair<void*,size_t>> hostData;
        hostData.emplace_back(copy.data(), copy.size() * sizeof(int));							// 0 -- dimensions
        hostData.emplace_back(tad.tadOnlyShapeInfo, shape::shapeInfoByteLength(tad.tadOnlyShapeInfo));	// 1 -- xTadShapeInfo
        hostData.emplace_back(tad.tadOffsets, tad.numTads * sizeof(Nd4jLong));							// 2 -- xTadOffsets
        std::vector<void*> devicePtrs(hostData.size(), nullptr);

        // create cuda stream and LaunchContext
        hipError_t cudaResult;
        //hipStream_t stream;
        //cudaResult = hipStreamCreate(&stream);	ASSERT_EQ(0, cudaResult);
        hipStream_t* stream = this->getContext()->getCudaStream();
        // allocate required amount of global device memory and copy host data to it
//    cudaResult = allocateDeviceMem(*pLc, devicePtrs, hostData);	ASSERT_EQ(0, cudaResult);
        for(int i = 0; i < devicePtrs.size(); ++i) {

            cudaResult = hipMalloc(reinterpret_cast<void **>(&devicePtrs[i]), hostData[i].second);
            if(cudaResult != 0) throw cuda_exception::build("Cannot allocate memory for tads on device", cudaResult);
            hipMemcpyAsync(devicePtrs[i], hostData[i].first, hostData[i].second, hipMemcpyHostToDevice, *stream);
        }

        // call cuda kernel which calculates result
        //NDArray::registerSpecialUse({result}, {this, tadArray});
        // TODO: eventually we want separate tads here
        NativeOpExecutioner::execBroadcastBool(_context, op, this->_buffer, this->_shapeInfo, this->_bufferD, this->_shapeInfoD,
                                               tadArray->_buffer, tadArray->_shapeInfo, tadArray->_bufferD, tadArray->_shapeInfoD,
                                               result->_buffer, result->_shapeInfo, result->_bufferD, result->_shapeInfoD, (int*)devicePtrs[0], (int)copy.size(), (Nd4jLong*)devicePtrs[1], (Nd4jLong*)devicePtrs[2], nullptr, nullptr);
        result->tickWriteDevice();
    }

    //////////////////////////////////////////////////////////////////////////
    NDArray NDArray::applyTrueBroadcast(nd4j::BroadcastOpsTuple op, const NDArray& other, ExtraArguments *extraArgs) const {
        Nd4jLong* newShapeInfo = nullptr;
        if(!ShapeUtils::evalBroadcastShapeInfo(*this, &other, true, newShapeInfo, _context->getWorkspace()))          // the rank of new array = max->rankOf)()
            throw std::runtime_error("NDArray::applyTrueBroadcast method: the shapes of this and other arrays are not suitable for broadcast operation !");
        NDArray result(newShapeInfo, true, this->_context);

        // if workspace is not null - do not call delete.
        if (_context->getWorkspace() == nullptr)
            delete[] newShapeInfo;

        this->applyTrueBroadcast(op, &other, &result, false, extraArgs);

        return result;
    }
  
    ////////////////////////////////////////////////////////////////////////
    void NDArray::applyIndexReduce(nd4j::indexreduce::Ops op, NDArray* target, const std::vector<int>& dimensions, const ExtraArguments *extraParams) const {
        if (isS())
            throw std::runtime_error("NDArray::applyIndexReduce: you can't use this method on String array!");

        if (target->dataType() != nd4j::DataType::INT64)
            throw std::runtime_error("NDArray::applyIndexReduce operations return INT64");
        
        void* params = extraParams != nullptr ? const_cast<ExtraArguments*>(extraParams)->argumentsAsT(this->dataType()) : nullptr;

        if (target->isScalar()) {
            //target->_buffer[0] = functions::indexreduce::IndexReduce<T>::template execScalar<OpName>(_buffer, _shapeInfo, const_cast<T*>(extraParams));
            NativeOpExecutioner::execIndexReduceScalar(_context, op, _buffer, _shapeInfo, _bufferD, _shapeInfoD, params, target->_buffer, target->_shapeInfo, target->_bufferD, target->_shapeInfoD);

            auto cudaResult = hipStreamSynchronize(*_context->getCudaStream());        
            if (cudaResult != 0) throw cuda_exception::build("NDArray::applyIndexReduce cuda failed !", cudaResult);            
        } 
        else {

            std::vector<int> copy(dimensions);
            shape::checkDimensions(rankOf(), copy);

            shape::TAD xTad(_shapeInfo, copy.data(), copy.size());
            xTad.createTadOnlyShapeInfo();
            xTad.createOffsets();

            // device memory allocation for tads
            Nd4jLong *xTadShapeInfo, *xTadOffsets;
            int* dims;
            
            auto cudaResult = hipMalloc(reinterpret_cast<void **>(&dims), copy.size() * sizeof(int));
            if (cudaResult != 0) throw cuda_exception::build("NDArray::applyIndexReduce cuda memory allocation failed !", cudaResult);
            
            cudaResult = hipMalloc(reinterpret_cast<void **>(&xTadShapeInfo), shape::shapeInfoByteLength(xTad.tadOnlyShapeInfo));
            if (cudaResult != 0) throw cuda_exception::build("NDArray::applyIndexReduce cuda memory allocation failed !", cudaResult);
            
            cudaResult = hipMalloc(reinterpret_cast<void **>(&xTadOffsets), xTad.numTads * sizeof(Nd4jLong));
            if (cudaResult != 0) throw cuda_exception::build("NDArray::applyIndexReduce cuda memory allocation failed !", cudaResult);

            hipMemcpyAsync(dims, copy.data(), copy.size() * sizeof(int), hipMemcpyHostToDevice, *_context->getCudaStream());
            hipMemcpyAsync(xTadShapeInfo, xTad.tadOnlyShapeInfo, shape::shapeInfoByteLength(xTad.tadOnlyShapeInfo), hipMemcpyHostToDevice, *_context->getCudaStream());
            hipMemcpyAsync(xTadOffsets, xTad.tadOffsets, xTad.numTads * sizeof(Nd4jLong), hipMemcpyHostToDevice, *_context->getCudaStream());

            NativeOpExecutioner::execIndexReduce(_context, op, _buffer, _shapeInfo, _bufferD, _shapeInfoD, params,target->_buffer, target->_shapeInfo, target->_bufferD, target->_shapeInfoD,dims, copy.size(),xTadShapeInfo, xTadOffsets);

            cudaResult = hipStreamSynchronize(*_context->getCudaStream());        
            if (cudaResult != 0) throw cuda_exception::build("NDArray::applyIndexReduce cuda failed !", cudaResult);

            hipFree(dims); hipFree(xTadShapeInfo); hipFree(xTadOffsets);
        }
        NDArray::registerSpecialUse({target}, {this});
    }
    
    ////////////////////////////////////////////////////////////////////////
    // reduce dimensions in this array relying on index operations
    NDArray* NDArray::applyIndexReduce(nd4j::indexreduce::Ops op,const std::vector<int>& dimensions, const ExtraArguments* extraParams ) const {
        if (isS())
            throw std::runtime_error("NDArray::applyIndexReduce: you can't use this method on String array!");

        std::vector<int> copy(dimensions);
        shape::checkDimensions(rankOf(), copy);

        auto newShape = ShapeUtils::evalReduceShapeInfo('c', copy, *this, false, false, _context->getWorkspace());
        ArrayOptions::setDataType(newShape, nd4j::INT64);
        auto result = new NDArray(newShape, true, _context, true);

        void* params = extraParams != nullptr ? const_cast<ExtraArguments*>(extraParams)->argumentsAsT(this->dataType()) : nullptr;        

        if (rankOf() == copy.size()) {
            NativeOpExecutioner::execIndexReduceScalar(_context, op, _buffer, _shapeInfo, _bufferD, _shapeInfoD, params, result->_buffer, result->_shapeInfo, result->_bufferD, result->_shapeInfoD);

            auto cudaResult = hipStreamSynchronize(*_context->getCudaStream());        
            if (cudaResult != 0) throw cuda_exception::build("NDArray::applyIndexReduce cuda failed !", cudaResult);            
        } 
        else {            

            shape::TAD xTad(_shapeInfo, copy.data(), copy.size());
            xTad.createTadOnlyShapeInfo();
            xTad.createOffsets();

            // device memory allocation for tads
            Nd4jLong *xTadShapeInfo, *xTadOffsets;
            int* dims;
            
            auto cudaResult = hipMalloc(reinterpret_cast<void **>(&dims), copy.size() * sizeof(int));
            if (cudaResult != 0) throw cuda_exception::build("NDArray::applyIndexReduce cuda memory allocation failed !", cudaResult);
            
            cudaResult = hipMalloc(reinterpret_cast<void **>(&xTadShapeInfo), shape::shapeInfoByteLength(xTad.tadOnlyShapeInfo));
            if (cudaResult != 0) throw cuda_exception::build("NDArray::applyIndexReduce cuda memory allocation failed !", cudaResult);
            
            cudaResult = hipMalloc(reinterpret_cast<void **>(&xTadOffsets), xTad.numTads * sizeof(Nd4jLong));
            if (cudaResult != 0) throw cuda_exception::build("NDArray::applyIndexReduce cuda memory allocation failed !", cudaResult);

            hipMemcpyAsync(dims, copy.data(), copy.size() * sizeof(int), hipMemcpyHostToDevice, *_context->getCudaStream());
            hipMemcpyAsync(xTadShapeInfo, xTad.tadOnlyShapeInfo, shape::shapeInfoByteLength(xTad.tadOnlyShapeInfo), hipMemcpyHostToDevice, *_context->getCudaStream());
            hipMemcpyAsync(xTadOffsets, xTad.tadOffsets, xTad.numTads * sizeof(Nd4jLong), hipMemcpyHostToDevice, *_context->getCudaStream());

            NativeOpExecutioner::execIndexReduce(_context, op, _buffer, _shapeInfo, _bufferD, _shapeInfoD,
                                                params,
                                                result->_buffer, result->_shapeInfo, result->_bufferD, result->_shapeInfoD,
                                                dims, copy.size(),
                                                xTadShapeInfo, xTadOffsets);

            cudaResult = hipStreamSynchronize(*_context->getCudaStream());        
            if (cudaResult != 0) throw cuda_exception::build("NDArray::applyIndexReduce cuda failed !", cudaResult);

            hipFree(dims); hipFree(xTadShapeInfo); hipFree(xTadOffsets);
        }
        
        NDArray::registerSpecialUse({result}, {this});
        
        return result;
    }

    ////////////////////////////////////////////////////////////////////////
    // apply reduce3 operations to this and other array, return result in new output array
    NDArray* NDArray::applyReduce3(nd4j::reduce3::Ops op, const NDArray* other, const ExtraArguments* extraParams) const {
        if (isS())
            throw std::runtime_error("NDArray::applyReduce3 method: you can't use this method on String array!");
        if(_dataType != other->_dataType)
            throw std::runtime_error("NDArray::applyReduce3 method: the types of this and other arrays must be the same !");
        // check shapes consistency
        if(!isSameShape(other))
            throw std::runtime_error("NDArray::applyReduce3 method: the shapes of this and other arrays must be the same !");
        // create shapeInfo for scalar
        auto newShape = ShapeBuilders::createScalarShapeInfo(DataTypeUtils::pickFloatingType(_dataType), _context->getWorkspace());
        // create output array (scalar)
        auto result = new NDArray(newShape, true, _context, true);
        // create dynamic array of extra parameters if array extraParams is empty (==nullptr)
        void* params = extraParams != nullptr ? const_cast<ExtraArguments*>(extraParams)->argumentsAsT(this->dataType()) : nullptr;        

        NativeOpExecutioner::execReduce3Scalar(_context, op, _buffer, _shapeInfo, _bufferD, _shapeInfoD, params, other->_buffer, other->_shapeInfo, other->_bufferD, other->_shapeInfoD, result->_buffer, result->_shapeInfo, result->_bufferD, result->_shapeInfoD);

        auto cudaResult = hipStreamSynchronize(*_context->getCudaStream());        
        if (cudaResult != 0) throw cuda_exception::build("NDArray::applyReduce3 cuda failed !", cudaResult);

        NDArray::registerSpecialUse({result}, {this, other});

        return result;
    }

    ////////////////////////////////////////////////////////////////////////
    // apply reduce3 (exec) operations to this and other array, return result in new output array
    NDArray* NDArray::applyReduce3(nd4j::reduce3::Ops op, const NDArray* other, const std::vector<int>& dimensions, const ExtraArguments* extraParams) const {
        
        if (isS())
            throw std::runtime_error("NDArray::applyReduce3: you can't use this method on String array!");
        if(_dataType != other->_dataType)
            throw std::runtime_error("NDArray::applyReduce3 method: the types of this and other arrays must be the same !");

        std::vector<int> copy(dimensions);
        shape::checkDimensions(rankOf(), copy);
        shape::checkDimensions(other->rankOf(), copy);

        if(!isActualOnDeviceSide()) 
            syncToDevice();

        if(!other->isActualOnDeviceSide())
            other->syncToDevice();

        auto newShape = ShapeUtils::evalReduceShapeInfo('c', copy, *this, false, false, _context->getWorkspace());
        ArrayOptions::setDataType(newShape, DataTypeUtils::pickFloatingType(_dataType));
        auto result = new NDArray(newShape, true, _context, true);
        // create temporary dynamic array of extra parameters if array extraParams is empty (==nullptr)
        void* params = extraParams != nullptr ? const_cast<ExtraArguments*>(extraParams)->argumentsAsT(this->dataType()) : nullptr;

        // perform calculations
        if(rankOf() == copy.size() && other->rankOf() == copy.size()) {
            NativeOpExecutioner::execReduce3Scalar(_context, op, _buffer, _shapeInfo, _bufferD, _shapeInfoD, params, other->_buffer, other->_shapeInfo, other->_bufferD, other->_shapeInfoD, result->_buffer, result->shapeInfo(), result->specialBuffer(), result->specialShapeInfo());
            auto cudaResult = hipStreamSynchronize(*_context->getCudaStream());        
            if (cudaResult != 0) throw cuda_exception::build("NDArray::applyReduce3 cuda failed !", cudaResult);
        }
        else {
            
            // evaluate xTad data 
            shape::TAD xTad(_shapeInfo, copy.data(), copy.size());
            xTad.createTadOnlyShapeInfo();
            xTad.createOffsets();

            // evaluate yTad data
            shape::TAD yTad(other->_shapeInfo, copy.data(), copy.size());         
            yTad.createTadOnlyShapeInfo();
            yTad.createOffsets();

            if(!shape::equalsSoft(xTad.tadOnlyShapeInfo, yTad.tadOnlyShapeInfo) || (xTad.numTads != yTad.numTads && xTad.numTads != 1 && yTad.numTads != 1))
                throw std::runtime_error("NDArray::applyReduce3 cuda method: arrays tads are inconsistent !");

            // device memory allocation for tads
            Nd4jLong *xTadShapeInfo, *xTadOffsets, *yTadShapeInfo, *yTadOffsets;
            int *dims;

            auto cudaResult = hipMalloc(reinterpret_cast<void **>(&dims), copy.size() * sizeof(int));
            if (cudaResult != 0) throw cuda_exception::build("NDArray::applyReduce3 cuda memory allocation failed !", cudaResult);
            
            cudaResult = hipMalloc(reinterpret_cast<void **>(&xTadShapeInfo), shape::shapeInfoByteLength(xTad.tadOnlyShapeInfo));
            if (cudaResult != 0) throw cuda_exception::build("NDArray::applyReduce3 cuda memory allocation failed !", cudaResult);

            cudaResult = hipMalloc(reinterpret_cast<void **>(&yTadShapeInfo), shape::shapeInfoByteLength(yTad.tadOnlyShapeInfo));
            if (cudaResult != 0) throw cuda_exception::build("NDArray::applyReduce3 cuda memory allocation failed !", cudaResult);

            cudaResult = hipMalloc(reinterpret_cast<void **>(&xTadOffsets), xTad.numTads * sizeof(Nd4jLong));
            if (cudaResult != 0) throw cuda_exception::build("NDArray::applyReduce3 cuda memory allocation failed !", cudaResult);

            cudaResult = hipMalloc(reinterpret_cast<void **>(&yTadOffsets), yTad.numTads * sizeof(Nd4jLong));
            if (cudaResult != 0) throw cuda_exception::build("NDArray::applyReduce3 cuda memory allocation failed !", cudaResult);

            hipMemcpyAsync(dims, copy.data(), copy.size() * sizeof(int), hipMemcpyHostToDevice, *_context->getCudaStream());
            hipMemcpyAsync(xTadShapeInfo, xTad.tadOnlyShapeInfo, shape::shapeInfoByteLength(xTad.tadOnlyShapeInfo), hipMemcpyHostToDevice, *_context->getCudaStream());
            hipMemcpyAsync(yTadShapeInfo, yTad.tadOnlyShapeInfo, shape::shapeInfoByteLength(yTad.tadOnlyShapeInfo), hipMemcpyHostToDevice, *_context->getCudaStream());
            hipMemcpyAsync(xTadOffsets, xTad.tadOffsets, xTad.numTads * sizeof(Nd4jLong), hipMemcpyHostToDevice, *_context->getCudaStream());
            hipMemcpyAsync(yTadOffsets, yTad.tadOffsets, yTad.numTads * sizeof(Nd4jLong), hipMemcpyHostToDevice, *_context->getCudaStream());

            NativeOpExecutioner::execReduce3(_context, op, _buffer, _shapeInfo, _bufferD, _shapeInfoD, params, other->_buffer, other->_shapeInfo, other->_bufferD, other->_shapeInfoD, result->_buffer, result->_shapeInfo, result->_bufferD, result->_shapeInfoD, dims, copy.size(), xTadShapeInfo, xTadOffsets,  yTadShapeInfo, yTadOffsets);

            cudaResult = hipStreamSynchronize(*_context->getCudaStream());        
            if (cudaResult != 0) throw cuda_exception::build("NDArray::applyReduce3 cuda failed !", cudaResult);

            hipFree(dims); hipFree(xTadShapeInfo); hipFree(xTadOffsets); hipFree(yTadShapeInfo); hipFree(yTadOffsets);            
        }        

        NDArray::registerSpecialUse({result}, {this, other});

        return result;
    }

    ////////////////////////////////////////////////////////////////////////
    // apply reduce3 (execAll) operations to this and other array, return result in new output array
    NDArray* NDArray::applyAllReduce3(nd4j::reduce3::Ops op, const NDArray *other, const std::vector<int>& dimensions, const ExtraArguments* extraParams) const {
        if (isS())
            throw std::runtime_error("NDArray::applyAllReduce3: you can't use this method on String array!");
        if(_dataType != other->_dataType)
            throw std::runtime_error("NDArray::applyAllReduce3 method: the types of this and other arrays must be the same !");

        // be careful, copy array may undergo changes (sort, transformation of negative dimensions to positive, duplicates removing )
        std::vector<int> copy(dimensions);
        shape::checkDimensions(rankOf(), copy);
        shape::checkDimensions(other->rankOf(), copy);
        
        // create tads
        shape::TAD xTad(_shapeInfo, copy.data(), copy.size());
        xTad.createTadOnlyShapeInfo();
        xTad.createOffsets();

        shape::TAD yTad(other->_shapeInfo, copy.data(), copy.size());
        yTad.createTadOnlyShapeInfo();
        yTad.createOffsets();

        // check tads shapes
        if(!shape::equalsSoft(xTad.tadOnlyShapeInfo, yTad.tadOnlyShapeInfo))
            throw std::runtime_error("NDArray::applyAllReduce3 method: the shapes of array tads are different !");

        // set newShape for output array
        Nd4jLong *newShape = nullptr;
        ALLOCATE(newShape, _context->getWorkspace(), 8, Nd4jLong);
        newShape[0] = 2;        // output rank is always equal to 2 for execAll case
        newShape[1] = xTad.numTads;
        newShape[2] = yTad.numTads;
        ShapeUtils::updateStridesAndType(newShape, DataTypeUtils::pickFloatingType(_dataType), 'c');
        // create output array
        auto result = new NDArray(newShape, true, _context, true);

        NDArray::prepareSpecialUse({result}, {const_cast<NDArray*>(this), const_cast<NDArray*>(other)});

        // create dynamic array of extra parameters if array extraParams is empty (==nullptr)
        void* params = extraParams != nullptr ? const_cast<ExtraArguments*>(extraParams)->argumentsAsT(this->dataType()) : nullptr;

         // device memory allocation for tads
        Nd4jLong *xTadShapeInfo, *xTadOffsets, *yTadShapeInfo, *yTadOffsets;
        int *dims;     
        
        auto cudaResult = hipMalloc(reinterpret_cast<void **>(&dims), copy.size() * sizeof(int));
        if (cudaResult != 0) throw cuda_exception::build("NDArray::applyAllReduce3 cuda memory allocation failed !", cudaResult);

        cudaResult = hipMalloc(reinterpret_cast<void **>(&xTadShapeInfo), shape::shapeInfoByteLength(xTad.tadOnlyShapeInfo));
        if (cudaResult != 0) throw cuda_exception::build("NDArray::applyAllReduce3 cuda memory allocation failed !", cudaResult);

        cudaResult = hipMalloc(reinterpret_cast<void **>(&yTadShapeInfo), shape::shapeInfoByteLength(yTad.tadOnlyShapeInfo));
        if (cudaResult != 0) throw cuda_exception::build("NDArray::applyAllReduce3 cuda memory allocation failed !", cudaResult);

        cudaResult = hipMalloc(reinterpret_cast<void **>(&xTadOffsets), xTad.numTads * sizeof(Nd4jLong));
        if (cudaResult != 0) throw cuda_exception::build("NDArray::applyAllReduce3 cuda memory allocation failed !", cudaResult);

        cudaResult = hipMalloc(reinterpret_cast<void **>(&yTadOffsets), yTad.numTads * sizeof(Nd4jLong));
        if (cudaResult != 0) throw cuda_exception::build("NDArray::applyAllReduce3 cuda memory allocation failed !", cudaResult);

        hipMemcpyAsync(dims, copy.data(), copy.size() * sizeof(int), hipMemcpyHostToDevice, *_context->getCudaStream());  
        hipMemcpyAsync(xTadShapeInfo, xTad.tadOnlyShapeInfo, shape::shapeInfoByteLength(xTad.tadOnlyShapeInfo), hipMemcpyHostToDevice, *_context->getCudaStream());
        hipMemcpyAsync(yTadShapeInfo, yTad.tadOnlyShapeInfo, shape::shapeInfoByteLength(yTad.tadOnlyShapeInfo), hipMemcpyHostToDevice, *_context->getCudaStream());
        hipMemcpyAsync(xTadOffsets, xTad.tadOffsets, xTad.numTads * sizeof(Nd4jLong), hipMemcpyHostToDevice, *_context->getCudaStream());
        hipMemcpyAsync(yTadOffsets, yTad.tadOffsets, yTad.numTads * sizeof(Nd4jLong), hipMemcpyHostToDevice, *_context->getCudaStream());

        NativeOpExecutioner::execReduce3All(_context, op, _buffer, _shapeInfo, _bufferD, _shapeInfoD, params,other->_buffer, other->_shapeInfo, other->_bufferD, other->_shapeInfoD,result->_buffer,result->_shapeInfo, result->_bufferD, result->_shapeInfoD,dims, copy.size(), xTadShapeInfo, xTadOffsets, yTadShapeInfo, yTadOffsets);

        cudaResult = hipStreamSynchronize(*_context->getCudaStream());        
        if (cudaResult != 0) throw cuda_exception::build("NDArray::applyAllReduce3 cuda failed !", cudaResult);

        hipFree(dims); hipFree(xTadShapeInfo); hipFree(xTadOffsets); hipFree(yTadShapeInfo); hipFree(yTadOffsets);            

        NDArray::registerSpecialUse({result}, {this, other});
        
        return result;
    }

    void NDArray::prepareSpecialUse(const std::initializer_list<const NDArray*>& writeList, const std::initializer_list<const NDArray*>& readList, bool synchronizeWritables) {
        for (auto a:writeList) {
            if (synchronizeWritables && !a->isActualOnDeviceSide())
                a->syncToDevice();

            a->tickWriteDevice();
        }

        for (auto a:readList) {
            if (!a->isActualOnDeviceSide())
                a->syncToDevice();
        }
    }

    void NDArray::registerSpecialUse(const std::initializer_list<const NDArray*>& writeList, const std::initializer_list<const NDArray*>& readList) {
        // no-op
        for (auto p:writeList) {
            //if (!p->isActualOnDeviceSide())
            //    p->syncToDevice();

            p->tickWriteDevice();
        }

        for (auto p:readList) {
            //if (!p->isActualOnDeviceSide())
            //    p->syncToDevice();

            p->tickReadDevice();
        }
    }
    
////////////////////////////////////////////////////////////////////////
// default destructor
NDArray::~NDArray() noexcept {
    if (isS()) {
        if (_isBuffAlloc && _context->getWorkspace() == nullptr && _buffer != nullptr) {
            for (int e = 0; e < lengthOf(); e++) {
                auto t = reinterpret_cast<utf8string**>(_buffer);
                delete t[e];
            };
            delete[] _buffer;
        }
    }
    else
    if (_isBuffAlloc)
        RELEASE(_buffer, _context->getWorkspace());

    if (_isShapeAlloc)
        RELEASE(_shapeInfo, _context->getWorkspace());


    if (_isShapeDAlloc)
        RELEASE_SPECIAL(_shapeInfoD, _context->getWorkspace());

    if (_isBuffDAlloc)
        RELEASE_SPECIAL(_bufferD, _context->getWorkspace());
}

//////////////////////////////////////////////////////////////////////////
void NDArray::setShapeInfo(Nd4jLong *shapeInfo) {

    if(_context->getWorkspace() == nullptr) {            
        if(_isShapeAlloc) delete []_shapeInfo;
        if(_isShapeDAlloc) RELEASE_SPECIAL(_shapeInfoD, nullptr);
    }    
        
    _shapeInfo = shapeInfo;

    if (shapeInfo != nullptr) {

        if(ArrayOptions::arrayType(shapeInfo) == ArrayType::EMPTY)
            _length = 0;
        else
            _length = shape::length(shapeInfo);
        
        _dataType = ArrayOptions::dataType(shapeInfo);
        ALLOCATE_SPECIAL(_shapeInfoD, _context->getWorkspace(), shape::shapeInfoLength(_shapeInfo), Nd4jLong);
        _isShapeDAlloc = true;
        syncShape();
    } 
    else {
        _dataType = nd4j::DataType::INHERIT;    
        _shapeInfoD = nullptr;
        _isShapeDAlloc = false;
    }
}

////////////////////////////////////////////////////////////////////////
void NDArray::setShapeInfo(Nd4jLong *shapeInfo, const nd4j::DataType dtype) {
    
    if(_context->getWorkspace() == nullptr) {            
        if(_isShapeAlloc) delete []_shapeInfo;
        if(_isShapeDAlloc) RELEASE_SPECIAL(_shapeInfoD, nullptr);
    }

    _shapeInfo = shapeInfo;

    if (shapeInfo != nullptr) {

        ArrayOptions::setDataType(_shapeInfo, dtype);

        if(ArrayOptions::arrayType(_shapeInfo) == ArrayType::EMPTY)
            _length = 0;
        else
            _length = shape::length(shapeInfo);

        _dataType = dtype;
        ALLOCATE_SPECIAL(_shapeInfoD, _context->getWorkspace(), shape::shapeInfoLength(_shapeInfo), Nd4jLong);
        _isShapeDAlloc = true;
        syncShape();
    } 
    else {
        _dataType = nd4j::DataType::INHERIT;    
        _shapeInfoD = nullptr;
        _isShapeDAlloc = false;
    }
}

////////////////////////////////////////////////////////////////////////
    NDArray* NDArray::varianceAlongDimension(nd4j::variance::Ops op, const bool biasCorrected, const std::vector<int>& dimensions) const {
        if (isS())
            throw std::runtime_error("NDArray::varianceAlongDimension: you can't use this method on String array!");

        std::vector<int> copy(dimensions);
        if (copy.size() > 1)
            std::sort(copy.begin(), copy.end());

        auto newShape = ShapeUtils::evalReduceShapeInfo('c', copy, *this, false, false, _context->getWorkspace());
        ArrayOptions::setDataType(newShape, DataTypeUtils::pickFloatingType(_dataType));
        auto result = new NDArray(newShape, true, _context, true);

        NDArray::prepareSpecialUse({result}, {this});

        if(rankOf() == copy.size() || copy.empty())
            NativeOpExecutioner::execSummaryStatsScalar(_context, op, _buffer, _shapeInfo, _bufferD, _shapeInfoD, nullptr, result->buffer(), result->shapeInfo(), result->specialBuffer(), result->specialShapeInfo(), biasCorrected);
        else {
            Nd4jLong *xTadShapeInfo;
            Nd4jLong *xTadOffsets;
            int *tmp;
            ALLOCATE_SPECIAL(tmp, _context->getWorkspace(), copy.size(), int);

            hipMemcpyAsync(tmp, copy.data(), copy.size() * sizeof(int), hipMemcpyHostToDevice, *_context->getCudaStream());
            shape::TAD tad(this->getShapeInfo(), copy.data(), copy.size());
            tad.createTadOnlyShapeInfo();
            tad.createOffsets();
            ALLOCATE_SPECIAL(xTadOffsets, _context->getWorkspace(), tad.numTads, Nd4jLong);
            ALLOCATE_SPECIAL(xTadShapeInfo, _context->getWorkspace(), shape::shapeInfoLength(tad.tadOnlyShapeInfo), Nd4jLong);

            hipMemcpyAsync(xTadOffsets, tad.tadOffsets, tad.numTads * sizeof(Nd4jLong), hipMemcpyHostToDevice, *_context->getCudaStream());
            hipMemcpyAsync(xTadShapeInfo, tad.tadOnlyShapeInfo, shape::shapeInfoByteLength(tad.tadOnlyShapeInfo), hipMemcpyHostToDevice, *_context->getCudaStream());

            NativeOpExecutioner::execSummaryStats(_context, op, _buffer, _shapeInfo, _bufferD, _shapeInfoD, nullptr, result->_buffer, result->_shapeInfo, result->_bufferD, result->_shapeInfoD, tmp, copy.size(), xTadShapeInfo, xTadOffsets, biasCorrected);

            auto res = hipStreamSynchronize(*_context->getCudaStream());
            if (res != 0)
                throw cuda_exception::build("varianceAlongDimension failed", res);

            RELEASE_SPECIAL(tmp, _context->getWorkspace());
            RELEASE_SPECIAL(xTadShapeInfo, _context->getWorkspace());
            RELEASE_SPECIAL(xTadOffsets, _context->getWorkspace());
        }


        NDArray::registerSpecialUse({result}, {this});

        return result;
    }

    void NDArray::varianceAlongDimension(nd4j::variance::Ops op, const NDArray *target, const bool biasCorrected, const std::vector<int>& dimensions) {
        if (isS())
            throw std::runtime_error("NDArray::varianceAlongDimension: you can't use this method on String array!");

        std::vector<int> copy(dimensions);
        if (copy.size() > 1)
            std::sort(copy.begin(), copy.end());

        if (!target->isR())
            throw std::runtime_error("NDArray::varianceAlongDimension: target array must have FLOAT type");

        if(rankOf() == copy.size() || copy.empty())
            NativeOpExecutioner::execSummaryStatsScalar(_context, op, _buffer, _shapeInfo, _bufferD, _shapeInfoD, nullptr, target->getBuffer(), target->getShapeInfo(), target->getSpecialBuffer(), target->getSpecialShapeInfo(), biasCorrected);
        else {
            Nd4jLong *xTadShapeInfo;
            Nd4jLong *xTadOffsets;
            int *tmp;
            ALLOCATE_SPECIAL(tmp, _context->getWorkspace(), copy.size(), int);

            hipMemcpyAsync(tmp, copy.data(), copy.size() * sizeof(int), hipMemcpyHostToDevice, *_context->getCudaStream());
            shape::TAD tad(this->getShapeInfo(), copy.data(), copy.size());
            tad.createTadOnlyShapeInfo();
            tad.createOffsets();
            ALLOCATE_SPECIAL(xTadOffsets, _context->getWorkspace(), tad.numTads, Nd4jLong);
            ALLOCATE_SPECIAL(xTadShapeInfo, _context->getWorkspace(), shape::shapeInfoLength(tad.tadOnlyShapeInfo), Nd4jLong);

            NativeOpExecutioner::execSummaryStats(_context, op, _buffer, _shapeInfo, _bufferD, _shapeInfoD, nullptr, target->_buffer, target->_shapeInfo, target->_bufferD, target->_shapeInfoD, tmp, copy.size(), xTadShapeInfo, xTadOffsets, biasCorrected);

            auto res = hipStreamSynchronize(*_context->getCudaStream());
            if (res != 0)
                throw cuda_exception::build("varianceAlongDimension failed", res);

            RELEASE_SPECIAL(tmp, _context->getWorkspace());
            RELEASE_SPECIAL(xTadShapeInfo, _context->getWorkspace());
            RELEASE_SPECIAL(xTadOffsets, _context->getWorkspace());
        }
    }

////////////////////////////////////////////////////////////////////////
    // This method returns true if two arrays are equal, with custom or default Eps value of 1e-5, false otherwise
    bool NDArray::equalsTo(const NDArray *other, double eps) const {
        if (this->dataType() != other->dataType() || lengthOf() != other->lengthOf())
            return false;

        // we need to be able to compare [1, len] to [len]
        if ((rankOf() == 1 && other->rankOf() == 2) || (rankOf() == 2 && other->rankOf() == 1)) {
            // FIXME: do something here?
        } else if (!shape::equalsSoft(_shapeInfo, other->_shapeInfo))
            return false;

        NDArray tmp = NDArrayFactory::create<float>(0LL, _context); // scalar = 0
        NDArray::prepareSpecialUse({&tmp}, {this, other});

        ExtraArguments extras({eps}); 
        NativeOpExecutioner::execReduce3Scalar(_context, reduce3::EqualsWithEps, _buffer, _shapeInfo, _bufferD, _shapeInfoD, extras.argumentsAsT(DataType::FLOAT32), other->_buffer, other->_shapeInfo, other->_bufferD, other->_shapeInfoD, tmp.buffer(), tmp.shapeInfo(), tmp._bufferD, tmp._shapeInfoD);

        NDArray::registerSpecialUse({&tmp}, {this, other});

        auto res = hipStreamSynchronize(*_context->getCudaStream());
        if (res != 0)
            throw cuda_exception::build("NDArray::equalsTo failed", res);

        auto r = tmp.e<Nd4jLong>(0);
        //nd4j_printf("equalsTo result: [%lld]\n", r);
        if (r > 0LL)
            return false;

        return true;
    }

    //////////////////////////////////////////////////////////////////////////
    bool NDArray::permutei(const int* dimensions, const int rank) {

        // check if current object is _shapeInfo owner
        if (!_isShapeAlloc) {             // if _shapeInfo is not its own
            _shapeInfo = ShapeUtils::evalPermShapeInfo(dimensions, rank, *this, _context->getWorkspace());
            _isShapeAlloc = true;
        } 
        else {
            if (!nonNull())
                throw std::runtime_error("NDArray::permutei method cuda: wrong arguments in permutei method: array is nullptr!");
            if (rank != rankOf())
                throw std::runtime_error("NDArray::permutei method cuda: wrong arguments in permutei method: rank is not suitable!");
            shape::doPermuteShapeInfo(_shapeInfo, dimensions);
        }

        syncShape();

        return true;
    }

    //////////////////////////////////////////////////////////////////////////
    bool NDArray::permutei(const Nd4jLong* dimensions, const int rank) {

        // check if current object is _shapeInfo owner
        if (!_isShapeAlloc) {             // if _shapeInfo is not its own
            _shapeInfo = ShapeUtils::evalPermShapeInfo(dimensions, rank, *this, _context->getWorkspace());
            _isShapeAlloc = true;
        } 
        else {
            if (!nonNull())
                throw std::runtime_error("NDArray::permutei method: wrong arguments in permutei method: array is nullptr!");
            if (rank != rankOf())
                throw std::runtime_error("NDArray::permutei method: wrong arguments in permutei method: rank is not suitable!");
            shape::doPermuteShapeInfo(_shapeInfo, dimensions);
        }

        syncShape();
        
        return true;
    }

//////////////////////////////////////////////////////////////////////////
// method reduces array by excluding its shapes along axes present in dimensions vector
void NDArray::reduceAlongDimension(nd4j::reduce::FloatOps op, NDArray* target, const std::vector<int>& dimensions, const bool keepDims, const bool supportOldShapes, const bool checkTargetShape) const {

    if (isS())
        throw std::runtime_error("NDArray::reduceAlongDimension FloatOps cuda: you can't use this method on String array!");
    if (target == nullptr || !target->isR())
        throw std::invalid_argument("NDArray::reduceAlongDimension FloatOps cuda: requires target array to be present and have type form real space!");

    std::vector<int> copy(dimensions);
    shape::checkDimensions(rankOf(), copy);

    if(checkTargetShape) {
        auto newShape = ShapeUtils::evalReduceShapeInfo(target->ordering(), copy, *this, keepDims, supportOldShapes, _context->getWorkspace());
        if(!shape::shapeEquals(newShape, target->getShapeInfo()))
            throw std::runtime_error("NDArray::reduceAlongDimension FloatOps cuda: wrong target shape!");
        RELEASE(newShape, _context->getWorkspace());
    }

    if(rankOf() == copy.size() || copy.empty()) {        
        NativeOpExecutioner::execReduceFloatScalar(_context, op, _buffer, _shapeInfo, _bufferD, _shapeInfoD,nullptr, target->_buffer, target->_shapeInfo, target->_bufferD, target->_shapeInfoD);        

        auto cudaResult = hipStreamSynchronize(*_context->getCudaStream());
        if (cudaResult != 0) throw cuda_exception::build("NDArray::reduceAlongDimension FloatOps cuda failed !", cudaResult);
    }
    else {

        shape::TAD xTad(_shapeInfo, copy.data(), copy.size());
        xTad.createTadOnlyShapeInfo();
        xTad.createOffsets();

         // device memory allocation for tads
        Nd4jLong *xTadShapeInfo, *xTadOffsets;
        int *dims;     
        
        auto cudaResult = hipMalloc(reinterpret_cast<void **>(&dims), copy.size() * sizeof(int));
        if (cudaResult != 0) throw cuda_exception::build("NDArray::reduceAlongDimension FloatOps cuda memory allocation failed !", cudaResult);

        cudaResult = hipMalloc(reinterpret_cast<void **>(&xTadShapeInfo), shape::shapeInfoByteLength(xTad.tadOnlyShapeInfo));
        if (cudaResult != 0) throw cuda_exception::build("NDArray::reduceAlongDimension FloatOps cuda memory allocation failed !", cudaResult);

        cudaResult = hipMalloc(reinterpret_cast<void **>(&xTadOffsets), xTad.numTads * sizeof(Nd4jLong));
        if (cudaResult != 0) throw cuda_exception::build("NDArray::reduceAlongDimension FloatOps cuda memory allocation failed !", cudaResult);

        hipMemcpyAsync(dims, copy.data(), copy.size() * sizeof(int), hipMemcpyHostToDevice, *_context->getCudaStream());  
        hipMemcpyAsync(xTadShapeInfo, xTad.tadOnlyShapeInfo, shape::shapeInfoByteLength(xTad.tadOnlyShapeInfo), hipMemcpyHostToDevice, *_context->getCudaStream());
        hipMemcpyAsync(xTadOffsets, xTad.tadOffsets, xTad.numTads * sizeof(Nd4jLong), hipMemcpyHostToDevice, *_context->getCudaStream());

        NativeOpExecutioner::execReduceFloat(_context, op, _buffer, _shapeInfo, _bufferD, _shapeInfoD, nullptr, target->_buffer, target->_shapeInfo, target->_bufferD, target->_shapeInfoD, dims, copy.size(), xTadShapeInfo, xTadOffsets);

        cudaResult = hipStreamSynchronize(*_context->getCudaStream());
        if (cudaResult != 0) throw cuda_exception::build("NDArray::reduceAlongDimension FloatOps cuda failed !", cudaResult);

        hipFree(dims); hipFree(xTadShapeInfo); hipFree(xTadOffsets);
    }
    NDArray::registerSpecialUse({target}, {this});
}

//////////////////////////////////////////////////////////////////////////
// method reduces array by excluding its shapes along axes present in dimensions vector
void NDArray::reduceAlongDimension(nd4j::reduce::SameOps op, NDArray* target, const std::vector<int>& dimensions, const bool keepDims, const bool supportOldShapes, const bool checkTargetShape) const {

    if (isS())
        throw std::runtime_error("NDArray::reduceAlongDimension SameOps cuda: you can't use this method on String array!");
    if (target == nullptr || target->_dataType != _dataType)
        throw std::runtime_error("NDArray::reduceAlongDimension SameOps cuda: requires target array to be present and have same dtype as input");

    std::vector<int> copy(dimensions);
    shape::checkDimensions(rankOf(), copy);

    if(checkTargetShape) {
        auto newShape = ShapeUtils::evalReduceShapeInfo(target->ordering(), copy, *this, keepDims, supportOldShapes, _context->getWorkspace());
        if(!shape::shapeEquals(newShape, target->getShapeInfo()))
            throw std::runtime_error("NDArray::reduceAlongDimension SameOps cuda: wrong target shape!");
        RELEASE(newShape, _context->getWorkspace());
    }

    if(rankOf() == copy.size() || copy.empty()) {        
        NativeOpExecutioner::execReduceSameScalar(_context, op, _buffer, _shapeInfo, _bufferD, _shapeInfoD, nullptr, target->_buffer, target->_shapeInfo, target->_bufferD, target->_shapeInfoD);        

        auto cudaResult = hipStreamSynchronize(*_context->getCudaStream());
        if (cudaResult != 0) throw cuda_exception::build("NDArray::reduceAlongDimension SameOps cuda failed !", cudaResult);
    }
    else {

        shape::TAD xTad(_shapeInfo, copy.data(), copy.size());
        xTad.createTadOnlyShapeInfo();
        xTad.createOffsets();

         // device memory allocation for tads
        Nd4jLong *xTadShapeInfo, *xTadOffsets;
        int *dims;     
        
        auto cudaResult = hipMalloc(reinterpret_cast<void **>(&dims), copy.size() * sizeof(int));
        if (cudaResult != 0) throw cuda_exception::build("NDArray::reduceAlongDimension SameOps cuda memory allocation failed !", cudaResult);

        cudaResult = hipMalloc(reinterpret_cast<void **>(&xTadShapeInfo), shape::shapeInfoByteLength(xTad.tadOnlyShapeInfo));
        if (cudaResult != 0) throw cuda_exception::build("NDArray::reduceAlongDimension SameOps cuda memory allocation failed !", cudaResult);

        cudaResult = hipMalloc(reinterpret_cast<void **>(&xTadOffsets), xTad.numTads * sizeof(Nd4jLong));
        if (cudaResult != 0) throw cuda_exception::build("NDArray::reduceAlongDimension SameOps cuda memory allocation failed !", cudaResult);

        hipMemcpyAsync(dims, copy.data(), copy.size() * sizeof(int), hipMemcpyHostToDevice, *_context->getCudaStream());  
        hipMemcpyAsync(xTadShapeInfo, xTad.tadOnlyShapeInfo, shape::shapeInfoByteLength(xTad.tadOnlyShapeInfo), hipMemcpyHostToDevice, *_context->getCudaStream());
        hipMemcpyAsync(xTadOffsets, xTad.tadOffsets, xTad.numTads * sizeof(Nd4jLong), hipMemcpyHostToDevice, *_context->getCudaStream());

        NativeOpExecutioner::execReduceSame(_context, op, _buffer, _shapeInfo, _bufferD, _shapeInfoD, nullptr, target->_buffer, target->_shapeInfo, target->_bufferD, target->_shapeInfoD, dims, copy.size(), xTadShapeInfo, xTadOffsets);

        cudaResult = hipStreamSynchronize(*_context->getCudaStream());
        if (cudaResult != 0) throw cuda_exception::build("NDArray::reduceAlongDimension SameOps cuda failed !", cudaResult);

        hipFree(dims); hipFree(xTadShapeInfo); hipFree(xTadOffsets);
    }
    NDArray::registerSpecialUse({target}, {this});
}

//////////////////////////////////////////////////////////////////////////
// method reduces array by excluding its shapes along axes present in dimensions vector
void NDArray::reduceAlongDimension(nd4j::reduce::BoolOps op, NDArray* target, const std::vector<int>& dimensions, const bool keepDims, const bool supportOldShapes, const bool checkTargetShape) const {

    if (isS())
        throw std::runtime_error("NDArray::reduceAlongDimension BoolOps cuda: you can't use this method on String array!");
    if (target == nullptr || !target->isB())
        throw std::invalid_argument("NDArray::reduceAlongDimension BoolOps cuda: requires target array to be present and have BOOL type!");    

    std::vector<int> copy(dimensions);
    shape::checkDimensions(rankOf(), copy);

    if(checkTargetShape) {
        auto newShape = ShapeUtils::evalReduceShapeInfo(target->ordering(), copy, *this, keepDims, supportOldShapes, _context->getWorkspace());
        if(!shape::shapeEquals(newShape, target->getShapeInfo()))
            throw std::runtime_error("NDArray::reduceAlongDimension BoolOps cuda: wrong target shape!");
        RELEASE(newShape, _context->getWorkspace());
    }

    if(rankOf() == copy.size() || copy.empty()) {        
        NativeOpExecutioner::execReduceBoolScalar(_context, op, _buffer, _shapeInfo, _bufferD, _shapeInfoD, nullptr, target->_buffer, target->_shapeInfo, target->_bufferD, target->_shapeInfoD);        

        auto cudaResult = hipStreamSynchronize(*_context->getCudaStream());
        if (cudaResult != 0) throw cuda_exception::build("NDArray::reduceAlongDimension BoolOps cuda failed !", cudaResult);
    }
    else {

        shape::TAD xTad(_shapeInfo, copy.data(), copy.size());
        xTad.createTadOnlyShapeInfo();
        xTad.createOffsets();

         // device memory allocation for tads
        Nd4jLong *xTadShapeInfo, *xTadOffsets;
        int *dims;     
        
        auto cudaResult = hipMalloc(reinterpret_cast<void **>(&dims), copy.size() * sizeof(int));
        if (cudaResult != 0) throw cuda_exception::build("NDArray::reduceAlongDimension BoolOps cuda memory allocation failed !", cudaResult);

        cudaResult = hipMalloc(reinterpret_cast<void **>(&xTadShapeInfo), shape::shapeInfoByteLength(xTad.tadOnlyShapeInfo));
        if (cudaResult != 0) throw cuda_exception::build("NDArray::reduceAlongDimension BoolOps cuda memory allocation failed !", cudaResult);

        cudaResult = hipMalloc(reinterpret_cast<void **>(&xTadOffsets), xTad.numTads * sizeof(Nd4jLong));
        if (cudaResult != 0) throw cuda_exception::build("NDArray::reduceAlongDimension BoolOps cuda memory allocation failed !", cudaResult);

        hipMemcpyAsync(dims, copy.data(), copy.size() * sizeof(int), hipMemcpyHostToDevice, *_context->getCudaStream());  
        hipMemcpyAsync(xTadShapeInfo, xTad.tadOnlyShapeInfo, shape::shapeInfoByteLength(xTad.tadOnlyShapeInfo), hipMemcpyHostToDevice, *_context->getCudaStream());
        hipMemcpyAsync(xTadOffsets, xTad.tadOffsets, xTad.numTads * sizeof(Nd4jLong), hipMemcpyHostToDevice, *_context->getCudaStream());

        NativeOpExecutioner::execReduceBool(_context, op, _buffer, _shapeInfo, _bufferD, _shapeInfoD, nullptr, target->_buffer, target->_shapeInfo, target->_bufferD, target->_shapeInfoD, dims, copy.size(), xTadShapeInfo, xTadOffsets);

        cudaResult = hipStreamSynchronize(*_context->getCudaStream());
        if (cudaResult != 0) throw cuda_exception::build("NDArray::reduceAlongDimension BoolOps cuda failed !", cudaResult);

        hipFree(dims); hipFree(xTadShapeInfo); hipFree(xTadOffsets);
    }
    NDArray::registerSpecialUse({target}, {this});
}

//////////////////////////////////////////////////////////////////////////
// method reduces array by excluding its shapes along axes present in dimensions vector
void NDArray::reduceAlongDimension(nd4j::reduce::LongOps op, NDArray* target, const std::vector<int>& dimensions, const bool keepDims, const bool supportOldShapes, const bool checkTargetShape) const {

    if (isS())
        throw std::runtime_error("NDArray::reduceAlongDimension LongOps cuda: you can't use this method on String array!");
    if (target == nullptr || target->_dataType != DataType::INT64)
        throw std::runtime_error("NDArray::reduceAlongDimension LongOps cuda: requires target array to be present and have type of INT64");

    std::vector<int> copy(dimensions);
    shape::checkDimensions(rankOf(), copy);

    if(checkTargetShape) {
        auto newShape = ShapeUtils::evalReduceShapeInfo(target->ordering(), copy, *this, keepDims, supportOldShapes, _context->getWorkspace());
        if(!shape::shapeEquals(newShape, target->getShapeInfo()))
            throw std::runtime_error("NDArray::reduceAlongDimension LongOps cuda: wrong target shape!");
        RELEASE(newShape, _context->getWorkspace());
    }

    if(rankOf() == copy.size() || copy.empty()) {        
        NativeOpExecutioner::execReduceLongScalar(_context, op, _buffer, _shapeInfo, _bufferD, _shapeInfoD, nullptr, target->_buffer, target->_shapeInfo, target->_bufferD, target->_shapeInfoD);        

        auto cudaResult = hipStreamSynchronize(*_context->getCudaStream());
        if (cudaResult != 0) throw cuda_exception::build("NDArray::reduceAlongDimension LongOps cuda failed !", cudaResult);
    }
    else {

        shape::TAD xTad(_shapeInfo, copy.data(), copy.size());
        xTad.createTadOnlyShapeInfo();
        xTad.createOffsets();

         // device memory allocation for tads
        Nd4jLong *xTadShapeInfo, *xTadOffsets;
        int *dims;     
        
        auto cudaResult = hipMalloc(reinterpret_cast<void **>(&dims), copy.size() * sizeof(int));
        if (cudaResult != 0) throw cuda_exception::build("NDArray::reduceAlongDimension LongOps cuda memory allocation failed !", cudaResult);

        cudaResult = hipMalloc(reinterpret_cast<void **>(&xTadShapeInfo), shape::shapeInfoByteLength(xTad.tadOnlyShapeInfo));
        if (cudaResult != 0) throw cuda_exception::build("NDArray::reduceAlongDimension LongOps cuda memory allocation failed !", cudaResult);

        cudaResult = hipMalloc(reinterpret_cast<void **>(&xTadOffsets), xTad.numTads * sizeof(Nd4jLong));
        if (cudaResult != 0) throw cuda_exception::build("NDArray::reduceAlongDimension LongOps cuda memory allocation failed !", cudaResult);

        hipMemcpyAsync(dims, copy.data(), copy.size() * sizeof(int), hipMemcpyHostToDevice, *_context->getCudaStream());  
        hipMemcpyAsync(xTadShapeInfo, xTad.tadOnlyShapeInfo, shape::shapeInfoByteLength(xTad.tadOnlyShapeInfo), hipMemcpyHostToDevice, *_context->getCudaStream());
        hipMemcpyAsync(xTadOffsets, xTad.tadOffsets, xTad.numTads * sizeof(Nd4jLong), hipMemcpyHostToDevice, *_context->getCudaStream());

        NativeOpExecutioner::execReduceLong(_context, op, _buffer, _shapeInfo, _bufferD, _shapeInfoD, nullptr, target->_buffer, target->_shapeInfo, target->_bufferD, target->_shapeInfoD, dims, copy.size(), xTadShapeInfo, xTadOffsets);

        cudaResult = hipStreamSynchronize(*_context->getCudaStream());
        if (cudaResult != 0) throw cuda_exception::build("NDArray::reduceAlongDimension LongOps cuda failed !", cudaResult);

        hipFree(dims); hipFree(xTadShapeInfo); hipFree(xTadOffsets);
    }
    NDArray::registerSpecialUse({target}, {this});
}

//////////////////////////////////////////////////////////////////////////
// This method sets value in linear buffer to position i
    template <typename T>
    void NDArray::p(const Nd4jLong i, const T value) {
        lazyAllocateBuffer();
        if (!isActualOnHostSide())
            syncToHost();

        if (i >= _length)
            throw std::invalid_argument("NDArray::p(i, value): input index is out of array length !");

        auto rp = getOffset(i);
        const void *pV = reinterpret_cast<const void*>(const_cast<T *>(&value));
        BUILD_SINGLE_PARTIAL_SELECTOR(this->dataType(), templatedSet<, T>(this->_buffer, rp, pV), LIBND4J_TYPES);
        tickWriteHost();
    }
    template void NDArray::p(const Nd4jLong i, const double value);
    template void NDArray::p(const Nd4jLong i, const float value);
    template void NDArray::p(const Nd4jLong i, const float16 value);
    template void NDArray::p(const Nd4jLong i, const bfloat16 value);
    template void NDArray::p(const Nd4jLong i, const Nd4jLong value);
    template void NDArray::p(const Nd4jLong i, const int value);
    template void NDArray::p(const Nd4jLong i, const int8_t value);
    template void NDArray::p(const Nd4jLong i, const uint8_t value);
    template void NDArray::p(const Nd4jLong i, const int16_t value);
    template void NDArray::p(const Nd4jLong i, const bool value);

    void NDArray::p(const Nd4jLong i, const NDArray& scalar) {
        
        if(!scalar.isScalar())
            throw std::invalid_argument("NDArray::p method: input array must be scalar!");
        if (i >= _length)
            throw std::invalid_argument("NDArray::p(i, NDArray_scalar): input index is out of array length !");

        if (!scalar.isActualOnDeviceSide())
            scalar.syncToHost();

        auto rp = getOffset(i);        
        BUILD_SINGLE_SELECTOR(scalar.dataType(), templatedSet, (_buffer, rp, scalar.dataType(), scalar.getBuffer()), LIBND4J_TYPES);
        tickWriteHost();
    }


//////////////////////////////////////////////////////////////////////////
// This method sets value in 2D matrix to position i, j

    template <typename T>
    void NDArray::p(const Nd4jLong i, const Nd4jLong j, const T value) {
        //(*this)(i,j) = value;
        if (rankOf() != 2 || i >= shapeOf()[0] || j >= shapeOf()[1])
            throw std::invalid_argument("NDArray:pe(i,j, value): one of input indexes is out of array length or rank!=2 !");

        lazyAllocateBuffer();
        if (!isActualOnHostSide())
            syncToHost();

        void *p = reinterpret_cast<void *>(const_cast<T *>(&value));        
        Nd4jLong coords[2] = {i, j};
        auto xOffset = shape::getOffset(0, shapeOf(), stridesOf(), coords, rankOf());
        BUILD_SINGLE_PARTIAL_SELECTOR(dataType(), templatedSet<, T>(this->_buffer, xOffset, p), LIBND4J_TYPES);
        tickWriteHost();
    }
    template void NDArray::p(const Nd4jLong i, const Nd4jLong j, const double value);
    template void NDArray::p(const Nd4jLong i, const Nd4jLong j, const float value);
    template void NDArray::p(const Nd4jLong i, const Nd4jLong j, const float16 value);
    template void NDArray::p(const Nd4jLong i, const Nd4jLong j, const bfloat16 value);
    template void NDArray::p(const Nd4jLong i, const Nd4jLong j, const Nd4jLong value);
    template void NDArray::p(const Nd4jLong i, const Nd4jLong j, const int value);
    template void NDArray::p(const Nd4jLong i, const Nd4jLong j, const int8_t value);
    template void NDArray::p(const Nd4jLong i, const Nd4jLong j, const uint8_t value);
    template void NDArray::p(const Nd4jLong i, const Nd4jLong j, const int16_t value);
    template void NDArray::p(const Nd4jLong i, const Nd4jLong j, const bool value);
    // template void NDArray::p(const Nd4jLong i, const Nd4jLong j, const utf8string value);

//////////////////////////////////////////////////////////////////////////
// This method sets value in 3D matrix to position i,j,k
    template <typename T>
    void NDArray::p(const Nd4jLong i, const Nd4jLong j, const Nd4jLong k, const T value) {
        //(*this)(i,j,k) = value;
        if (rankOf() != 3 || i >= shapeOf()[0] || j >= shapeOf()[1] || k >= shapeOf()[2])
            throw std::invalid_argument("NDArray:pe(i,j,k, value): one of input indexes is out of array length or rank!=3 !");

        lazyAllocateBuffer();
        if (!isActualOnHostSide())
            syncToHost();
        
        void *p = reinterpret_cast<void *>(const_cast<T *>(&value));        
        Nd4jLong coords[3] = {i, j, k};        
        auto xOffset = shape::getOffset(0, shapeOf(), stridesOf(), coords, rankOf());
        BUILD_SINGLE_PARTIAL_SELECTOR(dataType(), templatedSet<, T>(this->_buffer, xOffset, p), LIBND4J_TYPES);
        tickWriteHost();
    }
    template void NDArray::p(const Nd4jLong i, const Nd4jLong j, const Nd4jLong k, const double value);
    template void NDArray::p(const Nd4jLong i, const Nd4jLong j, const Nd4jLong k, const float value);
    template void NDArray::p(const Nd4jLong i, const Nd4jLong j, const Nd4jLong k, const float16 value);
    template void NDArray::p(const Nd4jLong i, const Nd4jLong j, const Nd4jLong k, const bfloat16 value);
    template void NDArray::p(const Nd4jLong i, const Nd4jLong j, const Nd4jLong k, const Nd4jLong value);
    template void NDArray::p(const Nd4jLong i, const Nd4jLong j, const Nd4jLong k, const int value);
    template void NDArray::p(const Nd4jLong i, const Nd4jLong j, const Nd4jLong k, const int8_t value);
    template void NDArray::p(const Nd4jLong i, const Nd4jLong j, const Nd4jLong k, const uint8_t value);
    template void NDArray::p(const Nd4jLong i, const Nd4jLong j, const Nd4jLong k, const int16_t value);
    template void NDArray::p(const Nd4jLong i, const Nd4jLong j, const Nd4jLong k, const bool value);

//////////////////////////////////////////////////////////////////////////
    template <typename T>
    void NDArray::p(const Nd4jLong i, const Nd4jLong j, const Nd4jLong k, const Nd4jLong l, const T value) {
        //(*this)(i,j,k) = value;
        if (rankOf() != 4 || i >= shapeOf()[0] || j >= shapeOf()[1] || k >= shapeOf()[2] || l >= shapeOf()[3])
            throw std::invalid_argument("NDArray::p(i,j,k,l, value): one of input indexes is out of array length or rank!=4 !");

        lazyAllocateBuffer();
        if(!isActualOnHostSide())
            syncToHost();
        
        void *p = reinterpret_cast<void *>(const_cast<T *>(&value));        
        Nd4jLong coords[4] = {i, j, k, l};                
        auto xOffset = shape::getOffset(0, shapeOf(), stridesOf(), coords, rankOf());
        BUILD_SINGLE_PARTIAL_SELECTOR(dataType(), templatedSet<, T>(this->_buffer, xOffset, p), LIBND4J_TYPES);
        tickWriteHost();
    }
    template void NDArray::p(const Nd4jLong i, const Nd4jLong j, const Nd4jLong k, const Nd4jLong l, const double value);
    template void NDArray::p(const Nd4jLong i, const Nd4jLong j, const Nd4jLong k, const Nd4jLong l, const float value);
    template void NDArray::p(const Nd4jLong i, const Nd4jLong j, const Nd4jLong k, const Nd4jLong l, const float16 value);
    template void NDArray::p(const Nd4jLong i, const Nd4jLong j, const Nd4jLong k, const Nd4jLong l, const bfloat16 value);
    template void NDArray::p(const Nd4jLong i, const Nd4jLong j, const Nd4jLong k, const Nd4jLong l, const Nd4jLong value);
    template void NDArray::p(const Nd4jLong i, const Nd4jLong j, const Nd4jLong k, const Nd4jLong l, const int value);
    template void NDArray::p(const Nd4jLong i, const Nd4jLong j, const Nd4jLong k, const Nd4jLong l, const int8_t value);
    template void NDArray::p(const Nd4jLong i, const Nd4jLong j, const Nd4jLong k, const Nd4jLong l, const uint8_t value);
    template void NDArray::p(const Nd4jLong i, const Nd4jLong j, const Nd4jLong k, const Nd4jLong l, const int16_t value);
    template void NDArray::p(const Nd4jLong i, const Nd4jLong j, const Nd4jLong k, const Nd4jLong l, const bool value);

//////////////////////////////////////////////////////////////////////////
    void* NDArray::specialBufferWithOffset(Nd4jLong offset) const {
        return _bufferD + (offset * sizeOfT());
    }

//////////////////////////////////////////////////////////////////////////    
    NDArray* NDArray::tensorAlongDimension(Nd4jLong index, const std::vector<int>& dimensions) const {
        std::vector<int> copy(dimensions);
        shape::checkDimensions(rankOf(), copy);

        Nd4jLong tadLength = shape::tadLength(this->_shapeInfo, copy.data(), copy.size());
        Nd4jLong numTads = this->lengthOf() / tadLength;

        if (index >= numTads)
            throw std::runtime_error("Can't get index higher than total number of TADs");

        shape::TAD tad(this->_shapeInfo, copy.data(), copy.size());
        tad.createTadOnlyShapeInfo();
        tad.createOffsets();

        // FIXME MISTAKE PRESENT
        auto array = new NDArray(tad.tadOnlyShapeInfo, true, _context, false);
        //hipFree(array->_bufferD);
        array->_bufferD = (int8_t*)specialBufferWithOffset(tad.tadOffsets[index]); //, array->lengthOf() * DataTypeUtils::sizeOf(dataType()), hipMemcpyDeviceToDevice);        
        array->_isBuffDAlloc = false;        
        array->_isView = true;

        return array;
    }

//////////////////////////////////////////////////////////////////////////
    void NDArray::addRowVector(const NDArray *row, NDArray *target) const {

        if (isS())
            throw std::runtime_error("NDArray::addRowVector: you can't use this method on String array!");
        if (rankOf() != 2 || target->rankOf() != 2 || rows() != target->rows() || columns() != target->columns() || !row->isRowVector() || columns() != row->lengthOf())
            throw std::invalid_argument("NDArray::addRowVector: wrong arguments !");
        if(target->_dataType !=  DataTypeUtils::pickPairwiseResultType(_dataType, row->_dataType) && !(isR() && row->isR() && target->isR()))
            throw std::invalid_argument("NDArray::addRowVector: wrong type of target array !");

        int dimension[1] = {1};

        std::unique_ptr<shape::TAD> tad(new shape::TAD(_shapeInfo, dimension, 1));
        tad->createTadOnlyShapeInfo();
        tad->createOffsets();
        std::vector<std::pair<void*,size_t>> hostData;
        hostData.emplace_back(dimension, sizeof(int));							// 0 -- dimensions
        hostData.emplace_back(tad->tadOnlyShapeInfo, shape::shapeInfoByteLength(tad->tadOnlyShapeInfo));	// 1 -- xTadShapeInfo
        hostData.emplace_back(tad->tadOffsets, tad->numTads * sizeof(Nd4jLong));							// 2 -- xTadOffsets
        std::vector<void*> devicePtrs(hostData.size(), nullptr);

        // create cuda stream and LaunchContext
        hipError_t cudaResult;
        //hipStream_t stream;
        //cudaResult = hipStreamCreate(&stream);	ASSERT_EQ(0, cudaResult);
        hipStream_t* stream = this->getContext()->getCudaStream();
        // allocate required amount of global device memory and copy host data to it
//    cudaResult = allocateDeviceMem(*pLc, devicePtrs, hostData);	ASSERT_EQ(0, cudaResult);
        for(int i = 0; i < devicePtrs.size(); ++i) {

            cudaResult = hipMalloc(reinterpret_cast<void **>(&devicePtrs[i]), hostData[i].second);
            if(cudaResult != 0) throw cuda_exception::build("Cannot allocate memory for tads on device", cudaResult);
            hipMemcpy(devicePtrs[i], hostData[i].first, hostData[i].second, hipMemcpyHostToDevice);
        }

        NDArray::registerSpecialUse({target}, {this, row});
        NativeOpExecutioner::execBroadcast(_context, nd4j::broadcast::Ops::Add, _buffer, _shapeInfo, _bufferD, _shapeInfoD, row->_buffer, row->_shapeInfo, row->_bufferD, row->_shapeInfoD, target->getBuffer(), target->getShapeInfo(), target->getSpecialBuffer(), target->getSpecialShapeInfo(), (int*)devicePtrs[0], 1, (Nd4jLong*)devicePtrs[1], (Nd4jLong*)devicePtrs[2], nullptr, nullptr);
    }

//////////////////////////////////////////////////////////////////////////
    void NDArray::subRowVector(const NDArray *row, NDArray * target) const {

        if (isS())
            throw std::runtime_error("NDArray::subRowVector: you can't use this method on String array!");
        if (rankOf() != 2 || target->rankOf() != 2 || rows() != target->rows() || columns() != target->columns() || !row->isRowVector() || columns() != row->columns())
            throw std::invalid_argument("NDArray::subRowVector: wrong arguments !");
        if(target->_dataType !=  DataTypeUtils::pickPairwiseResultType(_dataType, row->_dataType))
            throw std::invalid_argument("NDArray::subRowVector: wrong type of target array !");

        int dimension[1] = {1};

        std::unique_ptr<shape::TAD> tad(new shape::TAD(_shapeInfo, dimension, 1));
        tad->createTadOnlyShapeInfo();
        tad->createOffsets();

        std::vector<std::pair<void*,size_t>> hostData;
        hostData.emplace_back(dimension, sizeof(int));							// 0 -- dimensions
        hostData.emplace_back(tad->tadOnlyShapeInfo, shape::shapeInfoByteLength(tad->tadOnlyShapeInfo));	// 1 -- xTadShapeInfo
        hostData.emplace_back(tad->tadOffsets, tad->numTads * sizeof(Nd4jLong));							// 2 -- xTadOffsets
        std::vector<void*> devicePtrs(hostData.size(), nullptr);

        // create cuda stream and LaunchContext
        hipError_t cudaResult;
        //hipStream_t stream;
        //cudaResult = hipStreamCreate(&stream);	ASSERT_EQ(0, cudaResult);
        hipStream_t* stream = this->getContext()->getCudaStream();
        // allocate required amount of global device memory and copy host data to it
//    cudaResult = allocateDeviceMem(*pLc, devicePtrs, hostData);	ASSERT_EQ(0, cudaResult);
        for(int i = 0; i < devicePtrs.size(); ++i) {

            cudaResult = hipMalloc(reinterpret_cast<void **>(&devicePtrs[i]), hostData[i].second);
            if(cudaResult != 0) throw cuda_exception::build("Cannot allocate memory for tads on device", cudaResult);
            hipMemcpy(devicePtrs[i], hostData[i].first, hostData[i].second, hipMemcpyHostToDevice);
        }

        NDArray::registerSpecialUse({target}, {this, row});
        NativeOpExecutioner::execBroadcast(_context, nd4j::broadcast::Ops::Subtract, _buffer, _shapeInfo, _bufferD, _shapeInfoD, row->_buffer, row->_shapeInfo, row->_bufferD, row->_shapeInfoD, target->getBuffer(), target->getShapeInfo(), target->getSpecialBuffer(), target->getSpecialShapeInfo(), (int*)devicePtrs[0], 1, (Nd4jLong*)devicePtrs[1], (Nd4jLong*)devicePtrs[2], nullptr, nullptr);
    }

//////////////////////////////////////////////////////////////////////////
    void NDArray::mulRowVector(const NDArray *row, NDArray *target) const {

        if (isS())
            throw std::runtime_error("NDArray::mulRowVector: you can't use this method on String array!");
        if (rankOf() != 2 || target->rankOf() != 2 || rows() != target->rows() || columns() != target->columns() || !row->isRowVector() || columns() != row->columns())
            throw std::invalid_argument("NDArray::divRowVector: wrong arguments !");
        if(target->_dataType !=  DataTypeUtils::pickPairwiseResultType(_dataType, row->_dataType))
            throw std::invalid_argument("NDArray::mulRowVector: wrong type of target array !");

        int dimension[1] = {1};

        std::unique_ptr<shape::TAD> tad(new shape::TAD(_shapeInfo, dimension, 1));
        tad->createTadOnlyShapeInfo();
        tad->createOffsets();
        std::vector<std::pair<void*,size_t>> hostData;
        hostData.emplace_back(dimension, sizeof(int));							// 0 -- dimensions
        hostData.emplace_back(tad->tadOnlyShapeInfo, shape::shapeInfoByteLength(tad->tadOnlyShapeInfo));	// 1 -- xTadShapeInfo
        hostData.emplace_back(tad->tadOffsets, tad->numTads * sizeof(Nd4jLong));							// 2 -- xTadOffsets
        std::vector<void*> devicePtrs(hostData.size(), nullptr);

        // create cuda stream and LaunchContext
        hipError_t cudaResult;
        //hipStream_t stream;
        //cudaResult = hipStreamCreate(&stream);	ASSERT_EQ(0, cudaResult);
        hipStream_t* stream = this->getContext()->getCudaStream();
        // allocate required amount of global device memory and copy host data to it
//    cudaResult = allocateDeviceMem(*pLc, devicePtrs, hostData);	ASSERT_EQ(0, cudaResult);
        for(int i = 0; i < devicePtrs.size(); ++i) {

            cudaResult = hipMalloc(reinterpret_cast<void **>(&devicePtrs[i]), hostData[i].second);
            if(cudaResult != 0) throw cuda_exception::build("Cannot allocate memory for tads on device", cudaResult);
            hipMemcpy(devicePtrs[i], hostData[i].first, hostData[i].second, hipMemcpyHostToDevice);
        }

        NDArray::registerSpecialUse({target}, {this, row});
        NativeOpExecutioner::execBroadcast(_context, nd4j::broadcast::Ops::Multiply, _buffer, _shapeInfo, _bufferD, _shapeInfoD, row->_buffer, row->_shapeInfo, row->_bufferD, row->_shapeInfoD, target->getBuffer(), target->getShapeInfo(), target->getSpecialBuffer(), target->getSpecialShapeInfo(), (int*)devicePtrs[0], 1, (Nd4jLong*)devicePtrs[1], (Nd4jLong*)devicePtrs[2], nullptr, nullptr);
    }

//////////////////////////////////////////////////////////////////////////
    void NDArray::divRowVector(const NDArray *row, NDArray *target) const {

        if (isS())
            throw std::runtime_error("NDArray::divRowVector: you can't use this method on String array!");
        if (row->isB())
            throw std::runtime_error("NDArray::divRowVector: you can't divide by bool row!");
        if (rankOf() != 2 || target->rankOf() != 2 || rows() != target->rows() || columns() != target->columns() || !row->isRowVector() || columns() != row->columns())
            throw std::invalid_argument("NDArray::divRowVector: wrong arguments !");
        if(target->_dataType !=  DataTypeUtils::pickPairwiseResultType(_dataType, row->_dataType))
            throw std::invalid_argument("NDArray::divRowVector: wrong type of target array !");

        int dimension[1] = {1};

        std::unique_ptr<shape::TAD> tad(new shape::TAD(_shapeInfo, dimension, 1));
        tad->createTadOnlyShapeInfo();
        tad->createOffsets();
        std::vector<std::pair<void*,size_t>> hostData;
        hostData.emplace_back(dimension, sizeof(int));							// 0 -- dimensions
        hostData.emplace_back(tad->tadOnlyShapeInfo, shape::shapeInfoByteLength(tad->tadOnlyShapeInfo));	// 1 -- xTadShapeInfo
        hostData.emplace_back(tad->tadOffsets, tad->numTads * sizeof(Nd4jLong));							// 2 -- xTadOffsets
        std::vector<void*> devicePtrs(hostData.size(), nullptr);

        // create cuda stream and LaunchContext
        hipError_t cudaResult;
        //hipStream_t stream;
        //cudaResult = hipStreamCreate(&stream);	ASSERT_EQ(0, cudaResult);
        hipStream_t* stream = this->getContext()->getCudaStream();
        // allocate required amount of global device memory and copy host data to it
//    cudaResult = allocateDeviceMem(*pLc, devicePtrs, hostData);	ASSERT_EQ(0, cudaResult);
        for(int i = 0; i < devicePtrs.size(); ++i) {

            cudaResult = hipMalloc(reinterpret_cast<void **>(&devicePtrs[i]), hostData[i].second);
            if(cudaResult != 0) throw cuda_exception::build("Cannot allocate memory for tads on device", cudaResult);
            hipMemcpy(devicePtrs[i], hostData[i].first, hostData[i].second, hipMemcpyHostToDevice);
        }
        NDArray::registerSpecialUse({target}, {this, row});

        NativeOpExecutioner::execBroadcast(_context, nd4j::broadcast::Divide, _buffer, _shapeInfo, _bufferD, _shapeInfoD, row->_buffer, row->_shapeInfo, row->_bufferD, row->_shapeInfoD, target->getBuffer(), target->getShapeInfo(), target->getSpecialBuffer(), target->getSpecialShapeInfo(), (int*)devicePtrs[0], 1, (Nd4jLong*)devicePtrs[1], (Nd4jLong*)devicePtrs[2], nullptr, nullptr);

    }

//////////////////////////////////////////////////////////////////////////
// This method adds given row to all rows in this NDArray, this array becomes affected
    void NDArray::addiRowVector(const NDArray *row) {

        if (isS())
            throw std::runtime_error("NDArray::addiRowVector: you can't use this method on String array!");
        if (rankOf() != 2 || !row->isRowVector() || columns() != row->lengthOf())
            throw std::invalid_argument("NDArray::addiRowVector: wrong arguments !");

        int dimension[1] = {1};

        std::unique_ptr<shape::TAD> tad(new shape::TAD(_shapeInfo, dimension, 1));
        tad->createTadOnlyShapeInfo();
        tad->createOffsets();

        std::vector<std::pair<void*,size_t>> hostData;
        hostData.emplace_back(dimension, sizeof(int));							// 0 -- dimensions
        hostData.emplace_back(tad->tadOnlyShapeInfo, shape::shapeInfoByteLength(tad->tadOnlyShapeInfo));	// 1 -- xTadShapeInfo
        hostData.emplace_back(tad->tadOffsets, tad->numTads * sizeof(Nd4jLong));							// 2 -- xTadOffsets
        std::vector<void*> devicePtrs(hostData.size(), nullptr);

        // create cuda stream and LaunchContext
        hipError_t cudaResult;
        //hipStream_t stream;
        //cudaResult = hipStreamCreate(&stream);	ASSERT_EQ(0, cudaResult);
        hipStream_t* stream = this->getContext()->getCudaStream();
        // allocate required amount of global device memory and copy host data to it
//    cudaResult = allocateDeviceMem(*pLc, devicePtrs, hostData);	ASSERT_EQ(0, cudaResult);
        for(int i = 0; i < devicePtrs.size(); ++i) {

            cudaResult = hipMalloc(reinterpret_cast<void **>(&devicePtrs[i]), hostData[i].second);
            if(cudaResult != 0) throw cuda_exception::build("Cannot allocate memory for tads on device", cudaResult);
            hipMemcpy(devicePtrs[i], hostData[i].first, hostData[i].second, hipMemcpyHostToDevice);
        }
        NDArray::registerSpecialUse({this}, {row});
        NativeOpExecutioner::execBroadcast(_context, nd4j::broadcast::Ops::Add, _buffer, _shapeInfo, _bufferD, _shapeInfoD, row->_buffer, row->_shapeInfo, row->_bufferD, row->_shapeInfoD, this->buffer(), this->shapeInfo(), this->specialBuffer(), this->specialShapeInfo(), (int*)devicePtrs[0], 1, (Nd4jLong*)devicePtrs[1], (Nd4jLong*)devicePtrs[2], nullptr, nullptr);
    }

//////////////////////////////////////////////////////////////////////////
    void NDArray::addColumnVector(const NDArray *column, NDArray *target) const {
        if (isS())
            throw std::runtime_error("NDArray::addColumnVector: you can't use this method on String array!");
        if (rankOf() != 2 || target->rankOf() != 2 || rows() != target->rows() || columns() != target->columns() || !column->isColumnVector() || rows() != column->lengthOf())
            throw std::invalid_argument("NDArray::addColumnVector: wrong arguments !");
        if(target->_dataType !=  DataTypeUtils::pickPairwiseResultType(_dataType, column->_dataType))
            throw std::invalid_argument("NDArray::addColumnVector: wrong type of target array !");

        int dimension[1] = {0};

        std::unique_ptr<shape::TAD> tad(new shape::TAD(_shapeInfo, dimension, 1));
        tad->createTadOnlyShapeInfo();
        tad->createOffsets();
        std::vector<std::pair<void*,size_t>> hostData;
        hostData.emplace_back(dimension, sizeof(int));							// 0 -- dimensions
        hostData.emplace_back(tad->tadOnlyShapeInfo, shape::shapeInfoByteLength(tad->tadOnlyShapeInfo));	// 1 -- xTadShapeInfo
        hostData.emplace_back(tad->tadOffsets, tad->numTads * sizeof(Nd4jLong));							// 2 -- xTadOffsets
        std::vector<void*> devicePtrs(hostData.size(), nullptr);

        // create cuda stream and LaunchContext
        hipError_t cudaResult;
        //hipStream_t stream;
        //cudaResult = hipStreamCreate(&stream);	ASSERT_EQ(0, cudaResult);
        hipStream_t* stream = this->getContext()->getCudaStream();
        // allocate required amount of global device memory and copy host data to it
//    cudaResult = allocateDeviceMem(*pLc, devicePtrs, hostData);	ASSERT_EQ(0, cudaResult);
        for(int i = 0; i < devicePtrs.size(); ++i) {

            cudaResult = hipMalloc(reinterpret_cast<void **>(&devicePtrs[i]), hostData[i].second);
            if(cudaResult != 0) throw cuda_exception::build("Cannot allocate memory for tads on device", cudaResult);
            hipMemcpy(devicePtrs[i], hostData[i].first, hostData[i].second, hipMemcpyHostToDevice);
        }

        NDArray::registerSpecialUse({target}, {this, column});
        NativeOpExecutioner::execBroadcast(_context, nd4j::broadcast::Ops::Add, _buffer, _shapeInfo, _bufferD, _shapeInfoD, column->_buffer, column->_shapeInfo, column->_bufferD, column->_shapeInfoD, target->getBuffer(), target->getShapeInfo(), target->getSpecialBuffer(), target->getSpecialShapeInfo(), (int*)devicePtrs[0], 1, (Nd4jLong*)devicePtrs[1], (Nd4jLong*)devicePtrs[2], nullptr, nullptr);
    }

//////////////////////////////////////////////////////////////////////////
// This method adds given column to all columns in this NDArray, this array becomes affected
    void NDArray::addiColumnVector(const NDArray *column) {
        if (isS())
            throw std::runtime_error("NDArray::addiColumnVector: you can't use this method on String array!");
        if (rankOf() != 2 || !column->isColumnVector() || rows() != column->lengthOf())
            throw std::invalid_argument("NDArray::addiColumnVector: wrong arguments !");

        int dimension[1] = {0};

        std::unique_ptr<shape::TAD> tad(new shape::TAD(_shapeInfo, dimension, 1));
        tad->createTadOnlyShapeInfo();
        tad->createOffsets();
        std::vector<std::pair<void*,size_t>> hostData;
        hostData.emplace_back(dimension, sizeof(int));							// 0 -- dimensions
        hostData.emplace_back(tad->tadOnlyShapeInfo, shape::shapeInfoByteLength(tad->tadOnlyShapeInfo));	// 1 -- xTadShapeInfo
        hostData.emplace_back(tad->tadOffsets, tad->numTads * sizeof(Nd4jLong));							// 2 -- xTadOffsets
        std::vector<void*> devicePtrs(hostData.size(), nullptr);

        // create cuda stream and LaunchContext
        hipError_t cudaResult;
        //hipStream_t stream;
        //cudaResult = hipStreamCreate(&stream);	ASSERT_EQ(0, cudaResult);
        hipStream_t* stream = this->getContext()->getCudaStream();
        // allocate required amount of global device memory and copy host data to it
//    cudaResult = allocateDeviceMem(*pLc, devicePtrs, hostData);	ASSERT_EQ(0, cudaResult);
        for(int i = 0; i < devicePtrs.size(); ++i) {

            cudaResult = hipMalloc(reinterpret_cast<void **>(&devicePtrs[i]), hostData[i].second);
            if(cudaResult != 0) throw cuda_exception::build("Cannot allocate memory for tads on device", cudaResult);
            hipMemcpy(devicePtrs[i], hostData[i].first, hostData[i].second, hipMemcpyHostToDevice);
        }

        NDArray::registerSpecialUse({this}, {column});
        NativeOpExecutioner::execBroadcast(_context, nd4j::broadcast::Ops::Add, _buffer, _shapeInfo, _bufferD, _shapeInfoD, column->_buffer, column->_shapeInfo, column->_bufferD, column->_shapeInfoD, this->buffer(), this->shapeInfo(), this->specialBuffer(), this->specialShapeInfo(), (int*)devicePtrs[0], 1, (Nd4jLong*)devicePtrs[1], (Nd4jLong*)devicePtrs[2], nullptr, nullptr);
    }

//////////////////////////////////////////////////////////////////////////
// This method multiplies each column of this array by given argument-column, this array becomes affected
    void NDArray::muliColumnVector(const NDArray *column) {
        if (isS())
            throw std::runtime_error("NDArray::muliColumnVector: you can't use this method on String array!");
        if (rankOf() != 2 || !column->isColumnVector() || rows() != column->lengthOf())
            throw std::invalid_argument("NDArray::muliColumnVector: wrong arguments !");

        int dimension[1] = {0};

        std::unique_ptr<shape::TAD> tad(new shape::TAD(_shapeInfo, dimension, 1));
        tad->createTadOnlyShapeInfo();
        tad->createOffsets();
        std::vector<std::pair<void*,size_t>> hostData;
        hostData.emplace_back(dimension, sizeof(int));							// 0 -- dimensions
        hostData.emplace_back(tad->tadOnlyShapeInfo, shape::shapeInfoByteLength(tad->tadOnlyShapeInfo));	// 1 -- xTadShapeInfo
        hostData.emplace_back(tad->tadOffsets, tad->numTads * sizeof(Nd4jLong));							// 2 -- xTadOffsets
        std::vector<void*> devicePtrs(hostData.size(), nullptr);

        // create cuda stream and LaunchContext
        hipError_t cudaResult;
        //hipStream_t stream;
        //cudaResult = hipStreamCreate(&stream);	ASSERT_EQ(0, cudaResult);
        hipStream_t* stream = this->getContext()->getCudaStream();
        // allocate required amount of global device memory and copy host data to it
//    cudaResult = allocateDeviceMem(*pLc, devicePtrs, hostData);	ASSERT_EQ(0, cudaResult);
        for(int i = 0; i < devicePtrs.size(); ++i) {

            cudaResult = hipMalloc(reinterpret_cast<void **>(&devicePtrs[i]), hostData[i].second);
            if(cudaResult != 0) throw cuda_exception::build("Cannot allocate memory for tads on device", cudaResult);
            hipMemcpy(devicePtrs[i], hostData[i].first, hostData[i].second, hipMemcpyHostToDevice);
        }
        NDArray::registerSpecialUse({this}, {column});
        NativeOpExecutioner::execBroadcast(_context, nd4j::broadcast::Ops::Multiply, _buffer, _shapeInfo, _bufferD, _shapeInfoD, column->_buffer, column->_shapeInfo, column->_bufferD, column->_shapeInfoD, this->buffer(), this->shapeInfo(), this->specialBuffer(), this->specialShapeInfo(), (int*)devicePtrs[0], 1, (Nd4jLong*)devicePtrs[1], (Nd4jLong*)devicePtrs[2], nullptr, nullptr);
    }
    

    ////////////////////////////////////////////////////////////////////////////////////////////////////////////////////
    // change an array by repeating it the number of times given by reps.
    NDArray NDArray::tile(const std::vector<Nd4jLong>& reps) const {
        int dim = reps.size();
        int product = 1;
        for(const auto& item : reps)
            product *= item;
        if(product == 0)
            throw std::runtime_error("NDArray::tile method: one of the elements in reps array is zero !");

        if (!isActualOnDeviceSide())
            syncToDevice();

        int rankOld = rankOf();
        int diff = rankOld - dim;
        if(product==1) {        // in this case 2 possibilities are present: just reshape or nothing to do
            NDArray result(*this);
            if(diff < 0) {      // reshape to higher dimension
                std::vector<Nd4jLong> shapeNew = reps;               // need to have unities at first "diff" positions of new shape
                memcpy(&shapeNew[-diff], result._shapeInfo+1, rankOld * sizeof(Nd4jLong));   // put old shape numbers at rest of positions
                result.reshapei(ordering(), shapeNew);
            }
            return result;             // nothing to do, if diff >= 0 -> identity tile
        }

        // evaluate shapeInfo for resulting array
        auto newShapeInfo = ShapeUtils::evalTileShapeInfo(*this, reps, _context->getWorkspace());
        // create new buffer, in any case the memory amount new buffer points to is bigger then those for old _buffer
        int8_t * newBuff = nullptr;
        ALLOCATE(newBuff, _context->getWorkspace(), shape::length(newShapeInfo) * sizeOfT(), int8_t);
        // assign new shape and new buffer to resulting array
        NDArray result(newBuff, newShapeInfo, _context, true, true);
//        if (!isActualOnHostSide())
//            syncToHost();
        // fill newBuff, loop through all elements of newBuff
        // looping through _buffer goes automatically by means of getSubArrayIndex applying
        const auto resultLen = result.lengthOf();
        auto xType = this->dataType();
        auto stream = _context->getCudaStream();
        BUILD_SINGLE_SELECTOR(xType, tileKernelH, (this->_bufferD, this->_shapeInfoD, result._bufferD, result._shapeInfoD, resultLen, *stream), LIBND4J_TYPES);
        result.tickWriteDevice();
        return result;
    }
//                *(reinterpret_cast<double*>(newBuff) + i) = *(reinterpret_cast<double*>(_buffer) + yOffset);
    template <typename T>
    void NDArray::templatedAssign(void *xBuffer, Nd4jLong xOffset, const void *yBuffer, const Nd4jLong yOffset) const {
        if (xBuffer != nullptr && yBuffer != nullptr)
            *(reinterpret_cast<T*>(xBuffer) + xOffset) = *(reinterpret_cast<T const*>(yBuffer) + yOffset);
    }
    BUILD_SINGLE_TEMPLATE(template void NDArray::templatedAssign, (void *xBuffer, const Nd4jLong xOffset, const void *yBuffer, const Nd4jLong yOffset) const, LIBND4J_TYPES);


    //////////////////////////////////////////////////////////////////////////
    // change an array by repeating it the number of times given by reps.
    void NDArray::tile(const std::vector<Nd4jLong>& reps, NDArray& target) const {

        // evaluate true tile shapeInfo for comparison with target shapeInfo
        auto newShapeInfo = ShapeUtils::evalTileShapeInfo(*this, reps, _context->getWorkspace());
        if(!shape::equalsSoft(newShapeInfo, target.getShapeInfo()))  {
            delete []newShapeInfo;
            throw std::runtime_error("NDArray::tile method - shapeInfo of target array is not suitable for tile operation !");
        }
        RELEASE(newShapeInfo, _context->getWorkspace());

        // fill newBuff, loop through all elements of newBuff
        // looping through _buffer goes automatically by means of getSubArrayIndex applying
        const int ews = target.ews();
        const int targetLen = target.lengthOf();
        auto stream = _context->getCudaStream();
        BUILD_DOUBLE_SELECTOR(target.dataType(), dataType(), tileKernelHH, (_bufferD, _shapeInfoD, target._bufferD, target._shapeInfoD, targetLen, ews, *stream), LIBND4J_TYPES, LIBND4J_TYPES);
    }

    //////////////////////////////////////////////////////////////////////////
    void NDArray::tile(NDArray& target) const {
        if(rankOf() > target.rankOf())
            throw std::runtime_error("NDArray::tile method - rank of target array must be bigger or equal to the rank of this array !");

        if(!ShapeUtils::areShapesBroadcastable(*this, target))
            throw std::runtime_error("NDArray::tile method - shapeInfo of target array is not suitable for tile operation !");

        // fill newBuff, loop through all elements of newBuff
        // looping through _buffer goes automatically by means of getSubArrayIndex applying
        const auto ews = target.ews();
        const auto targetLen = target.lengthOf();
        auto stream = _context->getCudaStream();
        BUILD_DOUBLE_SELECTOR(target.dataType(), dataType(), tileKernelHH, (_bufferD, _shapeInfoD, target._bufferD, target._shapeInfoD, targetLen, ews, *stream), LIBND4J_TYPES, LIBND4J_TYPES);
    }

    ////////////////////////////////////////////////////////////////////////////////////////////////////////////////////
    // create new  array by repeating it the number of times given by reps
    ////////////////////////////////////////////////////////////////////////////////////////////////////////////////////
    NDArray* NDArray::repeat(int dimension, const std::vector<Nd4jLong>& repeats) const {
        auto outShape = ShapeUtils::evalRepeatShape(dimension, repeats, *this);

        // the size of outShape == rank
        int rank = rankOf();            // = outShape.size()

        std::vector<Nd4jLong> newShape(rank);
        for (int i = 0; i < rank; i++)
            newShape[i] = outShape[i];

        auto ret = new NDArray('c', outShape, _dataType,  _context);

        auto repeatDelta = shape::prodLong(newShape.data(), rank) / this->lengthOf();
        std::vector<int> dimsToExclude = ShapeUtils::evalDimsToExclude(rankOf(), {dimension});
        const Nd4jLong numTads = ShapeUtils::getNumOfSubArrs(_shapeInfo, dimsToExclude); //this->tensorsAlongDimension({dimension});
        //printf("Repeat delta %lld, numTads %lld\n", repeatDelta, numTads);
        //tadOnlyInputShapeInfo, tadInputOffsets, tadOnlyOutputShapeInfo, tadOutputOffsets;
        std::vector<int> copy({dimension});
        shape::TAD tadInput(this->_shapeInfo, copy.data(), copy.size());
        tadInput.createTadOnlyShapeInfo();
        tadInput.createOffsets();
        if (!this->isActualOnDeviceSide())
            this->syncToDevice();

        shape::TAD tadOutput(ret->_shapeInfo, copy.data(), copy.size());
        tadOutput.createTadOnlyShapeInfo();
        tadOutput.createOffsets();
        if (!this->isActualOnDeviceSide())
            this->syncToDevice();

        // prepare input arrays for prepareDataForCuda function
        std::vector<std::pair<void*,size_t>> hostData;
        hostData.emplace_back(tadInput.tadOnlyShapeInfo, shape::shapeInfoByteLength(tadInput.tadOnlyShapeInfo));	// 1 -- xTadShapeInfo
        hostData.emplace_back(tadInput.tadOffsets, tadInput.numTads * sizeof(Nd4jLong));							// 2 -- xTadOffsets
        hostData.emplace_back(tadOutput.tadOnlyShapeInfo, shape::shapeInfoByteLength(tadOutput.tadOnlyShapeInfo));	// 1 -- xTadShapeInfo
        hostData.emplace_back(tadOutput.tadOffsets, tadOutput.numTads * sizeof(Nd4jLong));							// 2 -- xTadOffsets
        std::vector<void*> devicePtrs(hostData.size(), nullptr);

        // create cuda stream and LaunchContext
        hipError_t cudaResult;
        //hipStream_t stream;
        //cudaResult = hipStreamCreate(&stream);	ASSERT_EQ(0, cudaResult);
        //hipStream_t* stream = this->getContext()->getCudaStream();
        // allocate required amount of global device memory and copy host data to it
//    cudaResult = allocateDeviceMem(*pLc, devicePtrs, hostData);	ASSERT_EQ(0, cudaResult);
        for(int i = 0; i < devicePtrs.size(); ++i) {
            cudaResult = hipMalloc(reinterpret_cast<void **>(&devicePtrs[i]), hostData[i].second);
            if(cudaResult != 0) throw cuda_exception::build("Cannot allocate memory for tads on device", cudaResult);
            cudaResult = hipMemcpy(devicePtrs[i], hostData[i].first, hostData[i].second, hipMemcpyHostToDevice);
            if(cudaResult != 0) throw cuda_exception::build("Cannot copy memory block for tads on device", cudaResult);
        }
        auto stream = _context->getCudaStream();
        BUILD_SINGLE_SELECTOR(_dataType, repeatKernelH, (_bufferD, ret->_bufferD, numTads, lengthOf(), (Nd4jLong*)devicePtrs[0], (Nd4jLong*)devicePtrs[1], (Nd4jLong*)devicePtrs[2], (Nd4jLong*)devicePtrs[3], *stream), LIBND4J_TYPES);

        for(int i = 0; i < devicePtrs.size(); ++i) {
            cudaResult = hipFree(devicePtrs[i]);
            if(cudaResult != 0) throw cuda_exception::build("Cannot allocate memory for tads on device", cudaResult);
//            cudaResult = hipMemcpy(devicePtrs[i], hostData[i].first, hostData[i].second, hipMemcpyHostToDevice);
//            if(cudaResult != 0) throw cuda_exception::build("Cannot copy memory block for tads on device", cudaResult);
        }

        return ret;
    }

    //////////////////////////////////////////////////////////////////////////
    // fill array by repeating it the number of times given by reps
    void NDArray::repeat(int dimension, NDArray& target) const {

        if(dimension < 0)
            dimension += rankOf();

        if(rankOf() != target.rankOf())
            throw std::invalid_argument("NDArray::repeat(int dimension, NDArray& target) method: wrong rank of target array it must be equal to this array rank!");

        Nd4jLong repeatDelta = target.sizeAt(dimension) / sizeAt(dimension);

        if(repeatDelta == 0)
            throw std::invalid_argument("NDArray::repeat(int dimension, NDArray& target) method: wrong shape of target array!");


        std::vector<int> dimsToExclude = ShapeUtils::evalDimsToExclude(rankOf(), {dimension});
        const Nd4jLong numTads = ShapeUtils::getNumOfSubArrs(_shapeInfo, dimsToExclude);

        std::vector<int> copy({dimension});
        shape::TAD tadInput(this->_shapeInfo, copy.data(), copy.size());
        tadInput.createTadOnlyShapeInfo();
        tadInput.createOffsets();
        if (!this->isActualOnDeviceSide())
            this->syncToDevice();

        shape::TAD tadOutput(target._shapeInfo, copy.data(), copy.size());
        tadOutput.createTadOnlyShapeInfo();
        tadOutput.createOffsets();
        if (!this->isActualOnDeviceSide())
            this->syncToDevice();
        if (!target.isActualOnDeviceSide())
            target.syncToDevice();
        // prepare input arrays for prepareDataForCuda function
        std::vector<std::pair<void*,size_t>> hostData;
        hostData.emplace_back(tadInput.tadOnlyShapeInfo, shape::shapeInfoByteLength(tadInput.tadOnlyShapeInfo));	// 1 -- xTadShapeInfo
        hostData.emplace_back(tadInput.tadOffsets, tadInput.numTads * sizeof(Nd4jLong));							// 2 -- xTadOffsets
        hostData.emplace_back(tadOutput.tadOnlyShapeInfo, shape::shapeInfoByteLength(tadOutput.tadOnlyShapeInfo));	// 1 -- xTadShapeInfo
        hostData.emplace_back(tadOutput.tadOffsets, tadOutput.numTads * sizeof(Nd4jLong));							// 2 -- xTadOffsets
        std::vector<void*> devicePtrs(hostData.size(), nullptr);

        // create cuda stream and LaunchContext
        hipError_t cudaResult;
        //hipStream_t stream;
        //cudaResult = hipStreamCreate(&stream);	ASSERT_EQ(0, cudaResult);
        //hipStream_t* stream = this->getContext()->getCudaStream();
        // allocate required amount of global device memory and copy host data to it
//    cudaResult = allocateDeviceMem(*pLc, devicePtrs, hostData);	ASSERT_EQ(0, cudaResult);
        for(int i = 0; i < devicePtrs.size(); ++i) {
            cudaResult = hipMalloc(reinterpret_cast<void **>(&devicePtrs[i]), hostData[i].second);
            if(cudaResult != 0) throw cuda_exception::build("Cannot allocate memory for tads on device", cudaResult);
            cudaResult = hipMemcpy(devicePtrs[i], hostData[i].first, hostData[i].second, hipMemcpyHostToDevice);
            if(cudaResult != 0) throw cuda_exception::build("Cannot copy memory block for tads on device", cudaResult);
        }
        auto stream = _context->getCudaStream();
        //BUILD_SINGLE_SELECTOR(_dataType, repeatKernelH, (_bufferD, target._bufferD, numTads, lengthOf(), (Nd4jLong*)devicePtrs[0], (Nd4jLong*)devicePtrs[1], (Nd4jLong*)devicePtrs[2], (Nd4jLong*)devicePtrs[3], *stream), LIBND4J_TYPES);
        BUILD_DOUBLE_SELECTOR(target._dataType, _dataType, repeatKernelHH, (_bufferD, target._bufferD, numTads, lengthOf(), (Nd4jLong*)devicePtrs[0], (Nd4jLong*)devicePtrs[1], (Nd4jLong*)devicePtrs[2], (Nd4jLong*)devicePtrs[3], *stream), LIBND4J_TYPES, LIBND4J_TYPES);
        for(int i = 0; i < devicePtrs.size(); ++i) {
            cudaResult = hipFree(devicePtrs[i]);
            if(cudaResult != 0) throw cuda_exception::build("Cannot allocate memory for tads on device", cudaResult);
        }
    }

 


} // end namespace nd4j



#endif

