#include "hip/hip_runtime.h"
/*******************************************************************************
 * Copyright (c) 2015-2018 Skymind, Inc.
 *
 * This program and the accompanying materials are made available under the
 * terms of the Apache License, Version 2.0 which is available at
 * https://www.apache.org/licenses/LICENSE-2.0.
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS, WITHOUT
 * WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied. See the
 * License for the specific language governing permissions and limitations
 * under the License.
 *
 * SPDX-License-Identifier: Apache-2.0
 ******************************************************************************/

//
// @author raver119@gmail.com
// @author Yurii Shyrma, created on 15.11.2018
//

#include <loops/special_kernels.h>


template <typename T>
__device__ void averagingKernelGeneric(void **vdx, void *vdz, int n, Nd4jLong length, bool propagate) {

	auto dx = reinterpret_cast<T**>(vdx);
	auto dz = reinterpret_cast<T*>(vdz);

    __shared__ T *shmem;

    if (threadIdx.x == 0) {
        extern __shared__ unsigned char sharedmem[];
        shmem = (T *) sharedmem;
    }
    __syncthreads();


    // each block cycles over it's own part of arrays
    for (int r = blockDim.x * blockIdx.x; r < length; r += blockDim.x * gridDim.x) {
        shmem[threadIdx.x] = (T) 0.0f;

        Nd4jLong baseIdx = r;

        // aggregation step, we roll over all arrays
        for (int ar = 0; ar < n; ar++) {
            T *cdata = (T *) dx[ar];
            cdata += baseIdx;

            if (baseIdx + threadIdx.x < length)
                shmem[threadIdx.x] += cdata[threadIdx.x];
        }


        // average data in shared memory
        if (baseIdx + threadIdx.x < length)
            shmem[threadIdx.x] /= n;

        // div step & write out step
        if (dz != nullptr) {
            T *wdata = dz + baseIdx;

            if (baseIdx + threadIdx.x < length) {
                wdata[threadIdx.x] = shmem[threadIdx.x];
            }
        }

        // propagate averaged data to all arrays
        if (propagate)
            for (int ar = 0; ar < n; ar++) {
                T *cdata = (T *) dx[ar];
                cdata += baseIdx;

                if (baseIdx + threadIdx.x < length)
                    cdata[threadIdx.x] = shmem[threadIdx.x];
            }
    }
}
