#include "hip/hip_runtime.h"
/*******************************************************************************
 * Copyright (c) 2015-2018 Skymind, Inc.
 *
 * This program and the accompanying materials are made available under the
 * terms of the Apache License, Version 2.0 which is available at
 * https://www.apache.org/licenses/LICENSE-2.0.
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS, WITHOUT
 * WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied. See the
 * License for the specific language governing permissions and limitations
 * under the License.
 *
 * SPDX-License-Identifier: Apache-2.0
 ******************************************************************************/

//
// @author raver119@gmail.com
// @author Yurii Shyrma, created on 15.11.2018
//

#include <loops/special_kernels.h>

template <typename T>
__device__ void fillDimensionalIsMaxGeneric(T *dX, Nd4jLong *xShapeInfo, 
                                            T *dZ, Nd4jLong *zShapeInfo, 
                                            Nd4jLong *tadOnlyShapeInfo, 
                                            int *dimension, int dimensionLength, 
                                            Nd4jLong *tadOffsets) {

    __shared__ int tadLength;
    __shared__ int tadEWS;
    __shared__ int numTads;

    if (threadIdx.x == 0) {
        tadLength = shape::tadLength(zShapeInfo, dimension, dimensionLength);
        tadEWS = shape::elementWiseStride(tadOnlyShapeInfo);
        numTads = shape::length(zShapeInfo) / tadLength;
    }
    __syncthreads();

    for (int r = blockIdx.x; r < numTads; r+= gridDim.x) {
        auto tadOffsetForBlock = tadOffsets[r];

        int highestElement = (int) dX[r];

        if (dimensionLength > 1 || tadEWS < 1) {

            for (int e = threadIdx.x; e < tadLength; e += blockDim.x) {
                
            	auto xOffset = tadOffsetForBlock + shape::getIndexOffset(e, tadOnlyShapeInfo, tadLength);
                dZ[xOffset] = (e == highestElement? (T) 1.0f : (T) 0.0f);
            }
        } else {
            for (int e = threadIdx.x; e < tadLength; e += blockDim.x) {
                // so, we just set dZ[e] for each TAD. Sure, e should be replaced with
                auto idx = tadOffsetForBlock + (e * tadEWS);
                dZ[idx] = (e == highestElement? (T) 1.0f : (T) 0.0f);
            }
        }

    }
}
