#include "hip/hip_runtime.h"
/*******************************************************************************
 * Copyright (c) 2015-2018 Skymind, Inc.
 *
 * This program and the accompanying materials are made available under the
 * terms of the Apache License, Version 2.0 which is available at
 * https://www.apache.org/licenses/LICENSE-2.0.
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS, WITHOUT
 * WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied. See the
 * License for the specific language governing permissions and limitations
 * under the License.
 *
 * SPDX-License-Identifier: Apache-2.0
 ******************************************************************************/

//
// @author raver119@gmail.com
// @author Yurii Shyrma, created on 15.11.2018
//

#include <loops/special_kernels.h>

////////////////////////////////////////////////////////////////////////
__device__ void fillDimensionalIsMax(void *vdX, Nd4jLong *xShapeInfo, 
                                    bool *dZ, Nd4jLong *zShapeInfo, 
                                    Nd4jLong *tadOnlyShapeInfo, 
                                    int *dimension, int dimensionLength, 
                                    Nd4jLong *tadOffsets) {

    auto dX = reinterpret_cast<int*>(vdX);

    __shared__ int tadLength;
    __shared__ int tadEWS;
    __shared__ int numTads;

    if (threadIdx.x == 0) {
        tadLength = shape::tadLength(zShapeInfo, dimension, dimensionLength);
        tadEWS = shape::elementWiseStride(tadOnlyShapeInfo);
        numTads = shape::length(zShapeInfo) / tadLength;
    }
    __syncthreads();

    for (int r = blockIdx.x; r < numTads; r+= gridDim.x) {
        auto tadOffsetForBlock = tadOffsets[r];

        int highestElement = (int) dX[r];

        if (dimensionLength > 1 || tadEWS < 1) {

            for (int e = threadIdx.x; e < tadLength; e += blockDim.x) {
                
            	auto xOffset = tadOffsetForBlock + shape::getIndexOffset(e, tadOnlyShapeInfo, tadLength);
                dZ[xOffset] = (e == highestElement? true : false);
            }
        } 
        else {
            for (int e = threadIdx.x; e < tadLength; e += blockDim.x) {
                // so, we just set dZ[e] for each TAD. Sure, e should be replaced with
                auto idx = tadOffsetForBlock + (e * tadEWS);
                dZ[idx] = (e == highestElement? true : false);
            }
        }
    }
}


////////////////////////////////////////////////////////////////////////
__global__ void execfillDimensionalIsMax(void *dX, Nd4jLong *xShapeInfo, 
                                    bool *dZ, Nd4jLong *zShapeInfo, 
                                    Nd4jLong *tadOnlyShapeInfo, 
                                    int *dimension, int dimensionLength, 
                                    Nd4jLong *tadOffsets) {

    fillDimensionalIsMax(dX, xShapeInfo, dZ, zShapeInfo, tadOnlyShapeInfo, dimension, dimensionLength, tadOffsets);
}

////////////////////////////////////////////////////////////////////////
__host__ void fillDimensionalIsMaxGeneric(dim3& launchDims, hipStream_t *stream,
                                    void *dX, Nd4jLong *xShapeInfo, 
                                    bool *dZ, Nd4jLong *zShapeInfo, 
                                    Nd4jLong *tadOnlyShapeInfo, 
                                    int *dimension, int dimensionLength, 
                                    Nd4jLong *tadOffsets) {
    
    execfillDimensionalIsMax<<<launchDims.x, launchDims.y, launchDims.z, *stream>>>(dX, xShapeInfo, dZ, zShapeInfo, tadOnlyShapeInfo, dimension, dimensionLength, tadOffsets);
}