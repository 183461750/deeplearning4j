#include "hip/hip_runtime.h"
/*******************************************************************************
 * Copyright (c) 2015-2018 Skymind, Inc.
 *
 * This program and the accompanying materials are made available under the
 * terms of the Apache License, Version 2.0 which is available at
 * https://www.apache.org/licenses/LICENSE-2.0.
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS, WITHOUT
 * WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied. See the
 * License for the specific language governing permissions and limitations
 * under the License.
 *
 * SPDX-License-Identifier: Apache-2.0
 ******************************************************************************/

//
// @author raver119@gmail.com
// @author Yurii Shyrma, created on 15.11.2018
//

#include <loops/special_kernels.h>


///////////////////////////////////////////////////////////////////////
template <typename T>
__device__ void concatKernelVStack(int dimension,
									int numArrays,
									Nd4jPointer *data, Nd4jPointer *inputShapeInfos,
									void *vz, Nd4jLong *zShapeInfo, 
									Nd4jPointer *tadPointers, Nd4jPointer *offsetPointers) {

    /*
     this is special case for concat: we group bunch of vectors into 2D matrix
     also: we expect each inputShapeInfo to have EWS, be a vector, and have equal size
     */
	auto z = static_cast<T*>(vz);

    auto inputShapes = (Nd4jLong**) inputShapeInfos;
	T **input = (T **) data;

    __shared__ int inputEWS;
    __shared__ int resultEWS;
    __shared__ int inputLength;

    if (threadIdx.x == 0) {
    	inputLength = shape::length(inputShapes[0]);
        inputEWS = shape::elementWiseStride(inputShapes[0]);
        resultEWS = shape::elementWiseStride(zShapeInfo);
    }
    __syncthreads();

    for (int r = blockIdx.x; r < numArrays; r+= gridDim.x) {

        int zOffset = r * inputLength * resultEWS;
        T *inputData = (T *) input[r];

        for(int i = threadIdx.x; i < inputLength; i += blockDim.x) {
            z[zOffset + i * resultEWS] = inputData[i * inputEWS];
        }
    }
}

///////////////////////////////////////////////////////////////////////
template <typename T>
__global__ void execConcatKernelVStack(int dimension,
                                    int numArrays,
                                    Nd4jPointer *data, Nd4jPointer *inputShapeInfos,
                                    void *vz, Nd4jLong *zShapeInfo, 
                                    Nd4jPointer *tadPointers, Nd4jPointer *offsetPointers) {
    
    concatKernelVStack<T>(dimension, numArrays, *data, inputShapeInfos, vz, zShapeInfo, tadPointers, offsetPointers);
}


///////////////////////////////////////////////////////////////////////
template <typename T>
__host__ void concatKernelVStackGeneric(dim3& launchDims, Nd4jPointer* extraPointers,
                                    int dimension,
                                    int numArrays,
                                    Nd4jPointer *data, Nd4jPointer *inputShapeInfos,
                                    void *vz, Nd4jLong *zShapeInfo, 
                                    Nd4jPointer *tadPointers, Nd4jPointer *offsetPointers) {

    hipStream_t *stream = reinterpret_cast<hipStream_t *>(&extraPointers[1]);
    
    execConcatKernelVStack<T><<<launchDims.x, launchDims.y, launchDims.z, stream>>>(dimension, numArrays, *data, inputShapeInfos, vz, zShapeInfo, tadPointers, offsetPointers);
}