#include "hip/hip_runtime.h"
/*******************************************************************************
 * Copyright (c) 2015-2018 Skymind, Inc.
 *
 * This program and the accompanying materials are made available under the
 * terms of the Apache License, Version 2.0 which is available at
 * https://www.apache.org/licenses/LICENSE-2.0.
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS, WITHOUT
 * WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied. See the
 * License for the specific language governing permissions and limitations
 * under the License.
 *
 * SPDX-License-Identifier: Apache-2.0
 ******************************************************************************/

//
//  @author raver119@gmail.com
//  @author Yurii Shyrma (iuriish@yahoo.com)
//

#include <op_boilerplate.h>
#include <loops/reduce_float.h>
#include <loops/legacy_ops.h>
#include <helpers/DebugHelper.h>
#include <types/types.h>

using namespace simdOps;

template <typename X, typename Z, typename OpType>
__device__ void reduceSimpleGeneric(
        void *dx,
        Nd4jLong *xShapeInfo,
        void *extraParams,
        void *result,
        Nd4jLong *resultShapeInfo,
        int *dimension,
        int dimensionLength,
        void *reductionBuffer, Nd4jLong *tadOnlyShapeInfo, Nd4jLong *tadOffsets) {

    __shared__ UnifiedSharedMemory *manager;

    if (threadIdx.x == 0) {
        extern __shared__ unsigned char shmem[];
        manager = new(shmem) UnifiedSharedMemory((int *) shmem);
        manager->init(sizeof(UnifiedSharedMemory), 0, sizeof(functions::reduce::ReduceFloatFunction<X,Z>), sizeof(shape::TAD), shape::rank(xShapeInfo));
    }
    __syncthreads();


    functions::reduce::ReduceFloatFunction<X, Z>::template transformCudaXD<OpType>(
            dx,
            xShapeInfo,
            extraParams,
            result,
            resultShapeInfo,
            dimension,
            dimensionLength,
            reductionBuffer,
            manager,
            tadOnlyShapeInfo,
            tadOffsets);
}

template <typename X, typename Z, typename OpType>
__device__ void reduceScalarGeneric(
        void *dx,
        Nd4jLong *xShapeInfo,
        void *extraParams,
        void *result,
        Nd4jLong *resultShapeInfo,
        int *dimension,
        int dimensionLength,
        void *reductionBuffer, Nd4jLong *tadOnlyShapeInfo) {

    __shared__ UnifiedSharedMemory *manager;

    if (threadIdx.x == 0) {
        extern __shared__ unsigned char shmem[];
        manager = new(shmem) UnifiedSharedMemory((int *) shmem);
        manager->init(sizeof(UnifiedSharedMemory), 0, sizeof(functions::reduce::ReduceFloatFunction<X,Z>), sizeof(shape::TAD), 0);
    }
    __syncthreads();

    functions::reduce::ReduceFloatFunction<X, Z>::template execScalarCuda<OpType>(
            dx,
            xShapeInfo,
            extraParams,
            result,
            resultShapeInfo,
            reductionBuffer,
            manager,
            tadOnlyShapeInfo);
};

    template <typename X, typename Z, typename OpType>
    __global__ void _simpleScalar(
        void *dx,
        Nd4jLong *xShapeInfo,
        void *extraParams,
        void *result,
        Nd4jLong *resultShapeInfo,
        int *dimension,
        int dimensionLength,
        void *reductionBuffer, Nd4jLong *tadOnlyShapeInfo) {
            reduceScalarGeneric<X, Z, OpType>(dx, xShapeInfo, extraParams, result, resultShapeInfo, dimension, dimensionLength, reductionBuffer, tadOnlyShapeInfo);
        }

// // reduceScalar
// DISPATCH_KERNEL_SIMPLE(reduceScalarSimple_, reduceScalarGeneric, float, INPUT(float *x, Nd4jLong *xShapeInfo, float *extraParams, float *z, Nd4jLong *zShapeInfo, int *dimension, int dimensionLength, float *reductionBuffer, Nd4jLong *tadOnlyShapeInfo), PARAMS(x, xShapeInfo, extraParams, z, zShapeInfo, dimension, dimensionLength, reductionBuffer, tadOnlyShapeInfo), OPS_A(REDUCE_OPS))
// DISPATCH_KERNEL_SIMPLE(reduceScalarSimple_, reduceScalarGeneric, double, INPUT(double *x, Nd4jLong *xShapeInfo, double *extraParams, double *z, Nd4jLong *zShapeInfo, int *dimension, int dimensionLength, double *reductionBuffer, Nd4jLong *tadOnlyShapeInfo), PARAMS(x, xShapeInfo, extraParams, z, zShapeInfo, dimension, dimensionLength, reductionBuffer, tadOnlyShapeInfo), OPS_A(REDUCE_OPS))
// DISPATCH_KERNEL_SIMPLE(reduceScalarSimple_, reduceScalarGeneric, float16, INPUT(float16 *x, Nd4jLong *xShapeInfo, float16 *extraParams, float16 *z, Nd4jLong *zShapeInfo, int *dimension, int dimensionLength, float16 *reductionBuffer, Nd4jLong *tadOnlyShapeInfo), PARAMS(x, xShapeInfo, extraParams, z, zShapeInfo, dimension, dimensionLength, reductionBuffer, tadOnlyShapeInfo), OPS_A(REDUCE_OPS))

	template <typename X, typename Z, typename OpType>
	__global__ void _simpleReduce(
		void *dx,
		Nd4jLong *xShapeInfo,
		void *extraParams,
		void *result,
		Nd4jLong *resultShapeInfo,
		int *dimension,
		int dimensionLength,
		void *reductionBuffer, Nd4jLong *tadOnlyShapeInfo, Nd4jLong *tadOffsets) {
			reduceSimpleGeneric<X, Z, OpType>(dx, xShapeInfo, extraParams, result, resultShapeInfo, dimension, dimensionLength, reductionBuffer, tadOnlyShapeInfo, tadOffsets);
	}



// // reduceXD
// DISPATCH_KERNEL_SIMPLE(reduceSimpleGenericXD_, reduceSimpleGeneric, float, INPUT(float *x, Nd4jLong *xShape, float *extraParams, float *z, Nd4jLong *zShape, int *dimension, int dimensionLength, float *reductionPointer, Nd4jLong *tadShapeInfo, Nd4jLong *tadOffsets), PARAMS(x, xShape, extraParams, z, zShape, dimension, dimensionLength, reductionPointer, tadShapeInfo, tadOffsets), OPS_A(REDUCE_OPS))
// DISPATCH_KERNEL_SIMPLE(reduceSimpleGenericXD_, reduceSimpleGeneric, double, INPUT(double *x, Nd4jLong *xShape, double *extraParams, double *z, Nd4jLong *zShape, int *dimension, int dimensionLength, double *reductionPointer, Nd4jLong *tadShapeInfo, Nd4jLong *tadOffsets), PARAMS(x, xShape, extraParams, z, zShape, dimension, dimensionLength, reductionPointer, tadShapeInfo, tadOffsets), OPS_A(REDUCE_OPS))
// DISPATCH_KERNEL_SIMPLE(reduceSimpleGenericXD_, reduceSimpleGeneric, float16, INPUT(float16 *x, Nd4jLong *xShape, float16 *extraParams, float16 *z, Nd4jLong *zShape, int *dimension, int dimensionLength, float16 *reductionPointer, Nd4jLong *tadShapeInfo, Nd4jLong *tadOffsets), PARAMS(x, xShape, extraParams, z, zShape, dimension, dimensionLength, reductionPointer, tadShapeInfo, tadOffsets), OPS_A(REDUCE_OPS))


namespace functions {
    namespace reduce {

			template <typename X, typename Z>
			template<typename OpType>
			__host__ void ReduceFloatFunction<X,Z>::intermediateXD(dim3 launchDims, hipStream_t *stream, void *x, Nd4jLong *xShape, void *extraParams, void *z, Nd4jLong *zShape, int *dimension, int dimensionLength, void *reductionPointer, Nd4jLong *tadShapeInfo, Nd4jLong *tadOffsets) {
				_simpleReduce<X, Z, OpType><<<launchDims.x, launchDims.y, launchDims.z, stream>>>(x, xShape, extraParams, z, zShape, dimension, dimensionLength, reductionPointer, tadShapeInfo, tadOffsets);
			}

            template <typename X, typename Z>
            template<typename OpType>
            __host__ void ReduceFloatFunction<X,Z>::intermediateScalar(dim3 launchDims, hipStream_t *stream, void *x, Nd4jLong *xShapeInfo, void *extraParams, void *z, Nd4jLong *zShapeInfo, int *dimension, int dimensionLength, void *reductionBuffer, Nd4jLong *tadOnlyShapeInfo) {
                _simpleScalar<X, Z, OpType><<<launchDims.x, launchDims.y, launchDims.z, stream>>>(x, xShapeInfo, extraParams, z, zShapeInfo, dimension, dimensionLength, reductionBuffer, tadOnlyShapeInfo);
            }

			template <typename X, typename Y>
            _CUDA_H void ReduceFloatFunction<X,Y>::execReduceScalar(dim3 launchDims, hipStream_t *stream, int opNum, void *x, Nd4jLong *xShapeInfo, void *extraParams, void *z, Nd4jLong *zShapeInfo, int *dimension, int dimensionLength, void *reductionBuffer, Nd4jLong *tadOnlyShapeInfo) {
                DISPATCH_BY_OPNUM_TT(intermediateScalar, PARAMS(launchDims, stream, x, xShapeInfo, extraParams, z, zShapeInfo, dimension, dimensionLength, reductionBuffer, tadOnlyShapeInfo), OPS_A(REDUCE_FLOAT_OPS));

				nd4j::DebugHelper::checkErrorCode(stream, "execReduceScalarFloat(...) failed");
            }



            template <typename X, typename Y>
            _CUDA_H void ReduceFloatFunction<X, Y>::execReduceXD(dim3 launchDims, hipStream_t *stream, int opNum, int rank, void *x, Nd4jLong *xShape, void *extraParams, void *z, Nd4jLong *zShape, int *dimension, int dimensionLength, void *reductionPointer, Nd4jLong *tadShapeInfo, Nd4jLong *tadOffsets) {
                DISPATCH_BY_OPNUM_TT(intermediateXD, PARAMS(launchDims, stream, x, xShape, extraParams, z, zShape, dimension, dimensionLength, reductionPointer, tadShapeInfo, tadOffsets), OPS_A(REDUCE_FLOAT_OPS));

                DEBUG_KERNEL(stream, opNum);
            }

            template <typename T>
            __device__ void initializeShared(T *extraParams, T **sPartials, int sMemSize) {
                int sPartialsLength = sMemSize / sizeof(T);
                T *sPartialsDeref = (T *) *sPartials;
                for (int i = 0; i < sPartialsLength; i++) {
                    sPartialsDeref[i] = extraParams[0];
                }
            }

            template <typename X, typename Z>
            template <typename OpType>
			__device__ void ReduceFloatFunction<X,Z>::execScalarCuda(
				void *vdx,
				Nd4jLong *xShapeInfo,
				void *vextraParams,
				void *vresult,
				Nd4jLong *resultShapeInfo,
				void *vreductionBuffer,
				UnifiedSharedMemory *manager,
				Nd4jLong *tadOnlyShapeInfo) {

                auto dx = reinterpret_cast<X*>(vdx);
                auto result = reinterpret_cast<Z*>(vresult);
                auto extraParams = reinterpret_cast<Z*>(vextraParams);
                auto reductionBuffer = reinterpret_cast<Z*>(vreductionBuffer);

				int elementWiseStride = shape::elementWiseStride(xShapeInfo);

				auto n = shape::length(xShapeInfo);

				auto tid = blockDim.x * blockIdx.x + threadIdx.x;

				//shared memory space for storing intermediate results
				Z *sPartials = reinterpret_cast<Z*>(manager->getSharedReductionBuffer());

				sPartials[threadIdx.x] = OpType::startingValue(dx);

				if (elementWiseStride >= 1) {
					for (int i = tid; i < n; i += (blockDim.x * gridDim.x)) {
						sPartials[threadIdx.x] = OpType::update(sPartials[threadIdx.x], OpType::op(dx[i * elementWiseStride], extraParams), extraParams);
					}
				}
				else {
				    __shared__ int rank;
				    __shared__ Nd4jLong *xShape;
				    __shared__ Nd4jLong *xStride;
				    if (threadIdx.x == 0) {
                        rank = shape::rank(xShapeInfo);
                        xShape = shape::shapeOf(xShapeInfo);
                        xStride = shape::stride(xShapeInfo);
				    }
				    __syncthreads();

					Nd4jLong ind2sub[MAX_RANK];

					for (int i = tid; i < n; i += blockDim.x * gridDim.x) {
						shape::ind2subC(rank, xShape, i, n, ind2sub);

						auto offset = shape::getOffset(0, xShape, xStride, ind2sub, rank);
						sPartials[threadIdx.x] = OpType::update(sPartials[threadIdx.x], OpType::op(dx[offset], extraParams), extraParams);
					}
				}

				__syncthreads();
				aggregatePartials<OpType>(sPartials, threadIdx.x, nd4j::math::nd4j_min<int>(blockDim.x, n), extraParams);


				__syncthreads();

				if (gridDim.x > 1) {
					unsigned int *tc = (unsigned int *)reductionBuffer;
					__shared__ bool amLast;
					tid = threadIdx.x;
					if (threadIdx.x == 0) {
						reductionBuffer[blockIdx.x] = sPartials[0];//this->postProcess(sPartials[0],n,extraParams);
					}
					__threadfence();
					__syncthreads();

					if (threadIdx.x == 0) {
						unsigned int ticket = atomicInc(&tc[16384], gridDim.x);
						amLast = (ticket == gridDim.x - 1);
					}

					__syncthreads();

					if (amLast) {
						tc[16384] = 0;

						sPartials[threadIdx.x] = OpType::startingValue(dx);

						for (int i = threadIdx.x; i < gridDim.x; i += blockDim.x) {
							sPartials[threadIdx.x] = OpType::update(sPartials[threadIdx.x], reductionBuffer[i], extraParams);
						}
						__syncthreads();



						aggregatePartials<OpType>(sPartials, threadIdx.x, nd4j::math::nd4j_min<int>(gridDim.x, blockDim.x), extraParams);

						__syncthreads();
						if (threadIdx.x == 0) {
							result[0] = OpType::postProcess(sPartials[0], n, extraParams);
						}
					}
				}
				else {
					if (threadIdx.x == 0) {
						unsigned int *tc = (unsigned *)reductionBuffer;
						tc[16384] = 0;
						result[0] = OpType::postProcess(sPartials[0], n, extraParams);
					}
				}
			}


            template <typename X, typename Z>
            template <typename OpType>
			__device__ void ReduceFloatFunction<X, Z>::transformCudaXD(
				void *vdx,
				Nd4jLong *xShapeInfo,
				void *vextraParams,
				void *vresult,
				Nd4jLong *resultShapeInfo,
				int *dimension,
				int dimensionLength,
				void *vreductionBuffer,
				UnifiedSharedMemory *manager,
				Nd4jLong *tadOnlyShapeInfo,
				Nd4jLong *tadOffsets) {

                auto dx = reinterpret_cast<X*>(vdx);
                auto result = reinterpret_cast<Z*>(vresult);
                auto extraParams = reinterpret_cast<Z*>(vextraParams);
                auto reductionBuffer = reinterpret_cast<Z*>(vreductionBuffer);


                if (OpType::requiresSpecialAccumulation) {
                    OpType::execSpecialCuda(dx, xShapeInfo, extraParams, result, resultShapeInfo, dimension, dimensionLength, reductionBuffer, manager, tadOnlyShapeInfo, tadOffsets);
                    return;
                }

				//shared memory space for storing intermediate results
				__shared__ Z *sPartials;

				//                __shared__ shape::TAD *tad;
				__shared__ int tadLength;
				__shared__ int tadRank;
				__shared__ int numTads;
				__shared__ Nd4jLong *tadShape;
				__shared__ Nd4jLong *tadStride;
				if (threadIdx.x == 0) {
				    extern __shared__ unsigned char shmem[];
				    sPartials = reinterpret_cast<Z*>(shmem);
					tadLength = shape::tadLength(xShapeInfo, dimension, dimensionLength);
					tadRank = shape::rank(tadOnlyShapeInfo);
					numTads = shape::length(xShapeInfo) / tadLength;

					tadShape = shape::shapeOf(tadOnlyShapeInfo);
					tadStride = shape::stride(tadOnlyShapeInfo);
				}
				__syncthreads();

				Nd4jLong xCoord[MAX_RANK];

				for (int r = blockIdx.x; r < numTads; r += gridDim.x) {
					Nd4jLong tadOffsetForBlock = tadOffsets[r];

					sPartials[threadIdx.x] = OpType::startingValue(dx + tadOffsetForBlock);

					for (int i = threadIdx.x; i < tadLength; i += blockDim.x) {
						shape::ind2subC(tadRank, tadShape, i, tadLength, xCoord);
						auto xOffset = shape::getOffset(tadOffsetForBlock, tadShape, tadStride, xCoord, tadRank);

						sPartials[threadIdx.x] = OpType::update(sPartials[threadIdx.x], OpType::op(dx[xOffset], extraParams), extraParams);
					}
					__syncthreads();

					// aggregate. do NOT reduce for elements > tadLength
					aggregatePartials<OpType>(sPartials, threadIdx.x, nd4j::math::nd4j_min<int>(blockDim.x, tadLength), extraParams);


					__syncthreads();
					if (threadIdx.x == 0)
						result[r] = OpType::postProcess(sPartials[threadIdx.x], tadLength, extraParams);
				}
			}

            template <typename X, typename Z>
            template <typename OpType>
			__device__ void ReduceFloatFunction<X, Z>::aggregatePartials(void *vsPartials, Nd4jLong tid, Nd4jLong numItems, void *vextraParams) {
				// start the shared memory loop on the next power of 2 less
				// than the block size.  If block size is not a power of 2,
				// accumulate the intermediate sums in the remainder range.
                auto sPartials = static_cast<Z*>(vsPartials);
                auto extraParams = static_cast<Z*>(vextraParams);

				Nd4jLong floorPow2 = numItems;

				if (floorPow2 & (floorPow2 - 1)) {
					while (floorPow2 & (floorPow2 - 1)) {
						floorPow2 &= floorPow2 - 1;
					}
					if (tid >= floorPow2) {
						sPartials[tid - floorPow2] = OpType::update(sPartials[tid - floorPow2], sPartials[tid], extraParams);
					}

					__syncthreads();
				}


				for (Nd4jLong activeThreads = floorPow2 >> 1; activeThreads; activeThreads >>= 1) {
					if (tid < activeThreads && tid + activeThreads < numItems) {
						sPartials[tid] = OpType::update(sPartials[tid], sPartials[tid + activeThreads], extraParams);
					}
                    __syncthreads();
				}
			}


        BUILD_DOUBLE_TEMPLATE(template class ND4J_EXPORT ReduceFloatFunction, , LIBND4J_TYPES, FLOAT_TYPES);
    }
}