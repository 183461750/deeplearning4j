#include "hip/hip_runtime.h"
/*******************************************************************************
 * Copyright (c) 2015-2018 Skymind, Inc.
 *
 * This program and the accompanying materials are made available under the
 * terms of the Apache License, Version 2.0 which is available at
 * https://www.apache.org/licenses/LICENSE-2.0.
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS, WITHOUT
 * WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied. See the
 * License for the specific language governing permissions and limitations
 * under the License.
 *
 * SPDX-License-Identifier: Apache-2.0
 ******************************************************************************/

//
// @author raver119@gmail.com
//

#include <Environment.h>
#include <loops/transform_same.h>
#include <types/types.h>
#include <op_boilerplate.h>

#include <loops/legacy_ops.h>
#include <helpers/DebugHelper.h>

using namespace simdOps;


template<typename X, typename OpClass>
__device__ void transformSameSimpleGeneric(
		Nd4jLong n,
		void *dy,
		Nd4jLong incy,
		void *params,
		void *result,
		Nd4jLong resultStride, int *allocationPointer, void *reductionPointer) {

	functions::transform::TransformSame<X>::template transformCuda<OpClass>(
		n,
		dy,
		incy,
		params,
		result,
		resultStride,
		allocationPointer,
		reductionPointer,
		nullptr);
}

template<typename X, typename OpClass>
__device__ void transformSameSimpleGeneric(
		void *dy,
		Nd4jLong *xShapeInfo, int xRank,
		void *params,
		void *result, Nd4jLong *resultShapeInfo, int zRank, int *allocationPointer, void *reductionPointer, Nd4jLong *tadShapeInfo, Nd4jLong *tadOffsets) {

	__shared__ UnifiedSharedMemory *manager;

	if (threadIdx.x == 0) {
		extern __shared__ unsigned char shmem[];
		manager = new(shmem) UnifiedSharedMemory((int *) shmem);
		manager->init(sizeof(UnifiedSharedMemory), 0, sizeof(functions::transform::TransformSame<X>), sizeof(shape::TAD), xRank);
	}
	__syncthreads();
	
    functions::transform::TransformSame<X>::template transformCuda<OpClass>(
	    dy,
	    xShapeInfo,
	    params,
	    result,
	    resultShapeInfo,
	    allocationPointer,
	    reductionPointer,
		manager, tadShapeInfo, tadOffsets);
}


template <typename X, typename OpType>
__global__ void transformSameSimple(void *dy, Nd4jLong *xShapeInfo, int xRank,
								void *params,
								void *result, Nd4jLong *resultShapeInfo, int zRank,
								int *allocationPointer,
								void *reductionPointer,
								Nd4jLong *tadShapeInfo, Nd4jLong *tadOffsets) {
	transformSameSimpleGeneric<X, OpType>(dy, xShapeInfo, xRank, params, result, resultShapeInfo, zRank, allocationPointer, reductionPointer, tadShapeInfo, tadOffsets);
}


namespace functions {
    namespace transform {

        template<typename X>
        _CUDA_H void TransformSame<X>::executeTransformShaped(dim3 launchDims, hipStream_t *stream, int opNum, void *x, Nd4jLong *xShape, int xRank, void *extraParams, void *z, Nd4jLong *zShape, int zRank, int *allocationPointer, void *reductionPointer,  Nd4jLong *tadShapeInfo, Nd4jLong *tadOffsets) {
			DISPATCH_BY_OPNUM_T(intermediateShaped, PARAMS(launchDims, stream, x, xShape, xRank, extraParams, z, zShape, zRank, allocationPointer, reductionPointer, tadShapeInfo, tadOffsets), TRANSFORM_SAME_OPS);

            DEBUG_KERNEL(stream, opNum);
        }


        template<typename X>
        template <typename OpType>
        __device__ void TransformSame<X>::transformCuda(
			void *vdy,
			Nd4jLong *shapeInfo,
			void *vparams,
			void *vresult,
			Nd4jLong *resultShapeInfo,
			int *allocationPointer, void *vreductionPointer, UnifiedSharedMemory *manager, Nd4jLong *tadShapeInfo, Nd4jLong *tadOffsets) {

        	auto dy = static_cast<X*>(vdy);
		    auto result = static_cast<X*>(vresult);
		    auto params = static_cast<X*>(vparams);
		    auto reductionPointer = static_cast<X*>(vreductionPointer);

		    if(OpType::requiresSpecial) {
			    OpType::execSpecialCuda(dy,shapeInfo,result,resultShapeInfo,params, allocationPointer, reductionPointer, manager, tadShapeInfo, tadOffsets);
			    return;
		    } else {

    		    auto xShape = shape::shapeOf(shapeInfo);
	    	    auto xStride = shape::stride(shapeInfo);
		        auto xOrder = shape::order(shapeInfo);
		        auto resultOrder = shape::order(resultShapeInfo);
    		    auto xRank = shape::rank(shapeInfo);

		        auto xElementWiseStride = shape::elementWiseStride(shapeInfo);
    		    auto resultElementWiseStride = shape::elementWiseStride(resultShapeInfo);
	    	    auto tid = blockIdx.x * blockDim.x + threadIdx.x;

                __shared__ Nd4jLong length;
		        if(threadIdx.x == 0)
			        length = shape::length(shapeInfo);
		        __syncthreads();

		        if(xElementWiseStride >= 1 && resultElementWiseStride >= 1 && xOrder == resultOrder) {
			        transformCuda<OpType>(
				    	length,
				    	dy,
				    	xElementWiseStride,
				    	params,
				    	result,
				    	resultElementWiseStride, allocationPointer, reductionPointer, manager);
		        }
		        else {
			        Nd4jLong xCoord[MAX_RANK];
			
		    	    for (Nd4jLong i = tid; i < length; i+= gridDim.x * blockDim.x) {
						shape::ind2subC(xRank,shape::shapeOf(shapeInfo),i, length, xCoord);
						
				        auto xOffset2 = shape::getOffset(0, xShape, xStride, xCoord, xRank);
						auto resultOffset2 = shape::getOffset(0,xShape,shape::stride(resultShapeInfo),xCoord,xRank);
						
	    			    result[resultOffset2] = OpType::op(dy[xOffset2], params);
		    	    }
		        }
	        }
	    };

        template<typename X>
        template <typename OpType>
	    __device__ void TransformSame<X>::transformCuda(
			Nd4jLong n,
			void *vdy,
			Nd4jLong incy,
			void *vparams,
			void *vresult,
			Nd4jLong resultStride,
			int *allocationPointer, void *vreductionPointer, UnifiedSharedMemory *manager) {
		
        	auto dy = static_cast<X*>(vdy);
		    auto result = static_cast<X*>(vresult);
		    auto params = static_cast<X*>(vparams);
		    auto reductionPointer = static_cast<X*>(vreductionPointer);

            int totalThreads = gridDim.x * blockDim.x;
		    Nd4jLong i = blockIdx.x * blockDim.x + threadIdx.x;

    		if(incy == 1 && resultStride == 1) {
	    		/* equal, positive, non-unit increments. */
			    for (; i < n; i += totalThreads) {
				    result[i] = OpType::op(dy[i], params);
			    }
		    }
		    else {
			    for (; i < n; i += totalThreads) {
				    result[i * resultStride] = OpType::op(dy[i * incy], params);
			    }
		    }
	    }


		template<typename X>
		template <typename OpType>
		_CUDA_H void TransformSame<X>::intermediateShaped(dim3 launchDims, hipStream_t *stream, void *x, Nd4jLong *xShape, int xRank, void *extraParams, void *z, Nd4jLong *zShape, int zRank, int *allocationPointer, void *reductionPointer,  Nd4jLong *tadShapeInfo, Nd4jLong *tadOffsets) {
			transformSameSimple<X, OpType><<<launchDims.x, launchDims.y, launchDims.z, stream>>>(x, xShape, xRank, extraParams, z, zShape, zRank, allocationPointer, reductionPointer, tadShapeInfo, tadOffsets);
		}

        BUILD_SINGLE_TEMPLATE(template class ND4J_EXPORT TransformSame, , LIBND4J_TYPES);
    }
}
