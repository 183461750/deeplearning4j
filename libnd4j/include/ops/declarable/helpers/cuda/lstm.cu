#include "hip/hip_runtime.h"
/*******************************************************************************
 * Copyright (c) 2015-2018 Skymind, Inc.
 *
 * This program and the accompanying materials are made available under the
 * terms of the Apache License, Version 2.0 which is available at
 * https://www.apache.org/licenses/LICENSE-2.0.
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS, WITHOUT
 * WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied. See the
 * License for the specific language governing permissions and limitations
 * under the License.
 *
 * SPDX-License-Identifier: Apache-2.0
 ******************************************************************************/

//
// @author Yurii Shyrma, created on 14.02.2018
//

// implementation of operation for LSTM cell with peep hole connections:
// http://www.bioinf.jku.at/publications/older/2604.pdf
// S. Hochreiter and J. Schmidhuber. "Long Short-Term Memory". Neural Computation, 9(8):1735-1780, 1997.
// and 
// https://research.google.com/pubs/archive/43905.pdf
// Hasim Sak, Andrew Senior, and Francoise Beaufays. "Long short-term memory recurrent neural network architectures for large scale acoustic modeling." INTERSPEECH, 2014.


#include<ops/declarable/helpers/lstm.h>
#include<ops/declarable/helpers/lstmBlock.h>
#include <ops/declarable/CustomOperations.h>
#include<ops/declarable/helpers/transforms.h>
#include <array/NDArrayList.h>
#include <iterator>

namespace nd4j 	  {
namespace ops 	  {
namespace helpers {


//////////////////////////////////////////////////////////////////////////
static FORCEINLINE NDArray sigmoid(const NDArray& arr) {
    return (const_cast<NDArray&>(arr)).transform(transform::Sigmoid);
}

//////////////////////////////////////////////////////////////////////////
static FORCEINLINE NDArray activation(const NDArray& arr) {
    
    return (const_cast<NDArray&>(arr)).transform(transform::Tanh);
}

//////////////////////////////////////////////////////////////////////////
template <typename T>
static void clipping(NDArray* arr, T limit) {
    
    if(limit < (T)0.f)
        limit *= (T)(-1.f);

    /*
    auto clip = LAMBDA_T(value, limit) {
        if(value < -limit || value > limit)
            value = limit;
        return value; 
    };

    arr->applyLambda(clip);
    */
    arr->applyScalar(scalar::LstmClip, limit);
}

//////////////////////////////////////////////////////////////////////////
void lstmCell(graph::LaunchContext* context, const NDArray* xt, const NDArray* ht_1, const NDArray* ct_1, const NDArray* Wx, const NDArray* Wh, const NDArray* Wc, const NDArray* Wp, const NDArray* b,
              NDArray* ht, NDArray* ct, const std::vector<double>& params) {
}


//////////////////////////////////////////////////////////////////////////
static NDArray* timeSubset(const NDArray* arr, const int t, const int dataFormat){
    if(dataFormat == 0){
        //TNS: shape [timeLength, numExamples, inOutSize]
        auto x = (*arr)({t,t+1, 0,0, 0,0});
        const std::vector<Nd4jLong> newShape({arr->sizeAt(1),arr->sizeAt(2)});
        return x.reshape(arr->ordering(), newShape);
    } else if(dataFormat == 1){
        //NST: shape [numExamples, inOutSize, timeLength]
        auto x = (*arr)({0,0, 0,0, t,t+1});
        const std::vector<Nd4jLong> newShape({arr->sizeAt(0),arr->sizeAt(1)});
        return x.reshape(arr->ordering(), newShape);
    } else {
        //NTS: shape [numExamples, timeLength, inOutSize] - TF "time_major=false" layout
        auto x = (*arr)({0,0, t,t+1, 0,0});
        const std::vector<Nd4jLong> newShape({arr->sizeAt(0),arr->sizeAt(2)});
        return x.reshape(arr->ordering(), newShape);
    }
}

//////////////////////////////////////////////////////////////////////////
void lstmTimeLoop(graph::LaunchContext* context, const NDArray* x, const NDArray* h0, const NDArray* c0, const NDArray* Wx, const NDArray* Wh, const NDArray* Wc, const NDArray* Wp, const NDArray* b,
                  NDArray* h, NDArray* c, const std::vector<double>& params) {

}


    void lstmBlockCell(const NDArray* xt, const NDArray* cLast, const NDArray* yLast,
                       const NDArray* W, const NDArray* Wci, const NDArray* Wcf, const NDArray* Wco, const NDArray* b,
                       NDArray* i, NDArray* c, NDArray* f, NDArray* o, NDArray* z, NDArray* h, NDArray* y, const std::vector<double>& params) {
        /* Input arrays:
        *    0: xt              - input [bS, inSize] at time t
        *    1: cLast (cs_prev) - previous cell state  [bS, numUnits], time t-1
        *    2: yLast (h_prev)  - previous output [bS, numUnits], time t-1
        *    3: W               - Weights - concatenated (input-to-hidden, hidden-to-hidden weights)  weights, [(inSize+numUnits), 4*numUnits]
        *    4: Wci             - weights - cell peephole (t-1) connections to input modulation gate, [numUnits]
        *    5: Wcf             - weights - cell peephole (t-1) connections to forget gate, [numUnits]
        *    6: Wco             - weights - cell peephole (t) connections to output gate, [numUnits]
        *    7: b               - biases, [4*numUnits]
        *
        *  Input integer arguments:
        *    0: if not zero, provide peephole connections
        *
        *  Input float arguments:
        *    0: the bias added to forget gates in order to reduce the scale of forgetting in the beginning of the training
        *    1: clipping value for cell state, if it is not equal to zero, then cell state is clipped
        *
        * Output arrays:
        *    0: i      - Input modulation gate activations [bS, numUnits]
        *    1: c (cs) - Cell state (pre tanh) [bs, numUnits] (cs)
        *    2: f      - Output - forget gate activations [bs, numUnits]
        *    3: o      - Output - output gate activations [bs, numUnits]
        *    4: z (ci) - Output - block input [bs, numUnits]
        *    5: h (co) - Cell state, post tanh [bs, numUnits]
        *    6: y (h)  - Current cell output [bS, numUnits], time t
        */
        const bool peephole   = (bool)params[0];        // if true, provide peephole connections
        const double forgetBias    = params[1];
        const double clippingCellValue   = params[2];              // clipping value for ct, if it is not equal to zero, then cell state is clipped


        const int bS   = xt->sizeAt(0);
        const int inSize      = xt->sizeAt(1);
        const int numUnits    = cLast->sizeAt(1);

        nd4j_printf("Batch: %lld, inSize: %lld, numUnits: %lld\n", bS, inSize, numUnits);

        //Concat inputs: [xt, yt-1]: concat([bs,nIn],[bs,nOut]) -> [bs, (nIn+nOut)]
        auto concatStart = std::chrono::system_clock::now();
        nd4j::ops::concat concat;
        Context cContext(119);
        auto concatOut = NDArrayFactory::create(xt->ordering(), {xt->sizeAt(0), xt->sizeAt(1) + yLast->sizeAt(1)}, xt->dataType(), xt->getContext());
        cContext.setInputArray(0, const_cast<NDArray*>(xt), false);
        cContext.setInputArray(1, const_cast<NDArray*>(yLast), false);
        cContext.setOutputArray(0, &concatOut, false);
        cContext.getIArguments()->emplace_back(1);

        concat.execute(&cContext);
        auto concatEnd = std::chrono::system_clock::now();

        auto mmulStart = std::chrono::system_clock::now();
        auto m = mmul(concatOut, *W);    //mmul: [bs, (nIn+numUnits)]* [(inSize+numUnits), 4*numUnits] = [bs, 4*numUnits]
        auto mmulEnd = std::chrono::system_clock::now();
        auto bAddStart = std::chrono::system_clock::now();
        m += (*b);
        auto bAddEnd = std::chrono::system_clock::now();

        //Note: weights are ordered [inputGate, blockInput, forgetGate, outputGate] to match TF (TF code comments state [i,f,z/ci,o] but behaviour is [i,z,f,o])
        auto zi = m({0,0, 0,            numUnits});      	// z for input modulation gate, [bS, numUnits]
        auto zz = m({0,0, numUnits, 2*numUnits});      	    // z for block input, [bS, numUnits]
        auto zf = m({0,0, 2*numUnits, 3*numUnits});      	// z for forget gate, [bS, numUnits]
        auto zo = m({0,0, 3*numUnits, 4*numUnits});      	// z for output gate, [bS, numUnits]

        if(peephole) {                                              // add peephole connections: z  +  ct_1*Wc
            zi += (*cLast) * (*Wci);       // add peephole connections to input gate
            zf += (*cLast) * (*Wcf);       // add peephole connections to forget gate
        }

        // current sell state = ft*cLast + it*tanh(mmul(Wxc,xt) + mmul(Whc,ht_1) + bc
        auto fbStart = std::chrono::system_clock::now();
        if(forgetBias != 0.0){
            zf += forgetBias;
        }
        auto fbEnd = std::chrono::system_clock::now();

        auto actTanhStart = std::chrono::system_clock::now();
        zz.applyTransform(transform::Tanh, z);      //z = tanh(zz)
        auto actTanhEnd = std::chrono::system_clock::now();
        auto actSigmoidStart = std::chrono::system_clock::now();
        zi.applyTransform(transform::Sigmoid, i);   //i = sigmoid(zi)
        zf.applyTransform(transform::Sigmoid, f);   //f = sigmoid(zf);
        auto actSigmoidEnd = std::chrono::system_clock::now();


        //cell state = blockInput .* inputGate + prevCellState .* forgetGate
        auto cellStateStart = std::chrono::system_clock::now();
        z->applyPairwiseTransform(pairwise::Multiply, i, c, nullptr);       //c = z * i
        auto temp = (*f) * (*cLast);
        *c += temp;                              //c = (i * z) + (zf * (*cLast))
        c->applyTransform(transform::Tanh, h);  //h = tanh(c)
        auto cellStateEnd = std::chrono::system_clock::now();


        // if clipping value is provided then cell state is clipped by this value prior to the cell output activation
        if(clippingCellValue > 0.0) {
            clipping(c, clippingCellValue);
        }

        if(peephole) {
            // add peephole connections to output gate zot + ct*Wc
            auto prod = *c * (*Wco);
            zo += prod;
        }
        zo.applyTransform(transform::Sigmoid, o);   // o = sigmoid(zo)

        // current cell output = ot*tanh(ct)
        c->applyTransform(transform::Tanh, h);  //h = tanh(c)
        auto mulStart = std::chrono::system_clock::now();
        o->applyPairwiseTransform(pairwise::Multiply, h, y, nullptr);   //y = o * h
        auto mulEnd = std::chrono::system_clock::now();

        auto concatTime = std::chrono::duration_cast<std::chrono::microseconds> ((concatEnd - concatStart)).count();
        auto mmulTime = std::chrono::duration_cast<std::chrono::microseconds> ((mmulEnd - mmulStart)).count();
        auto bAddTime = std::chrono::duration_cast<std::chrono::microseconds> ((bAddEnd - bAddStart)).count();
        auto phTime = std::chrono::duration_cast<std::chrono::microseconds> ((phEnd - phStart)).count();
        auto actTanhTime = std::chrono::duration_cast<std::chrono::microseconds> ((actTanhEnd - actTanhStart)).count();
        auto actSigmoidTime = std::chrono::duration_cast<std::chrono::microseconds> ((actSigmoidEnd - actSigmoidStart)).count();
        auto cellStateTime = std::chrono::duration_cast<std::chrono::microseconds> ((cellStateEnd - cellStateStart)).count();
        auto mulTime = std::chrono::duration_cast<std::chrono::microseconds> ((mulEnd - mulStart)).count();

        nd4j_printf("concat=%lld, mmul=%lld, bAdd=%lld, ph=%lld, actTanh=%lld, actSigmoid=%lld, cell=%lld, mul=%lld\n",concatTime, mmulTime, bAddTime, phTime,
                    actTanhTime, actSigmoidTime, cellStateTime, mulTime);
    }


    void lstmBlockTimeLoop(const NDArray* maxSeqLength, const NDArray* xSeq, const NDArray* c0, const NDArray* y0,
                           const NDArray* W, const NDArray* Wci, const NDArray* Wcf, const NDArray* Wco, const NDArray* b,
                           const NDArray* iSeq, const NDArray* cSeq, const NDArray* fSeq, const NDArray* oSeq, const NDArray* zSeq,
                           const NDArray* hSeq, const NDArray* ySeq, const std::vector<double>& params, const int dataFormat) {

        const int seqLen = xSeq->sizeAt(0);
        const int mb = xSeq->sizeAt(1);
        const int inSize = xSeq->sizeAt(2);
        const int outSize = iSeq->sizeAt(2);

        const std::vector<Nd4jLong> inSliceShape({mb,inSize});
        const std::vector<Nd4jLong> outSliceShape({mb,outSize});

        NDArray* c_t1 = const_cast<NDArray*>(c0);
        NDArray* y_t1 = const_cast<NDArray*>(y0);

        // loop through time steps
        for (int t = 0; t <seqLen; ++t) {
            auto xt = timeSubset(xSeq, t, dataFormat);

            auto it = timeSubset(iSeq, t, dataFormat);
            auto ct = timeSubset(cSeq, t, dataFormat);
            auto ft = timeSubset(fSeq, t, dataFormat);
            auto ot = timeSubset(oSeq, t, dataFormat);
            auto zt = timeSubset(zSeq, t, dataFormat);
            auto ht = timeSubset(hSeq, t, dataFormat);
            auto yt = timeSubset(ySeq, t, dataFormat);

            nd4j::ops::helpers::lstmBlockCell(xt, c_t1, y_t1, W, Wci, Wcf, Wco, b, it, ct, ft, ot, zt, ht, yt, params);

            c_t1 = ct;
            y_t1 = yt;
        }
    }
}
}
}

